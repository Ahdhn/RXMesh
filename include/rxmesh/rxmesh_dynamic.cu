#include "hip/hip_runtime.h"
#include "rxmesh/rxmesh_dynamic.h"

namespace rxmesh {

namespace detail {

__global__ static void calc_num_elements(const Context context,
                                         uint32_t*     sum_num_vertices,
                                         uint32_t*     sum_num_edges,
                                         uint32_t*     sum_num_faces)
{
    uint32_t thread_id = threadIdx.x + blockIdx.x * gridDim.x;

    if (thread_id < context.get_num_patches()) {
        ::atomicAdd(
            sum_num_vertices,
            uint32_t(context.get_patches_info()[thread_id].num_owned_vertices));

        ::atomicAdd(
            sum_num_edges,
            uint32_t(context.get_patches_info()[thread_id].num_owned_edges));

        ::atomicAdd(
            sum_num_faces,
            uint32_t(context.get_patches_info()[thread_id].num_owned_faces));
    }
}

__global__ static void check_uniqueness(const Context context,
                                        uint32_t*     d_check)
{
}

}  // namespace detail

bool RXMeshDynamic::validate()
{
    CUDA_ERROR(hipDeviceSynchronize());

    // check that the sum of owned vertices, edges, and faces per patch is equal
    // to the number of vertices, edges, and faces respectively.

    auto check_num_mesh_elements = [&]() -> bool {
        uint32_t *d_sum_num_vertices, *d_sum_num_edges, *d_sum_num_faces;
        CUDA_ERROR(hipMalloc((void**)&d_sum_num_vertices, sizeof(uint32_t)));
        CUDA_ERROR(hipMalloc((void**)&d_sum_num_edges, sizeof(uint32_t)));
        CUDA_ERROR(hipMalloc((void**)&d_sum_num_faces, sizeof(uint32_t)));

        CUDA_ERROR(hipMemset(d_sum_num_vertices, 0, sizeof(uint32_t)));
        CUDA_ERROR(hipMemset(d_sum_num_edges, 0, sizeof(uint32_t)));
        CUDA_ERROR(hipMemset(d_sum_num_faces, 0, sizeof(uint32_t)));


        uint32_t num_patches;
        CUDA_ERROR(hipMemcpy(&num_patches,
                              m_rxmesh_context.m_num_patches,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        const uint32_t block_size = 256;
        const uint32_t grid_size  = DIVIDE_UP(num_patches, block_size);

        detail::calc_num_elements<<<grid_size, block_size>>>(m_rxmesh_context,
                                                             d_sum_num_vertices,
                                                             d_sum_num_edges,
                                                             d_sum_num_faces);

        uint32_t num_vertices, num_edges, num_faces;
        CUDA_ERROR(hipMemcpy(&num_vertices,
                              m_rxmesh_context.m_num_vertices,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&num_edges,
                              m_rxmesh_context.m_num_edges,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&num_faces,
                              m_rxmesh_context.m_num_faces,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        uint32_t sum_num_vertices, sum_num_edges, sum_num_faces;

        CUDA_ERROR(hipMemcpy(&sum_num_vertices,
                              d_sum_num_vertices,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&sum_num_edges,
                              d_sum_num_edges,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&sum_num_faces,
                              d_sum_num_faces,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipFree(d_sum_num_vertices));
        CUDA_ERROR(hipFree(d_sum_num_edges));
        CUDA_ERROR(hipFree(d_sum_num_faces));

        if (num_vertices != sum_num_vertices || num_edges != sum_num_edges ||
            num_faces != sum_num_faces) {
            return false;
        } else {
            return true;
        }
    };

    // check that each edge is composed of two unique vertices and each face is
    // composed of three unique edges
    auto check_uniqueness = [&]() -> bool {
        uint32_t num_patches;
        CUDA_ERROR(hipMemcpy(&num_patches,
                              m_rxmesh_context.m_num_patches,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        uint32_t* d_check;
        CUDA_ERROR(hipMalloc((void**)&d_check, sizeof(uint32_t)));
        CUDA_ERROR(hipMemset(d_check, 0, sizeof(uint32_t)));

        const uint32_t block_size   = 256;
        const uint32_t grid_size    = num_patches;
        const uint32_t dynamic_smem = (3 * this->m_max_faces_per_patch +
                                       2 * this->m_max_edges_per_patch) *
                                      sizeof(uint16_t);

        detail::check_uniqueness<<<grid_size, block_size, dynamic_smem>>>(
            m_rxmesh_context, d_check);

        uint32_t h_check;
        CUDA_ERROR(hipMemcpy(
            &h_check, d_check, sizeof(uint32_t), hipMemcpyDeviceToHost));

        if (h_check != 0) {
            return false;
        } else {
            return true;
        }
    };

    if (!check_num_mesh_elements()) {
        return false;
    }

    if (!check_uniqueness()) {
        return false;
    }

    return true;
}
}  // namespace rxmesh