#include "hip/hip_runtime.h"
#include <numeric>

#include <hip/hip_cooperative_groups.h>

#include "rxmesh/bitmask.cuh"
#include "rxmesh/kernels/dynamic_util.cuh"
#include "rxmesh/kernels/for_each.cuh"
#include "rxmesh/kernels/loader.cuh"
#include "rxmesh/kernels/shmem_allocator.cuh"
#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_dynamic.h"
#include "rxmesh/util/bitmask_util.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/util.h"

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace rxmesh {

namespace detail {
template <uint32_t blockThreads, typename HandleT>
__device__ __inline__ void hashtable_calibration(const Context context,
                                                 PatchInfo&    pi)
{
    // TODO cleanup patch stash
    // TODO load the hashtable in shared memory
    // TODO cleanup the hash table for stall elements
    using LocalT = typename HandleT::LocalT;

    const uint16_t num_elements = *(pi.get_num_elements<HandleT>());

    const uint16_t num_elements_up =
        ROUND_UP_TO_NEXT_MULTIPLE(num_elements, blockThreads);

    for (uint16_t i = threadIdx.x; i < num_elements_up; i += blockThreads) {
        HandleT handle;
        bool    replace = false;

        // int probe = 0;

        if (i < num_elements) {

            if (!pi.is_owned(LocalT(i)) && !pi.is_deleted(LocalT(i))) {

                // This is the same implementation in
                // Context::get_owner_handle()

                uint32_t owner = pi.patch_id;
                uint16_t lid   = i;

                LPPair lp = pi.get_lp<HandleT>().find(lid);

                assert(!lp.is_sentinel());

                owner = pi.patch_stash.get_patch(lp);

                assert(owner != INVALID32);

                // This only happen when the element i resides in the cavity of
                // the owner where it will be cleaned up later in
                // remove_surplus_elements
                if (!context.m_patches_info[owner].is_deleted(
                        LocalT(lp.local_id_in_owner_patch()))) {

                    // assert(!context.m_patches_info[owner].is_deleted(
                    //    LocalT(lp.local_id_in_owner_patch())));


                    while (!context.m_patches_info[owner].is_owned(
                        LocalT(lp.local_id_in_owner_patch()))) {

                        // probe++;

                        replace = true;

                        lp = context.m_patches_info[owner]
                                 .get_lp<HandleT>()
                                 .find(lp.local_id_in_owner_patch());

                        assert(!lp.is_sentinel());

                        owner =
                            context.m_patches_info[owner].patch_stash.get_patch(
                                lp);

                        if (context.m_patches_info[owner].is_deleted(
                                LocalT(lp.local_id_in_owner_patch()))) {
                            replace = false;
                            // printf("\n probe = %d, p= %u, type= %s, owner=
                            // %u",
                            //       probe,
                            //       pi.patch_id,
                            //       LocalT::name(),
                            //       owner);
                            break;
                        }
                        assert(!context.m_patches_info[owner].is_deleted(
                            LocalT(lp.local_id_in_owner_patch())));
                    }

                    handle = HandleT(owner, lp.local_id_in_owner_patch());
                }
            }
        }

        __syncthreads();

        if (replace) {

            uint8_t o = pi.patch_stash.insert_patch(handle.patch_id());

            LPPair lp(i, handle.local_id(), o);

            pi.get_lp<HandleT>().replace(lp);
        }
    }
}

template <uint32_t blockThreads>
__global__ static void hashtable_calibration(const Context context)
{
    const uint32_t pid = blockIdx.x;
    if (pid >= context.m_num_patches[0]) {
        return;
    }

    PatchInfo pi = context.m_patches_info[pid];

    hashtable_calibration<blockThreads, VertexHandle>(context, pi);
    hashtable_calibration<blockThreads, EdgeHandle>(context, pi);
    hashtable_calibration<blockThreads, FaceHandle>(context, pi);
}


template <uint32_t blockThreads>
__global__ static void remove_surplus_elements(const Context context)
{
    auto block = cooperative_groups::this_thread_block();

    const uint32_t pid = blockIdx.x;
    if (pid >= context.m_num_patches[0]) {
        return;
    }

    PatchInfo pi = context.m_patches_info[pid];

    const uint16_t num_vertices = pi.num_vertices[0];
    const uint16_t num_edges    = pi.num_edges[0];
    const uint16_t num_faces    = pi.num_faces[0];

    ShmemAllocator shrd_alloc;

    uint16_t* s_fe = shrd_alloc.alloc<uint16_t>(3 * num_faces);
    load_async(
        block, reinterpret_cast<uint16_t*>(pi.fe), 3 * num_faces, s_fe, false);

    uint16_t* s_ev = shrd_alloc.alloc<uint16_t>(2 * num_edges);
    load_async(
        block, reinterpret_cast<uint16_t*>(pi.ev), 2 * num_edges, s_ev, false);

    Bitmask s_owned_v = Bitmask(num_vertices, shrd_alloc);
    s_owned_v.load_async(block, pi.owned_mask_v);

    Bitmask s_owned_e = Bitmask(num_edges, shrd_alloc);
    s_owned_e.load_async(block, pi.owned_mask_e);

    Bitmask s_owned_f = Bitmask(num_faces, shrd_alloc);
    s_owned_f.load_async(block, pi.owned_mask_f);

    Bitmask s_active_v = Bitmask(num_vertices, shrd_alloc);
    s_active_v.load_async(block, pi.active_mask_v);

    Bitmask s_active_e = Bitmask(num_edges, shrd_alloc);
    s_active_e.load_async(block, pi.active_mask_e);

    Bitmask s_active_f = Bitmask(num_faces, shrd_alloc);
    s_active_f.load_async(block, pi.active_mask_f, true);


    // indicate if an edge is incident to an owned face
    Bitmask s_edge_tag = Bitmask(num_edges, shrd_alloc);
    s_edge_tag.reset(block);

    Bitmask s_vert_tag = Bitmask(num_vertices, shrd_alloc);
    s_vert_tag.reset(block);

    Bitmask s_face_tag = Bitmask(num_faces, shrd_alloc);
    s_face_tag.reset(block);

    block.sync();

    auto tag_edges_and_vertices_through_face = [&]() {
        // mark edges that are incident to owned faces
        for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
            if (s_active_f(f) && (s_owned_f(f) || s_face_tag(f))) {
                for (int i = 0; i < 3; ++i) {
                    const uint16_t e = s_fe[3 * f + i] >> 1;
                    assert(e < num_edges);
                    assert(s_active_e(e));

                    const uint16_t v0 = s_ev[2 * e + 0];
                    assert(v0 < num_vertices);
                    assert(s_active_v(v0));

                    const uint16_t v1 = s_ev[2 * e + 1];
                    assert(v1 < num_vertices);
                    assert(s_active_v(v1));

                    s_edge_tag.set(e, true);
                    s_vert_tag.set(v0, true);
                    s_vert_tag.set(v1, true);
                }
            }
        }
    };


    tag_edges_and_vertices_through_face();
    block.sync();

    // tag faces through edges
    for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
        if (s_active_f(f)) {

            for (int i = 0; i < 3; ++i) {
                const uint16_t e = s_fe[3 * f + i] >> 1;
                assert(e < num_edges);
                assert(s_active_e(e));

                const uint16_t v0 = s_ev[2 * e + 0];
                assert(v0 < num_vertices);
                assert(s_active_v(v0));

                const uint16_t v1 = s_ev[2 * e + 1];
                assert(v1 < num_vertices);
                assert(s_active_v(v1));

                if (s_edge_tag(e) || s_vert_tag(v0) || s_vert_tag(v1) ||
                    s_owned_e(e) || s_owned_v(v0) || s_owned_v(v1)) {
                    s_face_tag.set(f, true);
                    break;
                }
            }
        }
    }
    block.sync();

    tag_edges_and_vertices_through_face();
    block.sync();


    // tag vertices through edges
    for (uint16_t e = threadIdx.x; e < num_edges; e += blockThreads) {
        if (s_active_e(e) && (s_owned_e(e) || s_edge_tag(e))) {

            const uint16_t v0 = s_ev[2 * e + 0];
            assert(v0 < num_vertices);
            assert(s_active_v(v0));

            const uint16_t v1 = s_ev[2 * e + 1];
            assert(v1 < num_vertices);
            assert(s_active_v(v1));

            s_vert_tag.set(v0, true);
            s_vert_tag.set(v1, true);
            s_edge_tag.set(e, true);
        }
    }

    block.sync();
    s_vert_tag.store<blockThreads>(pi.active_mask_v);
    s_edge_tag.store<blockThreads>(pi.active_mask_e);
    s_face_tag.store<blockThreads>(pi.active_mask_f);
}

template <uint32_t blockThreads>
__inline__ __device__ void bi_assignment2(
    cooperative_groups::thread_block& block,
    ShmemAllocator&                   shrd_alloc,
    const uint16_t                    num_vertices,
    const uint16_t                    num_edges,
    const uint16_t                    num_faces,
    const Bitmask&                    s_owned_v,
    const Bitmask&                    s_owned_e,
    const Bitmask&                    s_owned_f,
    const Bitmask&                    s_active_v,
    const Bitmask&                    s_active_e,
    const Bitmask&                    s_active_f,
    const uint16_t*                   s_ev,
    const uint16_t*                   s_fe,
    Bitmask&                          s_patch_v,
    Bitmask&                          s_patch_e,
    Bitmask&                          s_patch_f,
    const int                         max_num_iter = 10)
{
    // initially, all vertices/edges/face belongs to the same (this) patch
    // so, we only assigne vertices/edges/faces to the other (split) patch
    // this assignment is indicated by setting a bit in s_patch_v/e/f

    __shared__ int s_seed, s_num_1_faces, s_num_boundary_faces;


    Bitmask s_boundary_f(num_faces, shrd_alloc);
    Bitmask s_boundary_e(num_edges, shrd_alloc);

    s_patch_v.reset(block);
    s_patch_e.reset(block);
    s_patch_f.reset(block);
    block.sync();

    if (threadIdx.x == 0) {
        // pick two seeds that are active and owned faces
        for (uint16_t f = 0; f < num_faces; ++f) {
            if (s_active_f(f) && s_owned_f(f)) {
                s_seed = f;
                break;
            }
        }
    }
    block.sync();

    int num_iter = 0;

    while (num_iter < max_num_iter) {
        // seeding
        if (threadIdx.x == 0) {
            s_patch_f.set(uint16_t(s_seed), true);
            s_num_1_faces = 1;
        }
        block.sync();

        // cluster seed propagation
        while (s_num_1_faces < num_faces / 2) {
            // 1st
            for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
                if (s_active_f(f) && s_patch_f(f)) {
                    const uint16_t e0(s_fe[3 * f + 0] >> 1),
                        e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                    s_patch_e.set(e0, true);
                    s_patch_e.set(e1, true);
                    s_patch_e.set(e2, true);
                }
            }

            block.sync();

            // 2nd
            for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
                if (s_active_f(f)) {
                    const uint16_t e0(s_fe[3 * f + 0] >> 1),
                        e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                    int sum = int(s_patch_e(e0)) + int(s_patch_e(e1)) +
                              int(s_patch_e(e2));
                    if (sum >= 1) {
                        s_patch_f.set(f, true);
                        ::atomicAdd(&s_num_1_faces, int(1));
                    }
                }
            }
            block.sync();
        }


        // interior
        // find the most interior face and set it as seed

        s_boundary_f.reset(block);
        s_boundary_e.reset(block);
        if (threadIdx.x == 0) {
            s_num_boundary_faces = 0;
        }
        block.sync();


        // set boundary faces
        for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
            if (s_patch_f(f)) {
                const uint16_t e0(s_fe[3 * f + 0] >> 1),
                    e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                int sum = int(s_patch_e(e0)) + int(s_patch_e(e1)) +
                          int(s_patch_e(e2));
                if (sum == 1 || sum == 2) {
                    s_boundary_f.set(f, true);
                    ::atomicAdd(&s_num_boundary_faces, int(1));
                    atomicExch(&s_seed, int(f));
                }
            }
        }
        block.sync();

        while (s_num_boundary_faces != s_num_1_faces) {
            assert(s_num_boundary_faces < s_num_1_faces);
            block.sync();
            // set boundary edges
            for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
                if (s_boundary_f(f)) {
                    const uint16_t e0(s_fe[3 * f + 0] >> 1),
                        e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                    s_boundary_e.set(e0, true);
                    s_boundary_e.set(e1, true);
                    s_boundary_e.set(e2, true);
                }
            }
            block.sync();

            // set boundary faces
            for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
                if (s_patch_f(f)) {
                    const uint16_t e0(s_fe[3 * f + 0] >> 1),
                        e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                    if (s_boundary_e(e0) || s_boundary_e(e1) ||
                        s_boundary_e(e2)) {
                        s_boundary_f.set(f, true);
                        ::atomicAdd(&s_num_boundary_faces, int(1));
                        atomicExch(&s_seed, int(f));
                    }
                }
            }
            block.sync();
        }

        block.sync();
        s_patch_e.reset(block);
        s_patch_f.reset(block);
        block.sync();

        num_iter++;
    }

    // finally we assign vertices such that each edge assign its two vertices
    for (uint16_t e = threadIdx.x; e < num_edges; e += blockThreads) {
        if (s_active_e(e) && s_patch_e(e)) {
            const uint16_t v0(s_ev[2 * e + 0]), v1(s_ev[2 * e + 1]);
            s_patch_v.set(v0, true);
            s_patch_v.set(v1, true);
        }
    }
}

template <uint32_t blockThreads>
__inline__ __device__ void bi_assignment(
    cooperative_groups::thread_block& block,
    const uint16_t                    num_vertices,
    const uint16_t                    num_edges,
    const uint16_t                    num_faces,
    const Bitmask&                    s_owned_v,
    const Bitmask&                    s_owned_e,
    const Bitmask&                    s_owned_f,
    const Bitmask&                    s_active_v,
    const Bitmask&                    s_active_e,
    const Bitmask&                    s_active_f,
    const uint16_t*                   s_ev,
    const uint16_t*                   s_fe,
    Bitmask&                          s_patch_v,
    Bitmask&                          s_patch_e,
    Bitmask&                          s_patch_f)
{
    // assign mesh element to two partitions. the assignment partition the patch
    // into contiguous patches of almost equal size
    //
    // an element stay in the patch if its bitmask is zero

    // initially, all mesh elements stays in this patch
    s_patch_v.reset(block);
    s_patch_e.reset(block);
    s_patch_f.reset(block);
    block.sync();

    // number of faces that are assigned to 1
    __shared__ uint16_t s_num_1_faces;
    if (threadIdx.x == 0) {
        // we bootstrap the assignment by assigning a ribbon face to 1
        for (uint16_t f = 0; f < num_faces; ++f) {
            if (s_active_f(f) && !s_owned_f(f)) {
                s_patch_f.set(f);
            }
        }
        s_num_1_faces = 1;
    }
    block.sync();


    // we iterate over faces twice. First, every face atomically set its
    // three edges if the face is set. Second, every face set itself if there
    // are two edges incident to it that are set. we stop when the s_num_1_faces
    // is more than half num_faces
    while (s_num_1_faces < num_faces / 2) {

        // 1st
        for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
            if (s_active_f(f) && s_patch_f(f)) {
                const uint16_t e0(s_fe[3 * f + 0] >> 1),
                    e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);
                s_patch_e.set(e0, true);
                s_patch_e.set(e1, true);
                s_patch_e.set(e2, true);
            }
        }

        block.sync();

        // 2nd
        for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
            if (s_active_f(f)) {
                const uint16_t e0(s_fe[3 * f + 0] >> 1),
                    e1(s_fe[3 * f + 1] >> 1), e2(s_fe[3 * f + 2] >> 1);

                if (s_patch_e(e0) || s_patch_e(e1) || s_patch_e(e2)) {
                    s_patch_f.set(f, true);
                    atomicAdd(&s_num_1_faces, 1);
                }
            }
        }
        block.sync();
    }


    // finally we assign vertices such that each edge assign its two vertices
    for (uint16_t e = threadIdx.x; e < num_edges; e += blockThreads) {
        if (s_active_e(e) && s_patch_e(e)) {
            const uint16_t v0(s_ev[2 * e + 0]), v1(s_ev[2 * e + 1]);
            s_patch_v.set(v0, true);
            s_patch_v.set(v1, true);
        }
    }
}

template <uint32_t blockThreads>
__global__ static void slice_patches(const Context  context,
                                     const uint32_t current_num_patches,
                                     const uint32_t num_faces_threshold,
                                     rxmesh::FaceAttribute<int>   f_attr,
                                     rxmesh::EdgeAttribute<int>   e_attr,
                                     rxmesh::VertexAttribute<int> v_attr)
{
    // ev, fe, active_v/e/f, owned_v/e/f, patch_v/e/f
    auto block = cooperative_groups::this_thread_block();

    ShmemAllocator shrd_alloc;

    const uint32_t pid = blockIdx.x;
    if (pid >= current_num_patches) {
        return;
    }

    PatchInfo pi = context.m_patches_info[pid];

    const uint16_t num_vertices = pi.num_vertices[0];
    const uint16_t num_edges    = pi.num_edges[0];
    const uint16_t num_faces    = pi.num_faces[0];


    auto alloc_masks = [&](uint16_t        num_elements,
                           Bitmask&        owned,
                           Bitmask&        active,
                           Bitmask&        patch,
                           const uint32_t* g_owned,
                           const uint32_t* g_active) {
        owned  = Bitmask(num_elements, shrd_alloc);
        active = Bitmask(num_elements, shrd_alloc);
        patch  = Bitmask(num_elements, shrd_alloc);

        owned.reset(block);
        active.reset(block);

        // to remove the racecheck hazard report due to WAW on owned and active
        block.sync();

        detail::load_async(block,
                           reinterpret_cast<const char*>(g_owned),
                           owned.num_bytes(),
                           reinterpret_cast<char*>(owned.m_bitmask),
                           false);
        detail::load_async(block,
                           reinterpret_cast<const char*>(g_active),
                           active.num_bytes(),
                           reinterpret_cast<char*>(active.m_bitmask),
                           false);
    };


    if (num_faces >= num_faces_threshold) {
        Bitmask s_owned_v, s_owned_e, s_owned_f;
        Bitmask s_active_v, s_active_e, s_active_f;
        Bitmask s_patch_v, s_patch_e, s_patch_f;

        uint16_t* s_ev = shrd_alloc.alloc<uint16_t>(2 * num_edges);
        detail::load_async(block,
                           reinterpret_cast<uint16_t*>(pi.ev),
                           2 * num_edges,
                           s_ev,
                           false);
        uint16_t* s_fe = shrd_alloc.alloc<uint16_t>(3 * num_faces);
        detail::load_async(block,
                           reinterpret_cast<uint16_t*>(pi.fe),
                           3 * num_faces,
                           s_fe,
                           true);

        alloc_masks(num_vertices,
                    s_owned_v,
                    s_active_v,
                    s_patch_v,
                    pi.owned_mask_v,
                    pi.active_mask_v);
        alloc_masks(num_edges,
                    s_owned_e,
                    s_active_e,
                    s_patch_e,
                    pi.owned_mask_e,
                    pi.active_mask_e);
        alloc_masks(num_faces,
                    s_owned_f,
                    s_active_f,
                    s_patch_f,
                    pi.owned_mask_f,
                    pi.active_mask_f);


        bi_assignment<blockThreads>(block,
                                    // shrd_alloc,
                                    num_vertices,
                                    num_edges,
                                    num_faces,
                                    s_owned_v,
                                    s_owned_e,
                                    s_owned_f,
                                    s_active_v,
                                    s_active_e,
                                    s_active_f,
                                    s_ev,
                                    s_fe,
                                    s_patch_v,
                                    s_patch_e,
                                    s_patch_f);
        block.sync();

        if (pi.patch_id == 1) {
            for (uint16_t f = threadIdx.x; f < num_faces; f += blockThreads) {
                FaceHandle fh(pi.patch_id, f);
                if (!s_owned_f(f)) {
                    fh = pi.find<FaceHandle>(f);
                }

                if (s_patch_f(fh.local_id())) {
                    f_attr(fh) = 1;
                } else {
                    f_attr(fh) = 2;
                }
            }
            detail::for_each_face(pi, [&](const FaceHandle fh) {
                if (s_patch_f(fh.local_id())) {
                    f_attr(fh) = 1;
                } else {
                    f_attr(fh) = 2;
                }
            });

            detail::for_each_edge(pi, [&](const EdgeHandle eh) {
                if (s_patch_e(eh.local_id())) {
                    e_attr(eh) = 1;
                } else {
                    e_attr(eh) = 2;
                }
            });

            detail::for_each_vertex(pi, [&](const VertexHandle vh) {
                if (s_patch_v(vh.local_id())) {
                    v_attr(vh) = 1;
                } else {
                    v_attr(vh) = 2;
                }
            });
        }
    }
}


template <uint32_t blockThreads>
__global__ static void calc_num_elements(const Context context,
                                         uint32_t*     sum_num_vertices,
                                         uint32_t*     sum_num_edges,
                                         uint32_t*     sum_num_faces)
{
    auto sum_v = [&](VertexHandle& v_id) { ::atomicAdd(sum_num_vertices, 1u); };
    for_each<Op::V, blockThreads>(context, sum_v);


    auto sum_e = [&](EdgeHandle& e_id) { ::atomicAdd(sum_num_edges, 1u); };
    for_each<Op::E, blockThreads>(context, sum_e);


    auto sum_f = [&](FaceHandle& f_id) { ::atomicAdd(sum_num_faces, 1u); };
    for_each<Op::F, blockThreads>(context, sum_f);
}

template <uint32_t blockThreads>
__global__ static void check_uniqueness(const Context           context,
                                        unsigned long long int* d_check)
{
    auto block = cooperative_groups::this_thread_block();

    const uint32_t patch_id = blockIdx.x;

    if (patch_id < context.m_num_patches[0]) {

        PatchInfo patch_info = context.m_patches_info[patch_id];

        ShmemAllocator shrd_alloc;

        uint16_t* s_fe =
            shrd_alloc.alloc<uint16_t>(3 * patch_info.num_faces[0]);
        uint16_t* s_ev =
            shrd_alloc.alloc<uint16_t>(2 * patch_info.num_edges[0]);

        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.ev),
                   2 * patch_info.num_edges[0],
                   s_ev,
                   false);

        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.fe),
                   3 * patch_info.num_faces[0],
                   s_fe,
                   true);
        block.sync();

        // make sure an edge is connecting two unique vertices
        for (uint16_t e = threadIdx.x; e < patch_info.num_edges[0];
             e += blockThreads) {

            const LocalEdgeT el(e);

            uint16_t v0 = s_ev[2 * e + 0];
            uint16_t v1 = s_ev[2 * e + 1];

            if (!patch_info.is_deleted(el)) {

                if (v0 >= patch_info.num_vertices[0] ||
                    v1 >= patch_info.num_vertices[0] || v0 == v1) {
                    // printf("\n 1 unqiuness = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }

                if (patch_info.is_deleted(LocalVertexT(v0)) ||
                    patch_info.is_deleted(LocalVertexT(v1))) {

                    // printf("\n 2 unqiuness = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }
            }
        }

        // make sure a face is formed by three unique edges and these edges
        // gives three unique vertices
        for (uint16_t f = threadIdx.x; f < patch_info.num_faces[0];
             f += blockThreads) {

            const LocalFaceT fl(f);

            if (!patch_info.is_deleted(fl)) {
                uint16_t e0, e1, e2;
                flag_t   d0(0), d1(0), d2(0);
                Context::unpack_edge_dir(s_fe[3 * f + 0], e0, d0);
                Context::unpack_edge_dir(s_fe[3 * f + 1], e1, d1);
                Context::unpack_edge_dir(s_fe[3 * f + 2], e2, d2);

                if (e0 >= patch_info.num_edges[0] ||
                    e1 >= patch_info.num_edges[0] ||
                    e2 >= patch_info.num_edges[0] || e0 == e1 || e0 == e2 ||
                    e1 == e2) {
                    // printf("\n 3 unqiuness = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }

                if (patch_info.is_deleted(LocalEdgeT(e0)) ||
                    patch_info.is_deleted(LocalEdgeT(e1)) ||
                    patch_info.is_deleted(LocalEdgeT(e2))) {
                    // printf("\n 4 unqiuness = %u, f= %u", patch_id, f);
                    ::atomicAdd(d_check, 1);
                }

                uint16_t v0, v1, v2;
                v0 = s_ev[(2 * e0) + (1 * d0)];
                v1 = s_ev[(2 * e1) + (1 * d1)];
                v2 = s_ev[(2 * e2) + (1 * d2)];


                if (v0 >= patch_info.num_vertices[0] ||
                    v1 >= patch_info.num_vertices[0] ||
                    v2 >= patch_info.num_vertices[0] || v0 == v1 || v0 == v2 ||
                    v1 == v2) {
                    // printf("\n 5 unqiuness = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }

                if (patch_info.is_deleted(LocalVertexT(v0)) ||
                    patch_info.is_deleted(LocalVertexT(v1)) ||
                    patch_info.is_deleted(LocalVertexT(v2))) {
                    // printf("\n 6 unqiuness = %u, f=%u", patch_id, f);
                    ::atomicAdd(d_check, 1);
                }
            }
        }
    }
}


template <uint32_t blockThreads>
__global__ static void check_not_owned(const Context           context,
                                       unsigned long long int* d_check)
{
    auto block = cooperative_groups::this_thread_block();

    const uint32_t patch_id = blockIdx.x;

    if (patch_id < context.m_num_patches[0]) {

        const PatchInfo patch_info = context.m_patches_info[patch_id];

        ShmemAllocator shrd_alloc;
        uint16_t*      s_fe =
            shrd_alloc.alloc<uint16_t>(3 * patch_info.num_faces[0]);
        uint16_t* s_ev =
            shrd_alloc.alloc<uint16_t>(2 * patch_info.num_edges[0]);
        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.ev),
                   2 * patch_info.num_edges[0],
                   s_ev,
                   false);

        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.fe),
                   3 * patch_info.num_faces[0],
                   s_fe,
                   true);
        block.sync();


        // for every not-owned face, check that its three edges (possibly
        // not-owned) are the same as those in the face's owner patch
        for (uint16_t f = threadIdx.x; f < patch_info.num_faces[0];
             f += blockThreads) {
            const LocalFaceT fl(f);
            if (!patch_info.is_deleted(fl) && !patch_info.is_owned(fl)) {

                uint16_t e0, e1, e2;
                flag_t   d0(0), d1(0), d2(0);
                uint32_t p0(patch_id), p1(patch_id), p2(patch_id);
                Context::unpack_edge_dir(s_fe[3 * f + 0], e0, d0);
                Context::unpack_edge_dir(s_fe[3 * f + 1], e1, d1);
                Context::unpack_edge_dir(s_fe[3 * f + 2], e2, d2);

                // if the edge is not owned, grab its local index in the owner
                // patch
                auto get_owned_e = [&](uint16_t&       e,
                                       uint32_t&       p,
                                       const PatchInfo pi) {
                    EdgeHandle eh =
                        context.get_owner_handle(EdgeHandle(pi.patch_id, {e}));

                    e = eh.local_id();
                    p = eh.patch_id();
                };

                get_owned_e(e0, p0, patch_info);
                get_owned_e(e1, p1, patch_info);
                get_owned_e(e2, p2, patch_info);

                // get f's three edges from its owner patch

                // face handle of this face (f) in its owner patch
                FaceHandle f_owned = context.get_owner_handle(
                    FaceHandle(patch_info.patch_id, fl));
                PatchInfo owner_patch_info =
                    context.m_patches_info[f_owned.patch_id()];

                // the owner patch should have indicate that the owned face is
                // owned by it
                if (!owner_patch_info.is_owned(
                        LocalFaceT(f_owned.local_id()))) {
                    // printf("\n 1 owned = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }

                // If a face is deleted, it should also be deleted in the other
                // patches that have it as not-owned
                if (owner_patch_info.is_deleted(
                        LocalFaceT(f_owned.local_id()))) {
                    // printf("\n 2 owned = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                } else {
                    uint16_t ew0, ew1, ew2;
                    flag_t   dw0(0), dw1(0), dw2(0);
                    uint32_t pw0(f_owned.patch_id()), pw1(f_owned.patch_id()),
                        pw2(f_owned.patch_id());
                    Context::unpack_edge_dir(
                        owner_patch_info.fe[3 * f_owned.local_id() + 0].id,
                        ew0,
                        dw0);
                    Context::unpack_edge_dir(
                        owner_patch_info.fe[3 * f_owned.local_id() + 1].id,
                        ew1,
                        dw1);
                    Context::unpack_edge_dir(
                        owner_patch_info.fe[3 * f_owned.local_id() + 2].id,
                        ew2,
                        dw2);

                    get_owned_e(ew0, pw0, owner_patch_info);
                    get_owned_e(ew1, pw1, owner_patch_info);
                    get_owned_e(ew2, pw2, owner_patch_info);

                    if (e0 != ew0 || p0 != pw0 ||  //
                        e1 != ew1 || p1 != pw1 ||  //
                        e2 != ew2 || p2 != pw2) {
                        /*if (e0 != ew0 || p0 != pw0) {
                            printf(
                                "\n 3A owned patch= %u, f=%u, fw(%u, %u), "
                                "(p0=%u, e0=%u, pw0=%u, ew0=%u)",
                                patch_info.patch_id,
                                f,
                                f_owned.patch_id(),
                                f_owned.local_id(),
                                p0,
                                e0,
                                pw0,
                                ew0);
                        }

                        if (e1 != ew1 || p1 != pw1) {
                            printf(
                                "\n 3B owned patch= %u, f=%u, fw(%u, %u), "
                                "(p1=%u, e1=%u, pw1=%u, ew1=%u)",
                                patch_info.patch_id,
                                f,
                                f_owned.patch_id(),
                                f_owned.local_id(),
                                p1,
                                e1,
                                pw1,
                                ew1);
                        }

                        if (e2 != ew2 || p2 != pw2) {
                            printf(
                                "\n 3C owned patch= %u, f=%u, fw(%u, %u), "
                                "(p2=%u, e2=%u, pw2=%u, ew2=%u)",
                                patch_info.patch_id,
                                f,
                                f_owned.patch_id(),
                                f_owned.local_id(),
                                p2,
                                e2,
                                pw2,
                                ew2);
                        }*/
                        ::atomicAdd(d_check, 1);
                    }

                    if (d0 != dw0 || d1 != dw1 || d2 != dw2) {
                        // printf("\n 4 owned = %u", patch_id);
                        ::atomicAdd(d_check, 1);
                    }
                }
            }
        }

        // for every not-owned edge, check its two vertices (possibly
        // not-owned) are the same as those in the edge's owner patch
        for (uint16_t e = threadIdx.x; e < patch_info.num_edges[0];
             e += blockThreads) {

            const LocalEdgeT el(e);
            if (!patch_info.is_deleted(el) && !patch_info.is_owned(el)) {

                uint16_t v0 = s_ev[2 * e + 0];
                uint16_t v1 = s_ev[2 * e + 1];
                uint32_t p0(patch_id), p1(patch_id);

                auto get_owned_v =
                    [&](uint16_t& v, uint32_t& p, const PatchInfo pi) {
                        VertexHandle vh = context.get_owner_handle(
                            VertexHandle(pi.patch_id, {v}));

                        v = vh.local_id();
                        p = vh.patch_id();
                    };

                get_owned_v(v0, p0, patch_info);
                get_owned_v(v1, p1, patch_info);

                // get e's two vertices from its owner patch
                EdgeHandle e_owned = context.get_owner_handle(
                    EdgeHandle(patch_info.patch_id, el));

                PatchInfo owner_patch_info =
                    context.m_patches_info[e_owned.patch_id()];


                // the owner patch should have indicate that the owned face is
                // owned by it
                if (!owner_patch_info.is_owned(
                        LocalEdgeT(e_owned.local_id()))) {
                    // printf("\n 5 owned = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                }

                // If an edge is deleted, it should also be deleted in the other
                // patches that have it as not-owned
                if (owner_patch_info.is_deleted(
                        LocalEdgeT(e_owned.local_id()))) {
                    // printf("\n 6 owned = %u", patch_id);
                    ::atomicAdd(d_check, 1);
                } else {
                    uint16_t vw0 =
                        owner_patch_info.ev[2 * e_owned.local_id() + 0].id;
                    uint16_t vw1 =
                        owner_patch_info.ev[2 * e_owned.local_id() + 1].id;
                    uint32_t pw0(e_owned.patch_id()), pw1(e_owned.patch_id());

                    get_owned_v(vw0, pw0, owner_patch_info);
                    get_owned_v(vw1, pw1, owner_patch_info);

                    if (v0 != vw0 || p0 != pw0 || v1 != vw1 || p1 != pw1) {
                        // printf("\n 7 owned = %u", patch_id);
                        ::atomicAdd(d_check, 1);
                    }
                }
            }
        }
    }
}


template <uint32_t blockThreads>
__global__ static void check_ribbon_edges(const Context           context,
                                          unsigned long long int* d_check)
{
    auto block = cooperative_groups::this_thread_block();

    const uint32_t patch_id = blockIdx.x;

    if (patch_id < context.m_num_patches[0]) {
        PatchInfo patch_info = context.m_patches_info[patch_id];

        ShmemAllocator shrd_alloc;
        uint16_t*      s_fe =
            shrd_alloc.alloc<uint16_t>(3 * patch_info.num_faces[0]);
        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.fe),
                   3 * patch_info.num_faces[0],
                   s_fe,
                   true);
        uint16_t* s_mark_edges =
            shrd_alloc.alloc<uint16_t>(patch_info.num_edges[0]);

        for (uint16_t e = threadIdx.x; e < patch_info.num_edges[0];
             e += blockThreads) {
            s_mark_edges[e] = 0;
        }

        block.sync();

        // Check that each owned edge is incident to at least one owned
        // not-deleted face. We do that by iterating over faces, each face
        // (atomically) mark its incident edges only if they are owned. Then we
        // check the marked edges where we expect all owned edges to be marked.
        // If there is an edge that is owned but not marked, then this edge is
        // not incident to any owned faces
        for (uint16_t f = threadIdx.x; f < patch_info.num_faces[0];
             f += blockThreads) {
            const LocalFaceT fl(f);

            if (!patch_info.is_deleted(fl) && patch_info.is_owned(fl)) {

                uint16_t e0 = s_fe[3 * f + 0] >> 1;
                uint16_t e1 = s_fe[3 * f + 1] >> 1;
                uint16_t e2 = s_fe[3 * f + 2] >> 1;

                auto mark_if_owned = [&](uint16_t edge) {
                    if (patch_info.is_owned(LocalEdgeT(edge))) {
                        atomicAdd(s_mark_edges + edge, uint16_t(1));
                    }
                };

                mark_if_owned(e0);
                mark_if_owned(e1);
                mark_if_owned(e2);
            }
        }
        block.sync();
        for (uint16_t e = threadIdx.x; e < patch_info.num_edges[0];
             e += blockThreads) {
            const LocalEdgeT el(e);
            if (patch_info.is_owned(el) && !patch_info.is_deleted(el)) {
                if (s_mark_edges[e] == 0) {
                    // printf("\n ribbon edge = %u, %u", patch_id, e);
                    ::atomicAdd(d_check, 1);
                }
            }
        }
    }
}


template <uint32_t blockThreads>
__global__ static void compute_vf(const Context               context,
                                  VertexAttribute<FaceHandle> output)
{
    using namespace rxmesh;

    auto store_lambda = [&](VertexHandle& v_id, FaceIterator& iter) {
        for (uint32_t i = 0; i < iter.size(); ++i) {
            output(v_id, i) = iter[i];
        }
    };

    auto block = cooperative_groups::this_thread_block();

    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VF>(block, shrd_alloc, store_lambda);
}


template <uint32_t blockThreads>
__global__ static void compute_max_valence(const Context context,
                                           uint32_t*     d_max_valence)
{
    using namespace rxmesh;

    auto max_valence = [&](VertexHandle& v_id, VertexIterator& iter) {
        ::atomicMax(d_max_valence, iter.size());
    };

    auto block = cooperative_groups::this_thread_block();

    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, max_valence);
}

template <uint32_t blockThreads>
__global__ static void check_ribbon_faces(const Context               context,
                                          VertexAttribute<FaceHandle> global_vf,
                                          unsigned long long int*     d_check)
{
    auto block = cooperative_groups::this_thread_block();

    const uint32_t patch_id = blockIdx.x;

    if (patch_id < context.m_num_patches[0]) {
        PatchInfo patch_info = context.m_patches_info[patch_id];

        ShmemAllocator shrd_alloc;
        uint16_t*      s_fv =
            shrd_alloc.alloc<uint16_t>(3 * patch_info.num_faces[0]);
        uint16_t* s_fe =
            shrd_alloc.alloc<uint16_t>(3 * patch_info.num_faces[0]);
        uint16_t* s_ev =
            shrd_alloc.alloc<uint16_t>(2 * patch_info.num_edges[0]);
        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.ev),
                   2 * patch_info.num_edges[0],
                   s_ev,
                   false);
        load_async(block,
                   reinterpret_cast<uint16_t*>(patch_info.fe),
                   3 * patch_info.num_faces[0],
                   s_fv,
                   true);
        block.sync();


        // compute FV
        f_v<blockThreads>(patch_info.num_edges[0],
                          s_ev,
                          patch_info.num_faces[0],
                          s_fv,
                          patch_info.active_mask_f);
        block.sync();

        // copy FV
        for (uint16_t i = threadIdx.x; i < 3 * patch_info.num_faces[0];
             i += blockThreads) {
            s_fe[i] = s_fv[i];
        }
        block.sync();

        // compute (local) VF by transposing FV
        uint16_t* s_vf_offset = &s_fe[0];
        uint16_t* s_vf_value  = &s_ev[0];
        block_mat_transpose<3u, blockThreads>(patch_info.num_faces[0],
                                              patch_info.num_vertices[0],
                                              s_fe,
                                              s_ev,
                                              patch_info.active_mask_f,
                                              0);
        block.sync();

        // For every incident vertex V to an owned face, check if VF of V
        // using global_VF can be retrieved from local_VF
        for (uint16_t f = threadIdx.x; f < patch_info.num_faces[0];
             f += blockThreads) {

            const LocalFaceT fl(f);

            // Only if the face is owned, we do the check
            if (!patch_info.is_deleted(fl) && patch_info.is_owned(fl)) {

                // for the three vertices incident to this face
                for (uint16_t k = 0; k < 3; ++k) {
                    uint16_t v_id = s_fv[3 * f + k];

                    // get the vertex handle so we can index the attributes
                    assert(!patch_info.is_deleted(LocalVertexT(v_id)));

                    const VertexHandle vh = context.get_owner_handle(
                        VertexHandle(patch_id, {v_id}));

                    // for every incident face to this vertex
                    for (uint16_t i = 0; i < global_vf.get_num_attributes();
                         ++i) {

                        const auto fvh_global = global_vf(vh, i);

                        if (fvh_global.is_valid()) {

                            // look for the face incident to the vertex in local
                            // VF
                            bool found = false;
                            for (uint16_t j = s_vf_offset[v_id];
                                 j < s_vf_offset[v_id + 1];
                                 ++j) {

                                assert(!patch_info.is_deleted(
                                    LocalFaceT(s_vf_value[j])));

                                const FaceHandle fh = context.get_owner_handle(
                                    FaceHandle(patch_id, {s_vf_value[j]}));

                                if (fvh_global == fh) {
                                    found = true;
                                    break;
                                }
                            }

                            if (!found) {
                                /*printf(
                                    "\n T=%u, ribbon face = %u, f= %u, v_id= "
                                    "%u ",
                                    threadIdx.x,
                                    patch_id,
                                    f,
                                    v_id);*/
                                ::atomicAdd(d_check, 1);
                                break;
                            }
                        }
                    }
                }
            }
        }
    }
}

}  // namespace detail


void RXMeshDynamic::save(std::string filename)
{
    if (m_patcher->m_num_patches != get_num_patches()) {
        RXMESH_ERROR(
            "RXMeshDynamic:save() does not support changing number of "
            "patches in the mesh");
    }

    m_patcher->m_max_num_patches = get_num_patches();

    m_patcher->m_num_vertices = get_num_vertices();
    m_patcher->m_vertex_patch.resize(m_patcher->m_num_vertices);

    m_patcher->m_num_edges = get_num_edges();
    m_patcher->m_edge_patch.resize(m_patcher->m_num_edges);

    m_patcher->m_num_faces = get_num_faces();
    m_patcher->m_face_patch.resize(m_patcher->m_num_faces);


    for_each_vertex(HOST, [&](VertexHandle vh) {
        m_patcher->m_vertex_patch[linear_id(vh)] = vh.patch_id();
    });

    for_each_edge(HOST, [&](EdgeHandle eh) {
        m_patcher->m_edge_patch[linear_id(eh)] = eh.patch_id();
    });

    for_each_face(HOST, [&](FaceHandle fh) {
        m_patcher->m_face_patch[linear_id(fh)] = fh.patch_id();
    });


    m_patcher->m_patches_offset.resize(get_num_patches(), 0);
    std::fill(m_patcher->m_patches_offset.begin(),
              m_patcher->m_patches_offset.end(),
              0);
    m_patcher->m_patches_val.resize(get_num_faces());

    for_each_face(
        HOST,
        [&](FaceHandle fh) { m_patcher->m_patches_offset[fh.patch_id()]++; },
        NULL,
        false);

    std::inclusive_scan(m_patcher->m_patches_offset.begin(),
                        m_patcher->m_patches_offset.end(),
                        m_patcher->m_patches_offset.begin());

    std::vector<int> offset(get_num_patches(), 0);

    for_each_face(
        HOST,
        [&](FaceHandle fh) {
            uint32_t p_offset =
                (fh.patch_id() == 0) ?
                    0 :
                    m_patcher->m_patches_offset[fh.patch_id() - 1];
            m_patcher->m_patches_val[p_offset + (offset[fh.patch_id()]++)] =
                linear_id(fh);
        },
        NULL,
        false);

    // update m_ribbon_ext_val and m_ribbon_ext_offset
    m_patcher->m_ribbon_ext_offset.resize(get_num_patches(), 0);
    std::fill(m_patcher->m_ribbon_ext_offset.begin(),
              m_patcher->m_ribbon_ext_offset.end(),
              0);
    m_patcher->m_ribbon_ext_val.resize(get_num_faces(), 0);


    for (uint32_t p = 0; p < get_num_patches(); ++p) {
        uint16_t num_not_owned_faces =
            m_h_patches_info[p].num_faces[0] -
            m_h_patches_info[p].get_num_owned<FaceHandle>();
        m_patcher->m_ribbon_ext_offset[p] = num_not_owned_faces;
    }

    std::inclusive_scan(m_patcher->m_ribbon_ext_offset.begin(),
                        m_patcher->m_ribbon_ext_offset.end(),
                        m_patcher->m_ribbon_ext_offset.begin());


    for (uint32_t p = 0; p < get_num_patches(); ++p) {
        uint16_t offset = 0;
        uint32_t p_offset =
            (p == 0) ? 0 : m_patcher->m_ribbon_ext_offset[p - 1];

        for (uint16_t f = 0; f < m_h_patches_info[p].num_faces[0]; ++f) {
            LocalFaceT fl(f);
            if (!m_h_patches_info[p].is_owned(fl) &&
                !m_h_patches_info[p].is_deleted(fl)) {
                FaceHandle fh  = get_owner_handle<FaceHandle>({p, fl});
                uint32_t   fid = linear_id(fh);
                m_patcher->m_ribbon_ext_val[p_offset + offset++] = fid;
            }
        }
    }


    RXMesh::save(filename);
}


bool RXMeshDynamic::validate()
{
    bool cached_quite = this->m_quite;
    this->m_quite     = true;

    CUDA_ERROR(hipDeviceSynchronize());

    uint32_t num_patches;
    CUDA_ERROR(hipMemcpy(&num_patches,
                          m_rxmesh_context.m_num_patches,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    unsigned long long int* d_check;
    CUDA_ERROR(hipMalloc((void**)&d_check, sizeof(unsigned long long int)));

    assert(num_patches == get_num_patches());

    auto is_okay = [&]() {
        unsigned long long int h_check(0);
        CUDA_ERROR(hipMemcpy(&h_check,
                              d_check,
                              sizeof(unsigned long long int),
                              hipMemcpyDeviceToHost));
        if (h_check != 0) {
            return false;
        } else {
            return true;
        }
    };

    // check that the sum of owned vertices, edges, and faces per patch is equal
    // to the number of vertices, edges, and faces respectively
    auto check_num_mesh_elements = [&]() -> bool {
        uint32_t *d_sum_num_vertices, *d_sum_num_edges, *d_sum_num_faces;
        thrust::device_vector<uint32_t> d_sum_vertices(1, 0);
        thrust::device_vector<uint32_t> d_sum_edges(1, 0);
        thrust::device_vector<uint32_t> d_sum_faces(1, 0);

        constexpr uint32_t block_size = 256;
        const uint32_t     grid_size  = num_patches;

        detail::calc_num_elements<block_size>
            <<<grid_size, block_size>>>(m_rxmesh_context,
                                        d_sum_vertices.data().get(),
                                        d_sum_edges.data().get(),
                                        d_sum_faces.data().get());

        uint32_t num_vertices, num_edges, num_faces;
        CUDA_ERROR(hipMemcpy(&num_vertices,
                              m_rxmesh_context.m_num_vertices,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&num_edges,
                              m_rxmesh_context.m_num_edges,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(&num_faces,
                              m_rxmesh_context.m_num_faces,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));
        uint32_t sum_num_vertices, sum_num_edges, sum_num_faces;
        thrust::copy(
            d_sum_vertices.begin(), d_sum_vertices.end(), &sum_num_vertices);
        thrust::copy(d_sum_edges.begin(), d_sum_edges.end(), &sum_num_edges);
        thrust::copy(d_sum_faces.begin(), d_sum_faces.end(), &sum_num_faces);

        if (num_vertices != sum_num_vertices || num_edges != sum_num_edges ||
            num_faces != sum_num_faces) {
            return false;
        } else {
            return true;
        }
    };

    // check that each edge is composed of two unique vertices and each face is
    // composed of three unique edges that give three unique vertices.
    auto check_uniqueness = [&]() -> bool {
        CUDA_ERROR(hipMemset(d_check, 0, sizeof(unsigned long long int)));
        constexpr uint32_t block_size = 256;
        const uint32_t     grid_size  = num_patches;
        const uint32_t     dynamic_smem =
            rxmesh::ShmemAllocator::default_alignment * 2 +
            (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
            (2 * this->m_max_edges_per_patch) * sizeof(uint16_t);

        detail::check_uniqueness<block_size>
            <<<grid_size, block_size, dynamic_smem>>>(m_rxmesh_context,
                                                      d_check);

        return is_okay();
    };

    // check that every not-owned mesh elements' connectivity (faces and
    // edges) is equivalent to their connectivity in their owner patch.
    // if the mesh element is deleted in the owner patch, no check is done
    auto check_not_owned = [&]() -> bool {
        CUDA_ERROR(hipMemset(d_check, 0, sizeof(unsigned long long int)));
        CUDA_ERROR(hipMemset(d_check, 0, sizeof(unsigned long long int)));

        constexpr uint32_t block_size = 256;
        const uint32_t     grid_size  = num_patches;
        const uint32_t     dynamic_smem =
            ShmemAllocator::default_alignment * 2 +
            (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
            (2 * this->m_max_edges_per_patch) * sizeof(uint16_t);

        detail::check_not_owned<block_size>
            <<<grid_size, block_size, dynamic_smem>>>(m_rxmesh_context,
                                                      d_check);
        return is_okay();
    };

    // check if the ribbon construction is complete i.e., 1) each owned edge is
    // incident to an owned face, and 2) VF of the three vertices of an owned
    // face is inside the patch
    auto check_ribbon = [&]() {
        CUDA_ERROR(hipMemset(d_check, 0, sizeof(unsigned long long int)));
        constexpr uint32_t block_size = 512;
        const uint32_t     grid_size  = num_patches;
        uint32_t           dynamic_smem =
            ShmemAllocator::default_alignment * 3 +
            (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
            this->m_max_edges_per_patch * sizeof(uint16_t);

        detail::check_ribbon_edges<block_size>
            <<<grid_size, block_size, dynamic_smem>>>(m_rxmesh_context,
                                                      d_check);

        if (!is_okay()) {
            return false;
        }

        uint32_t* d_max_valence;
        CUDA_ERROR(hipMalloc((void**)&d_max_valence, sizeof(uint32_t)));
        CUDA_ERROR(hipMemset(d_max_valence, 0, sizeof(uint32_t)));

        LaunchBox<block_size> launch_box;
        RXMeshStatic::prepare_launch_box(
            {Op::VV},
            launch_box,
            (void*)detail::compute_max_valence<block_size>);
        detail::compute_max_valence<block_size>
            <<<launch_box.blocks, block_size, launch_box.smem_bytes_dyn>>>(
                m_rxmesh_context, d_max_valence);


        uint32_t h_max_valence = 0;
        CUDA_ERROR(hipMemcpy(&h_max_valence,
                              d_max_valence,
                              sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        GPU_FREE(d_max_valence);

        auto vf_global = this->add_vertex_attribute<FaceHandle>(
            "vf", h_max_valence, rxmesh::DEVICE);
        vf_global->reset(FaceHandle(), rxmesh::DEVICE);


        RXMeshStatic::prepare_launch_box(
            {Op::VF}, launch_box, (void*)detail::compute_vf<block_size>);

        detail::compute_vf<block_size>
            <<<launch_box.blocks, block_size, launch_box.smem_bytes_dyn>>>(
                m_rxmesh_context, *vf_global);

        dynamic_smem =
            ShmemAllocator::default_alignment * 3 +
            2 * (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
            std::max(3 * this->m_max_faces_per_patch,
                     2 * this->m_max_edges_per_patch) *
                sizeof(uint16_t);

        detail::check_ribbon_faces<block_size>
            <<<grid_size, block_size, dynamic_smem>>>(
                m_rxmesh_context, *vf_global, d_check);

        remove_attribute("vf");
        return is_okay();
    };


    // every mesh element in the ribbon of a patch should be mapped to an owner
    // that lives in a different. otherwise, the ribbon element is actually
    // duplicated. We check on this here
    auto check_unique_ribbon = [&]() {
        for (uint32_t p = 0; p < get_num_patches(); ++p) {
            PatchInfo pi = m_h_patches_info[p];

            for (uint32_t v = 0; v < pi.num_vertices[0]; v++) {
                const LocalVertexT vl(v);
                if (!pi.is_deleted(vl) && !pi.is_owned(vl)) {
                    const VertexHandle vh =
                        get_owner_handle<VertexHandle>({p, vl});
                    if (vh.patch_id() == p) {
                        return false;
                    }
                }
            }


            for (uint32_t e = 0; e < pi.num_edges[0]; e++) {
                const LocalEdgeT el(e);
                if (!pi.is_deleted(el) && !pi.is_owned(el)) {
                    const EdgeHandle eh = get_owner_handle<EdgeHandle>({p, el});
                    if (eh.patch_id() == p) {
                        return false;
                    }
                }
            }


            for (uint32_t f = 0; f < pi.num_faces[0]; f++) {
                const LocalFaceT fl(f);
                if (!pi.is_deleted(fl) && !pi.is_owned(fl)) {
                    const FaceHandle fh = get_owner_handle<FaceHandle>({p, fl});
                    if (fh.patch_id() == p) {
                        return false;
                    }
                }
            }
        }

        return true;
    };

    bool success = true;
    if (!check_num_mesh_elements()) {
        RXMESH_ERROR(
            "RXMeshDynamic::validate() check_num_mesh_elements failed");
        success = false;
    }

    if (!check_uniqueness()) {
        RXMESH_ERROR("RXMeshDynamic::validate() check_uniqueness failed");
        success = false;
    }

    if (!check_not_owned()) {
        RXMESH_ERROR("RXMeshDynamic::validate() check_not_owned failed");
        success = false;
    }

    if (!check_unique_ribbon()) {
        RXMESH_ERROR("RXMeshDynamic::validate() check_unique_ribbon failed");
        success = false;
    }

    if (!check_ribbon()) {
        RXMESH_ERROR("RXMeshDynamic::validate() check_ribbon failed");
        success = false;
    }

    CUDA_ERROR(hipFree(d_check));

    this->m_quite = cached_quite;

    return success;
}

void RXMeshDynamic::cleanup()
{
    constexpr uint32_t block_size = 256;
    const uint32_t     grid_size  = get_num_patches();

    CUDA_ERROR(hipMemcpy(&this->m_max_faces_per_patch,
                          this->m_rxmesh_context.m_max_num_vertices,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(&this->m_max_edges_per_patch,
                          this->m_rxmesh_context.m_max_num_edges,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    CUDA_ERROR(hipMemcpy(&this->m_max_faces_per_patch,
                          this->m_rxmesh_context.m_max_num_faces,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    uint32_t dyn_shmem = 2 * ShmemAllocator::default_alignment +
                         (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
                         (2 * this->m_max_edges_per_patch) * sizeof(uint16_t);

    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_vertices_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_edges_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_faces_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    detail::hashtable_calibration<block_size>
        <<<grid_size, block_size>>>(this->m_rxmesh_context);

    detail::remove_surplus_elements<block_size>
        <<<grid_size, block_size, dyn_shmem>>>(this->m_rxmesh_context);
}

void RXMeshDynamic::slice_patches(const uint32_t num_faces_threshold,
                                  rxmesh::FaceAttribute<int>&   f_attr,
                                  rxmesh::EdgeAttribute<int>&   e_attr,
                                  rxmesh::VertexAttribute<int>& v_attr)
{
    constexpr uint32_t block_size = 256;
    const uint32_t     grid_size  = get_num_patches();


    // ev, fe
    uint32_t dyn_shmem = 2 * ShmemAllocator::default_alignment +
                         (3 * this->m_max_faces_per_patch) * sizeof(uint16_t) +
                         (2 * this->m_max_edges_per_patch) * sizeof(uint16_t);

    // active_v/e/f, owned_v/e/f, patch_v/e/f
    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_vertices_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_edges_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    dyn_shmem += 3 * detail::mask_num_bytes(this->m_max_faces_per_patch) +
                 3 * ShmemAllocator::default_alignment;

    detail::slice_patches<block_size>
        <<<grid_size, block_size, dyn_shmem>>>(this->m_rxmesh_context,
                                               get_num_patches(),
                                               num_faces_threshold,
                                               f_attr,
                                               e_attr,
                                               v_attr);
}

void RXMeshDynamic::update_host()
{
    auto resize_masks = [&](uint16_t   size,
                            uint16_t&  capacity,
                            uint32_t*& active_mask,
                            uint32_t*& owned_mask) {
        if (size > capacity) {
            capacity = size;
            free(active_mask);
            free(owned_mask);
            active_mask = (uint32_t*)malloc(detail::mask_num_bytes(size));
            owned_mask  = (uint32_t*)malloc(detail::mask_num_bytes(size));
        }
    };

    uint32_t num_patches = 0;
    CUDA_ERROR(hipMemcpy(&num_patches,
                          m_rxmesh_context.m_num_patches,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    if (num_patches != m_num_patches) {
        RXMESH_ERROR(
            "RXMeshDynamic::update_host() does support changing number of "
            "patches in the mesh");
    }

    for (uint32_t p = 0; p < m_num_patches; ++p) {
        PatchInfo d_patch;
        CUDA_ERROR(hipMemcpy(&d_patch,
                              m_d_patches_info + p,
                              sizeof(PatchInfo),
                              hipMemcpyDeviceToHost));

        assert(d_patch.patch_id == p);

        CUDA_ERROR(hipMemcpy(m_h_patches_info[p].num_vertices,
                              d_patch.num_vertices,
                              sizeof(uint16_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(m_h_patches_info[p].num_edges,
                              d_patch.num_edges,
                              sizeof(uint16_t),
                              hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(m_h_patches_info[p].num_faces,
                              d_patch.num_faces,
                              sizeof(uint16_t),
                              hipMemcpyDeviceToHost));

        // resize topology (don't update capacity here)
        if (m_h_patches_info[p].num_edges[0] >
            m_h_patches_info[p].edges_capacity[0]) {
            free(m_h_patches_info[p].ev);
            m_h_patches_info[p].ev = (LocalVertexT*)malloc(
                m_h_patches_info[p].num_edges[0] * 2 * sizeof(LocalVertexT));
        }

        if (m_h_patches_info[p].num_faces[0] >
            m_h_patches_info[p].faces_capacity[0]) {
            free(m_h_patches_info[p].fe);
            m_h_patches_info[p].fe = (LocalEdgeT*)malloc(
                m_h_patches_info[p].num_faces[0] * 3 * sizeof(LocalEdgeT));
        }

        // copy topology
        CUDA_ERROR(hipMemcpy(
            m_h_patches_info[p].ev,
            d_patch.ev,
            2 * m_h_patches_info[p].num_edges[0] * sizeof(LocalVertexT),
            hipMemcpyDeviceToHost));

        CUDA_ERROR(hipMemcpy(
            m_h_patches_info[p].fe,
            d_patch.fe,
            3 * m_h_patches_info[p].num_faces[0] * sizeof(LocalEdgeT),
            hipMemcpyDeviceToHost));

        // resize mask (update capacity)
        resize_masks(m_h_patches_info[p].num_vertices[0],
                     m_h_patches_info[p].vertices_capacity[0],
                     m_h_patches_info[p].active_mask_v,
                     m_h_patches_info[p].owned_mask_v);

        resize_masks(m_h_patches_info[p].num_edges[0],
                     m_h_patches_info[p].edges_capacity[0],
                     m_h_patches_info[p].active_mask_e,
                     m_h_patches_info[p].owned_mask_e);

        resize_masks(m_h_patches_info[p].num_faces[0],
                     m_h_patches_info[p].faces_capacity[0],
                     m_h_patches_info[p].active_mask_f,
                     m_h_patches_info[p].owned_mask_f);

        // copy masks
        CUDA_ERROR(hipMemcpy(
            m_h_patches_info[p].active_mask_v,
            d_patch.active_mask_v,
            detail::mask_num_bytes(m_h_patches_info[p].num_vertices[0]),
            hipMemcpyDeviceToHost));
        CUDA_ERROR(hipMemcpy(
            m_h_patches_info[p].owned_mask_v,
            d_patch.owned_mask_v,
            detail::mask_num_bytes(m_h_patches_info[p].num_vertices[0]),
            hipMemcpyDeviceToHost));


        CUDA_ERROR(
            hipMemcpy(m_h_patches_info[p].active_mask_e,
                       d_patch.active_mask_e,
                       detail::mask_num_bytes(m_h_patches_info[p].num_edges[0]),
                       hipMemcpyDeviceToHost));
        CUDA_ERROR(
            hipMemcpy(m_h_patches_info[p].owned_mask_e,
                       d_patch.owned_mask_e,
                       detail::mask_num_bytes(m_h_patches_info[p].num_edges[0]),
                       hipMemcpyDeviceToHost));

        CUDA_ERROR(
            hipMemcpy(m_h_patches_info[p].active_mask_f,
                       d_patch.active_mask_f,
                       detail::mask_num_bytes(m_h_patches_info[p].num_faces[0]),
                       hipMemcpyDeviceToHost));
        CUDA_ERROR(
            hipMemcpy(m_h_patches_info[p].owned_mask_f,
                       d_patch.owned_mask_f,
                       detail::mask_num_bytes(m_h_patches_info[p].num_faces[0]),
                       hipMemcpyDeviceToHost));


        // copy patch stash
        CUDA_ERROR(hipMemcpy(m_h_patches_info[p].patch_stash.m_stash,
                              d_patch.patch_stash.m_stash,
                              PatchStash::stash_size * sizeof(uint32_t),
                              hipMemcpyDeviceToHost));

        // copy lp hashtable
        m_h_patches_info[p].lp_v.move(d_patch.lp_v);
        m_h_patches_info[p].lp_e.move(d_patch.lp_e);
        m_h_patches_info[p].lp_f.move(d_patch.lp_f);
    }


    CUDA_ERROR(hipMemcpy(&this->m_num_vertices,
                          m_rxmesh_context.m_num_vertices,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(&this->m_num_edges,
                          m_rxmesh_context.m_num_edges,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(&this->m_num_faces,
                          m_rxmesh_context.m_num_faces,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    // count and update num_owned and it prefix sum
    m_h_vertex_prefix[0] = 0;
    m_h_edge_prefix[0]   = 0;
    m_h_face_prefix[0]   = 0;
    for (uint32_t p = 0; p < m_num_patches; ++p) {
        m_h_num_owned_v[p] = m_h_patches_info[p].get_num_owned<VertexHandle>();
        m_h_vertex_prefix[p + 1] = m_h_vertex_prefix[p] + m_h_num_owned_v[p];

        m_h_num_owned_e[p] = m_h_patches_info[p].get_num_owned<EdgeHandle>();
        m_h_edge_prefix[p + 1] = m_h_edge_prefix[p] + m_h_num_owned_e[p];

        m_h_num_owned_f[p] = m_h_patches_info[p].get_num_owned<FaceHandle>();
        m_h_face_prefix[p + 1] = m_h_face_prefix[p] + m_h_num_owned_f[p];
    }

    if (m_h_vertex_prefix[m_num_patches] != this->m_num_vertices) {
        RXMESH_ERROR(
            "RXMeshDynamic::update_host error in updating host. m_num_vertices "
            "{} does not match m_h_vertex_prefix calculation {}",
            this->m_num_vertices,
            m_h_vertex_prefix[m_num_patches]);
    }

    if (m_h_edge_prefix[m_num_patches] != this->m_num_edges) {
        RXMESH_ERROR(
            "RXMeshDynamic::update_host error in updating host. m_num_edges "
            "{} does not match m_h_edge_prefix calculation {}",
            this->m_num_faces,
            m_h_face_prefix[m_num_patches]);
    }

    if (m_h_face_prefix[m_num_patches] != this->m_num_faces) {
        RXMESH_ERROR(
            "RXMeshDynamic::update_host error in updating host. m_num_faces "
            "{} does not match m_h_face_prefix calculation {}",
            this->m_num_edges,
            m_h_edge_prefix[m_num_patches]);
    }

    const uint32_t patches_1_bytes = (m_num_patches + 1) * sizeof(uint32_t);
    CUDA_ERROR(hipMemcpy(m_d_vertex_prefix,
                          m_h_vertex_prefix,
                          patches_1_bytes,
                          hipMemcpyHostToDevice));
    CUDA_ERROR(hipMemcpy(m_d_edge_prefix,
                          m_h_edge_prefix,
                          patches_1_bytes,
                          hipMemcpyHostToDevice));
    CUDA_ERROR(hipMemcpy(m_d_face_prefix,
                          m_h_face_prefix,
                          patches_1_bytes,
                          hipMemcpyHostToDevice));

    this->calc_max_elements();
}

void RXMeshDynamic::update_polyscope()
{
#if USE_POLYSCOPE
    // for polyscope, we just remove the mesh and re-add it since polyscope does
    // not support changing the mesh topology
    // if (this->m_polyscope_mesh_name.find("updated") != std::string::npos) {
    // polyscope::removeSurfaceMesh(this->m_polyscope_mesh_name, true);
    //}
    this->m_polyscope_mesh_name = this->m_polyscope_mesh_name + "updated";
    this->register_polyscope();
#endif
}

}  // namespace rxmesh
