#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdint.h>
#include <functional>
#include <iomanip>
#include <queue>
#include <unordered_map>
#include "cub/device/device_radix_sort.cuh"
#include "cub/device/device_scan.cuh"
#include "hip/hip_runtime_api.h"
#include "rxmesh/kernels/util.cuh"
#include "rxmesh/patcher/patcher.h"
#include "rxmesh/patcher/patcher_kernel.cuh"
#include "rxmesh/util/log.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/timer.h"
#include "rxmesh/util/util.h"


namespace rxmesh {


namespace patcher {

Patcher::Patcher(uint32_t                                        patch_size,
                 const std::vector<std::vector<uint32_t>>&       fvn,
                 const std::vector<std::vector<uint32_t>>&       ff,
                 const std::vector<std::vector<uint32_t>>&       fv,
                 const std::unordered_map<std::pair<uint32_t, uint32_t>,
                                          uint32_t,
                                          detail::edge_key_hash> edges_map,
                 const uint32_t                                  num_vertices,
                 const uint32_t                                  num_edges,
                 const bool                                      quite)
    : m_patch_size(patch_size),
      m_fvn(fvn),
      m_num_vertices(num_vertices),
      m_num_edges(num_edges),
      m_num_faces(fvn.size()),
      m_num_seeds(0),
      m_max_num_patches(0),
      m_num_components(0),
      m_patching_time_ms(0)
{

    m_num_patches =
        m_num_faces / m_patch_size + ((m_num_faces % m_patch_size) ? 1 : 0);

    m_max_num_patches = 5 * m_num_patches;

    m_num_seeds = m_num_patches;

    allocate_memory();

    // degenerate cases
    if (m_num_patches <= 1) {
        m_patches_offset[0] = m_num_faces;
        for (uint32_t i = 0; i < m_num_faces; ++i) {
            m_face_patch[i]  = 0;
            m_patches_val[i] = i;
        }
        m_neighbour_patches_offset.resize(1, 0);
        assign_patch(edges_map);
        if (!quite) {
            print_statistics();
        }
    } else {

        // parallel_execute(ef);
        postprocess();
        m_ribbon_ext_val.resize(m_ribbon_ext_offset[m_num_patches - 1]);

        // assign patches to vertices and edges
        assign_patch(edges_map);
        if (!quite) {
            print_statistics();
        }
    }
}

void Patcher::allocate_memory()
{
    m_seeds.reserve(m_num_seeds);

    // patches assigned to each face, vertex, and edge
    m_face_patch.resize(m_num_faces);
    std::fill(m_face_patch.begin(), m_face_patch.end(), INVALID32);

    m_vertex_patch.resize(m_num_vertices);
    std::fill(m_vertex_patch.begin(), m_vertex_patch.end(), INVALID32);

    m_edge_patch.resize(m_num_edges);
    std::fill(m_edge_patch.begin(), m_edge_patch.end(), INVALID32);

    // explicit patches in compressed format
    m_patches_val.resize(m_num_faces);

    // we allow up to double the number of faces due to patch bisecting
    m_patches_offset.resize(m_max_num_patches);

    // external ribbon. it assumes first that all faces will be in there and
    // then shrink to fit after the construction is done
    m_ribbon_ext_offset.resize(m_max_num_patches, 0);

    m_ribbon_ext_val.resize(m_num_faces);
}

void Patcher::print_statistics()
{
    RXMESH_TRACE("Patcher: num_patches = {}", m_num_patches);
    RXMESH_TRACE("Patcher: patches_size = {}", m_patch_size);
    RXMESH_TRACE("Patcher: num_components = {}", m_num_components);

    // patching time
    RXMESH_TRACE("Patcher: Num lloyd run = {}", m_num_lloyd_run);
    RXMESH_TRACE(
        "Patcher: Parallel patches construction time = {} (ms) and {} "
        "(ms/lloyd_run)",
        m_patching_time_ms,
        m_patching_time_ms / float(m_num_lloyd_run));

    // max-min patch size
    uint32_t max_patch_size(0), min_patch_size(m_num_faces), avg_patch_size(0);
    get_max_min_avg_patch_size(min_patch_size, max_patch_size, avg_patch_size);
    RXMESH_TRACE(
        "Patcher: max_patch_size= {}, min_patch_size= {}, avg_patch_size= {}",
        max_patch_size,
        min_patch_size,
        avg_patch_size);

    RXMESH_TRACE("Patcher: number external ribbon faces = {} ({:02.2f}%)",
                 get_num_ext_ribbon_faces(),
                 get_ribbon_overhead());
}

void Patcher::initialize_cluster_seeds()
{
    // cluster i.e., start from one triangle and grow in bfs style from it
    // for experiments only

    double   r = double(rand()) / double(RAND_MAX);
    uint32_t rand_face =
        static_cast<uint32_t>(r * static_cast<double>(m_num_faces - 1));
    std::queue<uint32_t> qu;
    qu.push(rand_face);

    std::vector<uint32_t> n_faces(3);
    std::vector<uint32_t> taken;
    taken.push_back(rand_face);

    while (true) {
        uint32_t current_face = qu.front();
        qu.pop();

        m_seeds.push_back(current_face);

        if (m_seeds.size() == m_num_seeds) {
            return;
        }

        get_adjacent_faces(current_face, n_faces);

        for (uint32_t i = 0; i < n_faces.size(); i++) {
            uint32_t ff = n_faces[i];
            if (ff == SPECIAL || ff == INVALID32 ||
                find_index(ff, taken) != std::numeric_limits<uint32_t>::max()) {
                continue;
            }
            qu.push(ff);
            taken.push_back(ff);
        }
    }
}

void Patcher::initialize_random_seeds()
{

    // 1) Identify the components i.e., for each component list the faces
    // that belong to that it
    // 2) Generate number of (random) seeds in each component
    // proportional to the number of faces it contain

    std::vector<std::vector<uint32_t>> components;
    get_multi_components(components);

    m_num_components = components.size();
    if (m_num_components == 1) {
        initialize_random_seeds_single_component();
    } else {
        if (m_num_seeds <= m_num_components) {
            // we have too many components so we increase the number of
            // seeds. this case should not be encountered frequently
            // since we generate only one seed per component
            m_num_seeds = m_num_components;
            for (auto& comp : components) {
                generate_random_seed_from_component(comp, 1);
            }
        } else {
            // if we have more seeds to give than the number of components,
            // then first secure that we have at least one seed per
            // component then we calculate the number of extra/remaining
            // seeds that will need be added. Every component then will have
            // a weight proportional to its size that tells how many of
            // these remaining seeds it can take

            uint32_t num_remaining_seeds      = m_num_seeds - m_num_components;
            uint32_t num_extra_seeds_inserted = 0;

            // sort the order of the component to be processed by their size
            std::vector<size_t> component_order(components.size());
            fill_with_sequential_numbers(component_order.data(),
                                         component_order.size());
            std::sort(component_order.begin(),
                      component_order.end(),
                      [&components](const size_t& a, const size_t& b) {
                          return components[a].size() > components[b].size();
                      });

            // process components in descending order with repsect to their
            // size
            for (size_t c = 0; c < component_order.size(); ++c) {

                std::vector<uint32_t>& comp = components[component_order[c]];

                uint32_t size = comp.size();
                // this weight tells how many extra faces this component
                // have from num_remaining_seeds
                float weight =
                    static_cast<float>(size) / static_cast<float>(m_num_faces);
                uint32_t component_num_seeds = static_cast<uint32_t>(std::ceil(
                    weight * static_cast<float>(num_remaining_seeds)));


                num_extra_seeds_inserted += component_num_seeds;
                if (num_extra_seeds_inserted > num_remaining_seeds) {
                    if (num_extra_seeds_inserted - num_remaining_seeds >
                        component_num_seeds) {
                        component_num_seeds = 0;
                    } else {
                        component_num_seeds -=
                            (num_extra_seeds_inserted - num_remaining_seeds);
                    }
                }

                component_num_seeds += 1;
                generate_random_seed_from_component(comp, component_num_seeds);
            }
        }
    }
}

void Patcher::initialize_random_seeds_single_component()
{
    // if not multi-component, just generate random number
    std::vector<uint32_t> rand_num(m_num_faces);
    fill_with_sequential_numbers(rand_num.data(), rand_num.size());
    random_shuffle(rand_num.data(), rand_num.size());
    m_seeds.resize(m_num_seeds);
    std::memcpy(
        m_seeds.data(), rand_num.data(), m_num_seeds * sizeof(uint32_t));
}

void Patcher::generate_random_seed_from_component(
    std::vector<uint32_t>& component,
    const uint32_t         num_seeds)
{
    // generate seeds from faces in component.
    // num_seeds is the number of seeds that will be generated
    uint32_t num_seeds_before = m_seeds.size();
    if (num_seeds < 1) {
        RXMESH_ERROR(
            "Patcher::generate_random_seed_in_component() num_seeds should be "
            "larger than 1");
    }

    random_shuffle(component.data(), component.size());
    m_seeds.resize(num_seeds_before + num_seeds);
    std::memcpy(m_seeds.data() + num_seeds_before,
                component.data(),
                num_seeds * sizeof(uint32_t));
}


void Patcher::get_multi_components(
    std::vector<std::vector<uint32_t>>& components)
{
    std::vector<bool>     visited(m_num_faces, false);
    std::vector<uint32_t> ff(3);
    for (uint32_t f = 0; f < m_num_faces; ++f) {
        if (!visited[f]) {
            std::vector<uint32_t> current_component;
            // just a guess
            current_component.reserve(
                static_cast<uint32_t>(static_cast<double>(m_num_faces) / 10.0));

            std::queue<uint32_t> face_queue;
            face_queue.push(f);
            while (!face_queue.empty()) {
                uint32_t current_face = face_queue.front();
                face_queue.pop();
                get_adjacent_faces(current_face, ff);

                for (const auto& f : ff) {
                    if (!visited[f]) {
                        current_component.push_back(f);
                        face_queue.push(f);
                        visited[f] = true;
                    }
                }
            }

            components.push_back(current_component);
        }
    }
}

void Patcher::postprocess()
{
    // Post process the patches by extracting the ribbons and populate the
    // neighbour patches storage
    //
    // For patch P, we start first by identifying boundary faces; faces that has
    // an edge on P's boundary. These faces are captured by querying the
    // adjacent faces for each face in P. If any of these adjacent faces are not
    // in the same patch, then this face is a boundary face. From these boundary
    // faces we can extract boundary vertices. We also now know which patch is
    // neighbor to P. Then we can use the boundary vertices to find the faces
    // that are incident to these vertices on the neighbor patches
    std::vector<uint32_t> tf(3);
    std::vector<uint32_t> frontier;
    frontier.reserve(m_num_faces);

    std::vector<uint32_t> bd_vertices;
    bd_vertices.reserve(m_patch_size);
    std::vector<uint32_t> vf1(3), vf2(3);

    m_neighbour_patches_offset.resize(m_num_patches);
    m_neighbour_patches.reserve(m_num_patches * 3);

    // build vertex incident faces
    std::vector<std::vector<uint32_t>> vertex_incident_faces(
        m_num_vertices, std::vector<uint32_t>(10));
    for (uint32_t i = 0; i < vertex_incident_faces.size(); ++i) {
        vertex_incident_faces[i].clear();
    }
    for (uint32_t face = 0; face < m_num_faces; ++face) {
        get_incident_vertices(face, vf1);
        for (uint32_t v = 0; v < vf1.size(); ++v) {
            vertex_incident_faces[vf1[v]].push_back(face);
        }
    }

    for (uint32_t cur_p = 0; cur_p < m_num_patches; ++cur_p) {

        uint32_t p_start = (cur_p == 0) ? 0 : m_patches_offset[cur_p - 1];
        uint32_t p_end   = m_patches_offset[cur_p];

        m_neighbour_patches_offset[cur_p] =
            (cur_p == 0) ? 0 : m_neighbour_patches_offset[cur_p - 1];
        uint32_t neighbour_patch_start = m_neighbour_patches_offset[cur_p];

        bd_vertices.clear();
        frontier.clear();


        //***** Pass One
        // 1) build a frontier of the boundary faces by loop over all faces and
        // add those that has an edge on the patch boundary
        for (uint32_t fb = p_start; fb < p_end; ++fb) {
            uint32_t face = m_patches_val[fb];

            get_adjacent_faces(face, tf);

            bool added = false;
            for (uint32_t g = 0; g < tf.size(); ++g) {
                uint32_t n       = tf[g];
                uint32_t n_patch = get_face_patch_id(n);

                // n is boundary face if its patch is not the current patch we
                // are processing
                if (n_patch != cur_p) {
                    if (!added) {
                        frontier.push_back(face);
                        added = true;
                    }

                    // add n_patch as a neighbour patch to the current patch
                    auto itt = std::find(
                        m_neighbour_patches.begin() + neighbour_patch_start,
                        m_neighbour_patches.end(),
                        n_patch);

                    if (itt == m_neighbour_patches.end()) {
                        m_neighbour_patches.push_back(n_patch);
                        ++m_neighbour_patches_offset[cur_p];
                        assert(m_neighbour_patches_offset[cur_p] ==
                               m_neighbour_patches.size());
                    }

                    // find/add the boundary vertices; these are the vertices
                    // that are shared between face and n
                    get_incident_vertices(face, vf1);
                    get_incident_vertices(n, vf2);

                    // add the common vertices in vf1 and vf2
                    for (uint32_t i = 0; i < vf1.size(); ++i) {
                        auto it_vf = std::find(vf2.begin(), vf2.end(), vf1[i]);
                        if (it_vf != vf2.end()) {
                            bd_vertices.push_back(vf1[i]);
                        }
                    }


                    // we don't break out of this loop because we want to get
                    // all the neighbour patches and boundary vertices
                    // break;
                }
            }
        }

        // Sort boundary vertices so we can use binary_search
        std::sort(bd_vertices.begin(), bd_vertices.end());
        // remove duplicated vertices
        inplace_remove_duplicates_sorted(bd_vertices);


        //***** Pass Two

        // 3) for every vertex on the patch boundary, we add all the faces
        // that are incident to it and not in the current patch

        m_ribbon_ext_offset[cur_p] =
            (cur_p == 0) ? 0 : m_ribbon_ext_offset[cur_p - 1];
        uint32_t r_start = m_ribbon_ext_offset[cur_p];

        for (uint32_t v = 0; v < bd_vertices.size(); ++v) {
            uint32_t vert = bd_vertices[v];

            for (uint32_t f = 0; f < vertex_incident_faces[vert].size(); ++f) {
                uint32_t face = vertex_incident_faces[vert][f];
                if (get_face_patch_id(face) != cur_p) {
                    // make sure we have not added face before
                    bool     added = false;
                    uint32_t r_end = m_ribbon_ext_offset[cur_p];
                    for (uint32_t r = r_start; r < r_end; ++r) {
                        if (m_ribbon_ext_val[r] == face) {
                            added = true;
                            break;
                        }
                    }
                    if (!added) {

                        m_ribbon_ext_val[m_ribbon_ext_offset[cur_p]] = face;
                        m_ribbon_ext_offset[cur_p]++;
                        if (m_ribbon_ext_offset[cur_p] == m_num_faces) {
                            // need to expand m_ribbon_ext_val. This occurs
                            // mostly for small meshes with small patch size
                            // such that the amount overlap between exterior
                            // ribbon of different patches is larger than
                            // m_num_faces
                            uint32_t new_size = m_ribbon_ext_val.size() * 2;
                            m_ribbon_ext_val.resize(new_size);
                        }
                        assert(m_ribbon_ext_offset[cur_p] <=
                               m_ribbon_ext_val.size());
                    }
                }
            }
        }
    }
}

void Patcher::get_adjacent_faces(uint32_t               face_id,
                                 std::vector<uint32_t>& ff) const
{
    if (m_fvn.size() != 0) {
        // We account here for non-manifold cases where a face might not be
        // adjacent to just three faces
        uint32_t size = m_fvn[face_id].size() - 3;
        ff.resize(size);
        std::memcpy(
            ff.data(), m_fvn[face_id].data() + 3, size * sizeof(uint32_t));
    } else {
        RXMESH_ERROR(
            "Patcher::get_adjacent_faces() can not get adjacent faces!!");
    }
}

void Patcher::get_incident_vertices(uint32_t face_id, std::vector<uint32_t>& fv)
{
    if (m_fvn.size() != 0) {
        fv.resize(3);
        std::memcpy(fv.data(), m_fvn[face_id].data(), 3 * sizeof(uint32_t));
    } else {
        RXMESH_ERROR(
            "Patcher::get_incident_vertices() can not get adjacent faces!!");
    }
}

void Patcher::assign_patch(
    const std::unordered_map<std::pair<uint32_t, uint32_t>,
                             uint32_t,
                             ::rxmesh::detail::edge_key_hash> edges_map)
{
    // For every patch p, for every face in the patch, find the three edges
    // that bound that face, and assign them to the patch. For boundary vertices
    // and edges assign them to one patch (TODO smallest face count). For now,
    // we assign it to the first patch

    std::vector<uint32_t> vf1(3);

    for (uint32_t cur_p = 0; cur_p < m_num_patches; ++cur_p) {

        uint32_t p_start = (cur_p == 0) ? 0 : m_patches_offset[cur_p - 1];
        uint32_t p_end   = m_patches_offset[cur_p];

        for (uint32_t f = p_start; f < p_end; ++f) {

            uint32_t face = m_patches_val[f];

            get_incident_vertices(face, vf1);

            uint32_t v1 = vf1.back();
            for (uint32_t v = 0; v < vf1.size(); ++v) {
                uint32_t v0 = vf1[v];

                std::pair<uint32_t, uint32_t> key =
                    ::rxmesh::detail::edge_key(v0, v1);
                uint32_t edge_id = edges_map.at(key);

                if (m_vertex_patch[v0] == INVALID32) {
                    m_vertex_patch[v0] = cur_p;
                }

                if (m_edge_patch[edge_id] == INVALID32) {
                    m_edge_patch[edge_id] = cur_p;
                }

                v1 = v0;
            }
        }
    }
}

void Patcher::assign_patch(
    std::function<uint32_t(uint32_t, uint32_t)> get_edge_id)
{
    // For every patch p, for every face in the patch, find the three edges
    // that bound that face, and assign them to the patch. For boundary vertices
    // and edges assign them to one patch (TODO smallest face count). For now,
    // we assign it to the first patch

    std::vector<uint32_t> vf1(3);

    for (uint32_t cur_p = 0; cur_p < m_num_patches; ++cur_p) {

        uint32_t p_start = (cur_p == 0) ? 0 : m_patches_offset[cur_p - 1];
        uint32_t p_end   = m_patches_offset[cur_p];

        for (uint32_t f = p_start; f < p_end; ++f) {

            uint32_t face = m_patches_val[f];

            get_incident_vertices(face, vf1);

            uint32_t v1 = vf1.back();
            for (uint32_t v = 0; v < vf1.size(); ++v) {
                uint32_t v0 = vf1[v];

                uint32_t edge_id = get_edge_id(v0, v1);

                if (m_vertex_patch[v0] == INVALID32) {
                    m_vertex_patch[v0] = cur_p;
                }

                if (m_edge_patch[edge_id] == INVALID32) {
                    m_edge_patch[edge_id] = cur_p;
                }

                v1 = v0;
            }
        }
    }
}

void Patcher::populate_ff(const std::vector<std::vector<uint32_t>>& ef,
                          std::vector<uint32_t>&                    h_ff_values,
                          std::vector<uint32_t>&                    h_ff_offset)
{
    assert(ef.size() == m_num_edges);
    uint32_t                           total_ff_values = 0;
    std::vector<std::vector<uint32_t>> h_ff_values_vec;
    for (uint32_t f = 0; f < m_num_faces; ++f) {
        std::vector<uint32_t> ff;
        ff.reserve(3);
        h_ff_values_vec.push_back(ff);
    }
    for (uint32_t e = 0; e < ef.size(); ++e) {
        for (uint32_t f0 = 0; f0 < ef[e].size() - 1; ++f0) {
            uint32_t face0 = ef[e][f0];
            for (uint32_t f1 = f0 + 1; f1 < ef[e].size(); ++f1) {
                uint32_t face1 = ef[e][f1];
                total_ff_values += 2;
                h_ff_values_vec[face0].push_back(face1);
                h_ff_values_vec[face1].push_back(face0);
            }
        }
    }

    h_ff_offset.clear();
    h_ff_offset.resize(m_num_faces);
    for (uint32_t f = 0; f < m_num_faces; ++f) {
        uint32_t s = 0;
        if (f != 0) {
            s = h_ff_offset[f - 1];
        }
        h_ff_offset[f] = s + h_ff_values_vec[f].size();
    }
    assert(h_ff_offset.back() == total_ff_values);
    h_ff_values.clear();
    h_ff_values.reserve(total_ff_values);
    for (uint32_t f = 0; f < m_num_faces; ++f) {
        for (uint32_t ff = 0; ff < h_ff_values_vec[f].size(); ff++) {
            h_ff_values.push_back(h_ff_values_vec[f][ff]);
        }
    }
}

void Patcher::parallel_execute(const std::vector<std::vector<uint32_t>>& ef)
{
    // TODO use streams
    // TODO we don't need ef. We only use it to compute FF which we already
    // compute in RXMesh build_local method before invoking patcher.

    // adjacent faces
    uint32_t *d_ff_values(nullptr), *d_ff_offset(nullptr);
    {
        std::vector<uint32_t> h_ff_values, h_ff_offset;
        populate_ff(ef, h_ff_values, h_ff_offset);
        assert(h_ff_offset.size() == m_num_faces);
        CUDA_ERROR(hipMalloc((void**)&d_ff_values,
                              h_ff_values.size() * sizeof(uint32_t)));
        CUDA_ERROR(hipMalloc((void**)&d_ff_offset,
                              h_ff_offset.size() * sizeof(uint32_t)));

        CUDA_ERROR(hipMemcpy(d_ff_values,
                              h_ff_values.data(),
                              h_ff_values.size() * sizeof(uint32_t),
                              hipMemcpyHostToDevice));
        CUDA_ERROR(hipMemcpy(d_ff_offset,
                              h_ff_offset.data(),
                              h_ff_offset.size() * sizeof(uint32_t),
                              hipMemcpyHostToDevice));
    }


    // faces patch
    uint32_t* d_face_patch = nullptr;
    CUDA_ERROR(
        hipMalloc((void**)&d_face_patch, m_num_faces * sizeof(uint32_t)));

    // seeds (allocate m_max_num_patches but copy only m_num_patches)
    initialize_random_seeds();
    uint32_t* d_seeds = nullptr;
    assert(m_num_patches == m_seeds.size());
    CUDA_ERROR(
        hipMalloc((void**)&d_seeds, m_max_num_patches * sizeof(uint32_t)));
    CUDA_ERROR(hipMemcpy(d_seeds,
                          m_seeds.data(),
                          m_num_patches * sizeof(uint32_t),
                          hipMemcpyHostToDevice));


    // queue of size num_faces
    // queue_start and queue_end
    uint32_t* d_queue = nullptr;
    CUDA_ERROR(hipMalloc((void**)&d_queue, m_num_faces * sizeof(uint32_t)));

    // 0 -> queue start
    // 1-> queue end
    // 2-> next queue end
    std::vector<uint32_t> h_queue_ptr{0, m_num_patches, m_num_patches};
    uint32_t*             d_queue_ptr;
    CUDA_ERROR(hipMalloc((void**)&d_queue_ptr, 3 * sizeof(uint32_t)));
    CUDA_ERROR(hipMemcpy(d_queue_ptr,
                          h_queue_ptr.data(),
                          3 * sizeof(uint32_t),
                          hipMemcpyHostToDevice));

    // patches offset, values, and size
    uint32_t *d_patches_offset, *d_patches_val, *d_patches_size,
        *d_max_patch_size;
    CUDA_ERROR(hipMalloc((void**)&d_patches_offset,
                          m_max_num_patches * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&d_patches_size,
                          m_max_num_patches * sizeof(uint32_t)));
    CUDA_ERROR(
        hipMalloc((void**)&d_patches_val, m_num_faces * sizeof(uint32_t)));
    CUDA_ERROR(hipMalloc((void**)&d_max_patch_size, sizeof(uint32_t)));
    void * d_cub_temp_storage_scan(nullptr), *d_cub_temp_storage_max(nullptr);
    size_t cub_temp_storage_bytes_scan = 0;
    size_t cub_temp_storage_bytes_max  = 0;
    ::hipcub::DeviceScan::InclusiveSum(d_cub_temp_storage_scan,
                                    cub_temp_storage_bytes_scan,
                                    d_patches_size,
                                    d_patches_offset,
                                    m_max_num_patches);
    ::hipcub::DeviceReduce::Max(d_cub_temp_storage_max,
                             cub_temp_storage_bytes_max,
                             d_patches_size,
                             d_max_patch_size,
                             m_max_num_patches);
    CUDA_ERROR(hipMalloc((void**)&d_cub_temp_storage_scan,
                          cub_temp_storage_bytes_scan));
    CUDA_ERROR(hipMalloc((void**)&d_cub_temp_storage_max,
                          cub_temp_storage_bytes_max));

    // Lloyd iterations loop
    uint32_t* d_new_num_patches = nullptr;
    CUDA_ERROR(hipMalloc((void**)&d_new_num_patches, sizeof(uint32_t)));
    CUDA_ERROR(hipMemcpy(d_new_num_patches,
                          &m_num_patches,
                          sizeof(uint32_t),
                          hipMemcpyHostToDevice));


    CUDA_ERROR(hipProfilerStart());
    GPUTimer timer;
    timer.start();

    m_num_lloyd_run = 0;
    while (true) {
        ++m_num_lloyd_run;

        const uint32_t threads_s = 256;
        const uint32_t blocks_s  = DIVIDE_UP(m_num_patches, threads_s);
        const uint32_t threads_f = 256;
        const uint32_t blocks_f  = DIVIDE_UP(m_num_faces, threads_f);

        // add more seeds if needed
        if (m_num_lloyd_run % 5 == 0 && m_num_lloyd_run > 0) {
            uint32_t threshold = m_patch_size;

            CUDA_ERROR(hipMemcpy(d_new_num_patches,
                                  &m_num_patches,
                                  sizeof(uint32_t),
                                  hipMemcpyHostToDevice));
            add_more_seeds<<<m_num_patches, 1>>>(m_num_patches,
                                                 d_new_num_patches,
                                                 d_seeds,
                                                 d_patches_offset,
                                                 d_patches_val,
                                                 threshold);

            CUDA_ERROR(hipMemcpy(&m_num_patches,
                                  d_new_num_patches,
                                  sizeof(uint32_t),
                                  hipMemcpyDeviceToHost));

            if (m_num_patches >= m_max_num_patches) {
                RXMESH_ERROR(
                    "Patcher::parallel_execute() m_num_patches exceeds "
                    "m_max_num_patches");
            }
        }
        h_queue_ptr[0] = 0;
        h_queue_ptr[1] = m_num_patches;
        h_queue_ptr[2] = m_num_patches;
        CUDA_ERROR(hipMemcpy(d_queue_ptr,
                              h_queue_ptr.data(),
                              3 * sizeof(uint32_t),
                              hipMemcpyHostToDevice));

        rxmesh::memset<<<blocks_f, threads_f>>>(
            d_face_patch, INVALID32, m_num_faces);

        rxmesh::memcpy<<<blocks_s, threads_s>>>(
            d_queue, d_seeds, m_num_patches);

        rxmesh::memset<<<blocks_s, threads_s>>>(
            d_patches_size, 0u, m_num_patches);

        write_initial_face_patch<<<blocks_s, threads_s>>>(
            m_num_patches, d_face_patch, d_seeds, d_patches_size);

        // Cluster seed propagation
        while (true) {
            // Launch enough threads to cover all the faces. However, only
            // subset will do actual work depending on the queue size
            cluster_seed_propagation<<<blocks_f, threads_f>>>(m_num_faces,
                                                              m_num_patches,
                                                              d_queue_ptr,
                                                              d_queue,
                                                              d_face_patch,
                                                              d_patches_size,
                                                              d_ff_offset,
                                                              d_ff_values);

            reset_queue_ptr<<<1, 1>>>(d_queue_ptr);

            CUDA_ERROR(hipMemcpy(h_queue_ptr.data(),
                                  d_queue_ptr,
                                  sizeof(uint32_t),
                                  hipMemcpyDeviceToHost));

            if (h_queue_ptr[0] >= m_num_faces) {
                break;
            }
        }


        uint32_t max_patch_size =
            construct_patches_compressed_parallel(d_cub_temp_storage_max,
                                                  cub_temp_storage_bytes_max,
                                                  d_patches_size,
                                                  d_max_patch_size,
                                                  d_cub_temp_storage_scan,
                                                  cub_temp_storage_bytes_scan,
                                                  d_patches_offset,
                                                  d_face_patch,
                                                  d_patches_val);
        // Interior
        uint32_t threads_i   = 512;
        uint32_t shmem_bytes = max_patch_size * (sizeof(uint32_t));
        rxmesh::memset<<<blocks_f, threads_f>>>(
            d_queue, INVALID32, m_num_faces);
        interior<<<m_num_patches, threads_i, shmem_bytes>>>(m_num_patches,
                                                            d_patches_offset,
                                                            d_patches_val,
                                                            d_face_patch,
                                                            d_seeds,
                                                            d_ff_offset,
                                                            d_ff_values,
                                                            d_queue);

        if (max_patch_size < m_patch_size) {
            break;
        }
    }


    timer.stop();
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());
    m_patching_time_ms = timer.elapsed_millis();
    CUDA_ERROR(hipProfilerStop());


    // move data to host
    m_num_seeds = m_num_patches;
    m_seeds.resize(m_num_seeds);
    CUDA_ERROR(hipMemcpy(m_seeds.data(),
                          d_seeds,
                          m_num_seeds * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(m_face_patch.data(),
                          d_face_patch,
                          sizeof(uint32_t) * m_num_faces,
                          hipMemcpyDeviceToHost));
    m_patches_offset.resize(m_num_patches);
    CUDA_ERROR(hipMemcpy(m_patches_offset.data(),
                          d_patches_offset,
                          sizeof(uint32_t) * m_num_patches,
                          hipMemcpyDeviceToHost));
    CUDA_ERROR(hipMemcpy(m_patches_val.data(),
                          d_patches_val,
                          sizeof(uint32_t) * m_num_faces,
                          hipMemcpyDeviceToHost));


    // draw();

    for (uint32_t i = 0; i < m_num_faces; ++i) {
        m_face_patch[i]  = m_face_patch[i] >> 1;
        m_patches_val[i] = m_patches_val[i] >> 1;
    }


    GPU_FREE(d_ff_values);
    GPU_FREE(d_ff_offset);
    GPU_FREE(d_face_patch);
    GPU_FREE(d_seeds);
    GPU_FREE(d_queue);
    GPU_FREE(d_patches_offset);
    GPU_FREE(d_patches_size);
    GPU_FREE(d_patches_val);
    GPU_FREE(d_queue_ptr);
    GPU_FREE(d_cub_temp_storage_scan);
    GPU_FREE(d_cub_temp_storage_max);
    GPU_FREE(d_max_patch_size);
    GPU_FREE(d_new_num_patches);
}

uint32_t Patcher::construct_patches_compressed_parallel(
    void*     d_cub_temp_storage_max,
    size_t    cub_temp_storage_bytes_max,
    uint32_t* d_patches_size,
    uint32_t* d_max_patch_size,
    void*     d_cub_temp_storage_scan,
    size_t    cub_temp_storage_bytes_scan,
    uint32_t* d_patches_offset,
    uint32_t* d_face_patch,
    uint32_t* d_patches_val)
{
    uint32_t       max_patch_size = 0;
    const uint32_t threads_s      = 256;
    const uint32_t blocks_s       = DIVIDE_UP(m_num_patches, threads_s);
    const uint32_t threads_f      = 256;
    const uint32_t blocks_f       = DIVIDE_UP(m_num_faces, threads_f);

    // Compute max patch size
    max_patch_size = 0;
    ::hipcub::DeviceReduce::Max(d_cub_temp_storage_max,
                             cub_temp_storage_bytes_max,
                             d_patches_size,
                             d_max_patch_size,
                             m_num_patches);
    CUDA_ERROR(hipMemcpy(&max_patch_size,
                          d_max_patch_size,
                          sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    // Construct compressed patches
    ::hipcub::DeviceScan::InclusiveSum(d_cub_temp_storage_scan,
                                    cub_temp_storage_bytes_scan,
                                    d_patches_size,
                                    d_patches_offset,
                                    m_num_patches);
    rxmesh::memset<<<blocks_s, threads_s>>>(d_patches_size, 0u, m_num_patches);

    construct_patches_compressed<<<blocks_f, threads_f>>>(m_num_faces,
                                                          d_face_patch,
                                                          m_num_patches,
                                                          d_patches_offset,
                                                          d_patches_size,
                                                          d_patches_val);

    return max_patch_size;
}

}  // namespace patcher
}  // namespace rxmesh