#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/rxmesh_attribute.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/vector.h"

/**
 * test_vector()
 */
__global__ static void test_vector(
    rxmesh::RXMeshAttribute<rxmesh::Vector3f> mesh_attr,
    uint32_t*                                 suceess)
{

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *suceess = 1;

        assert((mesh_attr.get_allocated() & rxmesh::DEVICE) == rxmesh::DEVICE);
        uint32_t num_mesh_elements = mesh_attr.get_num_mesh_elements();
        for (uint32_t i = 0; i < num_mesh_elements; ++i) {
            const auto& vec = mesh_attr(i);
            if (vec[0] != i + 0 || vec[1] != i + 1 || vec[2] != i + 2) {
                *suceess = 0;
                return;
            }
        }
    }
}

/**
 * test_values()
 */
template <class T>
__global__ static void test_values(rxmesh::RXMeshAttribute<T> mesh_attr,
                                   uint32_t*                  suceess)
{

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *suceess = 1;

        assert((mesh_attr.get_allocated() & rxmesh::DEVICE) == rxmesh::DEVICE);
        uint32_t num_mesh_elements = mesh_attr.get_num_mesh_elements();
        for (uint32_t i = 0; i < num_mesh_elements; ++i) {
            for (uint32_t j = 0; j < mesh_attr.get_num_attribute_per_element();
                 ++j) {
                if (mesh_attr(i, j) != i + j) {

                    *suceess = 0;
                    return;
                }
            }
        }
    }
}

/**
 * generate_values()
 */
template <class T>
__global__ static void generate_values(rxmesh::RXMeshAttribute<T> mesh_attr)
{

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        assert((mesh_attr.get_allocated() & rxmesh::DEVICE) == rxmesh::DEVICE);

        uint32_t num_mesh_elements = mesh_attr.get_num_mesh_elements();
        for (uint32_t i = 0; i < num_mesh_elements; ++i) {
            for (uint32_t j = 0; j < mesh_attr.get_num_attribute_per_element();
                 ++j) {
                mesh_attr(i, j) = i + j;
            }
        }
    }
}


bool test_host(uint32_t attributes_per_element)
{
    using namespace rxmesh;
    // mesh attr on host
    uint32_t                       num_mesh_elements = 2048;
    rxmesh::RXMeshAttribute<float> rxmesh_attr;

    rxmesh_attr.set_name("float_attr");
    rxmesh_attr.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            rxmesh_attr(i, j) = i + j;
        }
    }

    // change the layout to SoA (good for gpu)
    rxmesh_attr.change_layout(rxmesh::HOST);

    // move memory to device
    rxmesh_attr.move(rxmesh::HOST, rxmesh::DEVICE);


    // device success variable
    uint32_t* d_success = nullptr;
    CUDA_ERROR(hipMalloc((void**)&d_success, sizeof(uint32_t)));


    // actual testing
    test_values<float><<<1, 1>>>(rxmesh_attr, d_success);

    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());
    CUDA_ERROR(hipDeviceSynchronize());

    // host success variable
    uint32_t h_success(0);
    CUDA_ERROR(hipMemcpy(
        &h_success, d_success, sizeof(uint32_t), hipMemcpyDeviceToHost));

    // free device
    GPU_FREE(d_success);

    // release rxmesh_attribute memory on host and device
    rxmesh_attr.release();

    // reporting
    return h_success == 1;
}


bool test_device(uint32_t attributes_per_element)
{
    using namespace rxmesh;
    // Test generating values on device and processing it on host

    // mesh attr on host (but allocated on device)
    uint32_t                          num_mesh_elements = 2048;
    rxmesh::RXMeshAttribute<uint32_t> rxmesh_attr;
    rxmesh_attr.set_name("int_attr");
    rxmesh_attr.init(num_mesh_elements, attributes_per_element, rxmesh::DEVICE);


    // generate some numbers on device
    generate_values<<<1, 1>>>(rxmesh_attr);

    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipGetLastError());


    // move the generate values to host
    rxmesh_attr.move(rxmesh::DEVICE, rxmesh::HOST);

    // change the layout to SoA
    rxmesh_attr.change_layout(rxmesh::HOST);

    // testing
    bool suceess = true;
    assert((rxmesh_attr.get_allocated() & rxmesh::HOST) == rxmesh::HOST);
    num_mesh_elements = rxmesh_attr.get_num_mesh_elements();

    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            if (rxmesh_attr(i, j) != i + j) {
                suceess = false;
                break;
            }
        }
        if (!suceess) {
            break;
        }
    }

    // release rxmesh_attribute memory on host and device
    rxmesh_attr.release();

    return suceess;
}

/*bool test_vector()
{
    using namespace rxmesh;
    // mesh attr on host
    uint32_t                         num_mesh_elements = 2048;
    rxmesh::RXMeshAttribute<Vector3f> rxmesh_attr;

    rxmesh_attr.set_name("vector3f_attr");
    rxmesh_attr.init(num_mesh_elements, 1, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        auto& vec = rxmesh_attr(i);
        vec[0] = i + 0;
        vec[1] = i + 1;
        vec[2] = i + 2;
    }

    // move memory to device
    rxmesh_attr.move(rxmesh::HOST, rxmesh::DEVICE);


    // device success variable
    uint32_t* d_success = nullptr;
    CUDA_ERROR(hipMalloc((void**)&d_success, sizeof(uint32_t)));


    // actual testing
    test_vector<<<1, 1>>>(rxmesh_attr, d_success);

    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());
    CUDA_ERROR(hipDeviceSynchronize());

    // host success variable
    uint32_t h_success(0);
    CUDA_ERROR(hipMemcpy(&h_success, d_success, sizeof(uint32_t),
                          hipMemcpyDeviceToHost));

    // free device
    GPU_FREE(d_success);

    // release rxmesh_attribute memory on host and device
    rxmesh_attr.release();

    // reporting
    return h_success == 1;
}*/

bool test_axpy(uint32_t attributes_per_element)
{
    using namespace rxmesh;

    float x_val(1.0), y_val(3.0), alpha_val(5.0), beta_val(7.0);

    uint32_t                       num_mesh_elements = 2048;
    rxmesh::RXMeshAttribute<float> X;
    rxmesh::RXMeshAttribute<float> Y;

    X.set_name("X");
    Y.set_name("Y");
    X.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);
    Y.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            X(i, j) = x_val;
            Y(i, j) = y_val;
        }
    }

    X.change_layout(rxmesh::HOST);
    Y.change_layout(rxmesh::HOST);
    X.move(rxmesh::HOST, rxmesh::DEVICE);
    Y.move(rxmesh::HOST, rxmesh::DEVICE);

    // call axpy
    Vector<3, float> alpha(alpha_val);
    Vector<3, float> beta(beta_val);
    Y.axpy(X, alpha, beta);

    // sync
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());


    // move to host (don't need to move X
    Y.move(rxmesh::DEVICE, rxmesh::HOST);

    // check results
    bool is_passed = true;
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            if (std::abs(Y(i, j) - (alpha_val * x_val + beta_val * y_val)) >
                0.0001) {
                is_passed = false;
                break;
            }
        }
        if (!is_passed) {
            break;
        }
    }

    // release rxmesh_attribute memory on host and device
    X.release();
    Y.release();


    return is_passed;
}


bool test_reduce()
{
    using namespace rxmesh;
    constexpr uint32_t             attributes_per_element = 3;
    uint32_t                       num_mesh_elements      = 2048;
    rxmesh::RXMeshAttribute<float> X;

    X.set_name("X");
    X.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            X(i, j) = j + 1;
        }
    }

    X.change_layout(rxmesh::HOST);
    X.move(rxmesh::HOST, rxmesh::DEVICE);
    Vector<attributes_per_element, float> output;

    // call reduce
    X.reduce(output, rxmesh::SUM);


    // sync
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());

    bool is_passed = true;

    for (uint32_t j = 0; j < attributes_per_element; ++j) {
        if (output[j] != num_mesh_elements * (j + 1)) {
            is_passed = false;
            break;
        }
    }

    // release rxmesh_attribute memory on host and device
    X.release();


    return is_passed;
}


bool test_norm2()
{
    using namespace rxmesh;
    constexpr uint32_t             attributes_per_element = 3;
    uint32_t                       num_mesh_elements      = 2048;
    rxmesh::RXMeshAttribute<float> X;

    X.set_name("X");
    X.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            X(i, j) = 2;
        }
    }

    X.change_layout(rxmesh::HOST);
    X.move(rxmesh::HOST, rxmesh::DEVICE);
    Vector<attributes_per_element, float> output;

    // call reduce
    X.reduce(output, rxmesh::NORM2);


    // sync
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());

    bool is_passed = true;

    for (uint32_t j = 0; j < attributes_per_element; ++j) {
        if (output[j] != 4 * num_mesh_elements) {
            is_passed = false;
            break;
        }
    }

    // release rxmesh_attribute memory on host and device
    X.release();


    return is_passed;
}


bool test_dot()
{
    using namespace rxmesh;
    constexpr uint32_t             attributes_per_element = 3;
    uint32_t                       num_mesh_elements      = 2048;
    rxmesh::RXMeshAttribute<float> X;
    rxmesh::RXMeshAttribute<float> Y;

    X.set_name("X");
    Y.set_name("Y");
    X.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);
    Y.init(
        num_mesh_elements, attributes_per_element, rxmesh::HOST, rxmesh::AoS);

    // generate some numbers as AoS
    for (uint32_t i = 0; i < num_mesh_elements; ++i) {
        for (uint32_t j = 0; j < attributes_per_element; ++j) {
            X(i, j) = 2;
            Y(i, j) = 3;
        }
    }

    X.change_layout(rxmesh::HOST);
    X.move(rxmesh::HOST, rxmesh::DEVICE);
    Y.change_layout(rxmesh::HOST);
    Y.move(rxmesh::HOST, rxmesh::DEVICE);
    Vector<attributes_per_element, float> output;

    // call reduce
    X.reduce(output, rxmesh::DOT, &Y);


    // sync
    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipPeekAtLastError());
    CUDA_ERROR(hipGetLastError());

    bool is_passed = true;

    for (uint32_t j = 0; j < attributes_per_element; ++j) {
        if (output[j] != 6 * num_mesh_elements) {
            is_passed = false;
            break;
        }
    }

    // release rxmesh_attribute memory on host and device
    X.release();
    Y.release();


    return is_passed;
}


TEST(RXMesh, Attributes)
{
    using namespace rxmesh;
    EXPECT_TRUE(test_host(3u)) << " TestAttributes::tes_host failed";
    EXPECT_TRUE(test_device(3u)) << " TestAttributes::tes_device failed";
    // EXPECT_TRUE(test_vector()) << " TestAttributes::test_vector failed";
    EXPECT_TRUE(test_axpy(3u)) << " TestAttributes::test_axpy failed";
    EXPECT_TRUE(test_reduce()) << " TestAttributes::test_reduce failed";
    EXPECT_TRUE(test_norm2()) << " TestAttributes::test_norm2 failed";
    EXPECT_TRUE(test_dot()) << " TestAttributes::test_dot failed";

    CUDA_ERROR(hipDeviceSynchronize());
}