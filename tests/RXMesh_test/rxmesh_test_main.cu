#include "hip/hip_runtime.h"
#include <vector>
#include "gtest/gtest.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/report.h"
#include "rxmesh/util/vector.h"

using dataT = float;

struct RXMeshTestArg
{
    uint32_t    num_run       = 1;
    uint32_t    device_id     = 0;
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    bool        quite         = false;
    int         argc          = argc;
    char**      argv          = argv;
} rxmesh_args;

// clang-format off
#include "test_higher_queries.h"
#include "test_queries.h"
#include "test_attribute.cuh"
#include "test_for_each.h"
#include "test_edge_flip.cuh"
#include "test_validate.h"
#include "test_deletion.cuh"
// clang-format on

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    rxmesh_args.argc = argc;
    rxmesh_args.argv = argv;
    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: RXMesh_test.exe < -option X>\n"
                        " -h:          Display this massage and exits\n"
                        " -input:      Input file. Input file should under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accepts OBJ files\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -num_run:    Number of iterations for performance testing. Default is {} \n"                        
                        " -q:          Run in quite mode. Default is false\n"
                        " -device_id:  GPU device ID. Default is {}",
            rxmesh_args.obj_file_name, rxmesh_args.output_folder ,rxmesh_args.num_run,rxmesh_args.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }


        if (cmd_option_exists(argv, argc + argv, "-num_run")) {
            rxmesh_args.num_run =
                atoi(get_cmd_option(argv, argv + argc, "-num_run"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            rxmesh_args.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            rxmesh_args.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            rxmesh_args.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
        if (cmd_option_exists(argv, argc + argv, "-q")) {
            rxmesh_args.quite = true;
        }
    }

    if (!rxmesh_args.quite) {
        RXMESH_TRACE("input= {}", rxmesh_args.obj_file_name);
        RXMESH_TRACE("output_folder= {}", rxmesh_args.output_folder);
        RXMESH_TRACE("num_run= {}", rxmesh_args.num_run);
        RXMESH_TRACE("device_id= {}", rxmesh_args.device_id);
    }
    return RUN_ALL_TESTS();
}
