#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/util.h"

#include "rxmesh/geometry_factory.h"
struct arg
{
    std::string obj_file_name    = STRINGIFY(INPUT_DIR) "cloth.obj";
    std::string output_folder    = STRINGIFY(OUTPUT_DIR);
    uint32_t    nx               = 66;
    uint32_t    ny               = 66;
    float       relative_len     = 1.0;
    int         num_smooth_iters = 5;
    uint32_t    num_iter         = 3;
    uint32_t    device_id        = 0;
    char**      argv;
    int         argc;
} Arg;

#include "remesh_rxmesh.cuh"

TEST(Apps, Remesh)
{
    using namespace rxmesh;

    // Select device
    cuda_query(Arg.device_id);

    // std::vector<std::vector<float>>    verts;
    // std::vector<std::vector<uint32_t>> fv;
    // create_plane(verts, fv, Arg.nx, Arg.ny);
    // RXMeshDynamic rx(fv);
    // rx.add_vertex_coordinates(verts, "Coords");


    RXMeshDynamic rx(Arg.obj_file_name, "", 256, 4.0, 2);
    // rx.save(STRINGIFY(OUTPUT_DIR) + extract_file_name(Arg.obj_file_name) +
    //         "_patches");

    // RXMeshDynamic rx(Arg.obj_file_name,
    //                  STRINGIFY(OUTPUT_DIR) +
    //                      extract_file_name(Arg.obj_file_name) + "_patches");
    //

    ASSERT_TRUE(rx.is_edge_manifold());

    // rx.export_obj("grid_" + std::to_string(Arg.nx) + "_" +
    //                   std::to_string(Arg.ny) + ".obj",
    //               *rx.get_input_vertex_coordinates());

    remesh_rxmesh(rx);
}


int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;


    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: Remesh.exe < -option X>\n"
                        " -h:              Display this massage and exit\n"
                        " -input:          Input obj file. Default is {} \n"
                        "                  Hint: Only accept OBJ files\n"
                        " -num_iter:       Number of remeshing iterations. Default is {}\n"
                        " -relative_len:   Target edge length as a ratio of the input mesh average edge length. Default is {}\n"
                        "                  Hint: should be slightly less than the average edge length of the input mesh\n"
                        " -o:              JSON file output folder. Default is {} \n"
                        " -device_id:      GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.num_iter,Arg.relative_len, Arg.output_folder, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
        if (cmd_option_exists(argv, argc + argv, "-num_iter")) {
            Arg.num_iter = atoi(get_cmd_option(argv, argv + argc, "-num_iter"));
        }
        if (cmd_option_exists(argv, argc + argv, "-relative_len")) {
            Arg.relative_len =
                std::stof(get_cmd_option(argv, argv + argc, "-relative_len"));
        }

        if (cmd_option_exists(argv, argc + argv, "-nx")) {
            Arg.nx = atoi(get_cmd_option(argv, argv + argc, "-nx"));
        }
        if (cmd_option_exists(argv, argc + argv, "-ny")) {
            Arg.ny = atoi(get_cmd_option(argv, argv + argc, "-ny"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("device_id= {}", Arg.device_id);
    RXMESH_TRACE("num_iter= {}", Arg.num_iter);
    RXMESH_TRACE("relative_len= {}", Arg.relative_len);
    RXMESH_TRACE("nx= {}", Arg.nx);
    RXMESH_TRACE("ny= {}", Arg.ny);


    return RUN_ALL_TESTS();
}