#include "gtest/gtest.h"
#include "rxmesh/attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/import_obj.h"

#include "nd_cross_patch_oedering.cuh"
#include "nd_reorder_kernel.cuh"

struct arg
{
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "sphere3.obj";
    uint32_t    device_id     = 0;
} Arg;

template <uint32_t blockThreads>
void cross_patch_ordering(rxmesh::RXMeshStatic&             rx,
                          rxmesh::VertexAttribute<uint16_t> v_ordering,
                          uint32_t smem_bytes_dyn)
{
    using namespace rxmesh;

    bool is_coarsen_flag = true;

    while (is_coarsen_flag) {
        bool     is_matching_flag = true;
        uint16_t level            = 0;

        RXMESH_TRACE("Matching");
        match_patches_init<blockThreads>
            <<<rx.get_num_patches(), blockThreads, smem_bytes_dyn>>>(rx.get_context(), level);
        CUDA_ERROR(hipDeviceSynchronize());

        while (is_matching_flag) {


            // match_patches_confirm<blockThreads><<<rx.get_num_patches(),
            // blockThreads>>>(
            //     rx.get_context(), level);
            // CUDA_ERROR(hipDeviceSynchronize());

            // update the is_matching_flag here
            is_matching_flag = false;
        }

        // match_patches_result_update_level<blockThreads><<<rx.get_num_patches(),
        // blockThreads>>>(
        //         rx.get_context(), level);
        //     CUDA_ERROR(hipDeviceSynchronize());

        // update level counter
        ++level;

        // update the is_coarsen_flag here
        is_coarsen_flag = false;
    }
}

void nd_reorder()
{
    using namespace rxmesh;
    constexpr uint32_t blockThreads = 256;

    RXMeshStatic rx(Arg.obj_file_name);

    // rx.save(STRINGIFY(OUTPUT_DIR) + extract_file_name(Arg.obj_file_name) +
    //         "_nd_patches");

    // RXMeshDynamic rx(Arg.obj_file_name,
    //                  STRINGIFY(OUTPUT_DIR) +
    //                      extract_file_name(Arg.obj_file_name) +
    //                      "_nd_patches");

    // vertex color attribute
    auto attr_matched_v =
        rx.add_vertex_attribute<uint16_t>("attr_matched_v", 1);
    auto attr_active_e = rx.add_edge_attribute<uint16_t>("attr_active_e", 1);

    uint16_t req_levels     = 5;
    uint32_t blocks         = rx.get_num_patches();
    uint32_t threads        = blockThreads;
    size_t   smem_bytes_dyn = 0;

    smem_bytes_dyn += (1 + 1 * req_levels) * rx.max_bitmask_size<LocalEdgeT>();
    smem_bytes_dyn +=
        (6 + 4 * req_levels) * rx.max_bitmask_size<LocalVertexT>();
    smem_bytes_dyn +=
        (4 + 5 * req_levels) * rx.get_per_patch_max_edges() * sizeof(uint16_t);
    smem_bytes_dyn += (1 + 4 * req_levels) * rx.get_per_patch_max_vertices() *
                      sizeof(uint16_t);
    smem_bytes_dyn +=
        (11 + 11 * req_levels) * ShmemAllocator::default_alignment;

    RXMESH_TRACE("blocks: {}, threads: {}, smem_bytes: {}",
                 blocks,
                 threads,
                 smem_bytes_dyn);

    // vertex ordering attribute to store the result
    auto v_reorder =
        rx.add_vertex_attribute<uint16_t>("v_reorder", 1, rxmesh::LOCATION_ALL);

    // Phase: cross patch reordering
    cross_patch_ordering<blockThreads>(rx, *v_reorder, smem_bytes_dyn);

    // Phase: single patch reordering
    // nd_single_patch_main<blockThreads><<<blocks, threads, smem_bytes_dyn>>>(
    //     rx.get_context(), *v_reorder, *attr_matched_v, *attr_active_e,
    //     req_levels);

    CUDA_ERROR(hipDeviceSynchronize());

#if USE_POLYSCOPE
    // Tests using coloring
    // Move vertex color to the host
    attr_matched_v->move(rxmesh::DEVICE, rxmesh::HOST);
    attr_active_e->move(rxmesh::DEVICE, rxmesh::HOST);

    // polyscope instance associated with rx
    auto polyscope_mesh = rx.get_polyscope_mesh();

    // pass vertex color to polyscope
    polyscope_mesh->addVertexScalarQuantity("attr_matched_v", *attr_matched_v);
    polyscope_mesh->addEdgeScalarQuantity("attr_active_e", *attr_active_e);

    // render
    polyscope::show();
#endif

    RXMESH_TRACE("DONE!!!!!!!!!!!!!!");
}

TEST(Apps, NDReorder)
{
    using namespace rxmesh;

    // Select device
    cuda_query(Arg.device_id);

    // nd reorder implementation
    nd_reorder();
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);

    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: NDReorder.exe < -option X>\n"
                        " -h:          Display this massage and exits\n"
                        " -input:      Input file. Input file should under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accepts OBJ files\n"                                              
                        " -device_id:  GPU device ID. Default is {}",
            Arg.obj_file_name,  Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }

        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}


// batch info file