#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/util.h"

struct arg
{
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "Fennec_Fox.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    uint32_t    target        = 0;
    uint32_t    device_id     = 0;
    char**      argv;
    int         argc;
} Arg;

#include "simplification_rxmesh.cuh"

TEST(Apps, Simplification)
{
    using namespace rxmesh;

    // Select device
    cuda_query(Arg.device_id);

    RXMeshDynamic rx(Arg.obj_file_name, "", true);
    // rx.save(STRINGIFY(OUTPUT_DIR) + extract_file_name(Arg.obj_file_name) +
    //        "_patches");

    // RXMeshDynamic rx(Arg.obj_file_name,
    //                  STRINGIFY(OUTPUT_DIR) +
    //                      extract_file_name(Arg.obj_file_name) + "_patches",
    //                  true);

    ASSERT_TRUE(rx.is_edge_manifold());

    simplification_rxmesh(rx, Arg.target);
}


int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;


    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: Simplification.exe < -option X>\n"
                        " -h:          Display this massage and exit\n"
                        " -input:      Input file. Input file should be under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accept OBJ files\n"
                        " -target:     The final/target number of faces in the output mesh\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -device_id:  GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
        if (cmd_option_exists(argv, argc + argv, "-target")) {
            Arg.target = false;
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("device_id= {}", Arg.device_id);
    RXMESH_TRACE("target= {}", Arg.target);

    return RUN_ALL_TESTS();
}