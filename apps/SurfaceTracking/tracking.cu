#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/macros.h"
#include "rxmesh/util/util.h"

struct arg
{
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id     = 0;
    char**      argv;
    int         argc;
} Arg;

#include "tracking_rxmesh.cuh"

TEST(Apps, SurfaceTracking)
{
    using namespace rxmesh;

    // Select device
    cuda_query(Arg.device_id);

    RXMeshDynamic rx("");
    // rx.save(STRINGIFY(OUTPUT_DIR) + extract_file_name(Arg.obj_file_name) +
    //         "_patches");

    // RXMeshDynamic rx(Arg.obj_file_name,
    //                  STRINGIFY(OUTPUT_DIR) +
    //                      extract_file_name(Arg.obj_file_name) + "_patches");

    tracking_rxmesh(rx);
}


int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;


    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: ShortestEdgeCollapse.exe < -option X>\n"
                        " -h:          Display this massage and exit\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -device_id:  GPU device ID. Default is {}",
            Arg.output_folder, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }
                
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}