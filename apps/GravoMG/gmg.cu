#include "hip/hip_runtime.h"

#include "include/NeighborHandling.h"


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "include/GMGProlongatorCreation.h"
std::vector<int> intPointerArrayToVector(int* array, size_t size)
{
    return std::vector<int>(array, array + size);
}


void CreateNextLevelData(int      N,
                         int      numberOfSamples,
                         VertexData* vData_old,
                         VertexData* vData_new)
{

    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
        [=] __device__(int number) { vData_new[number].distance = 0;
                     });
}




void numberOfNeighbors(int              numberOfSamples,
                       VertexNeighbors* neighbors ,
    int              N,
                       CSR              csr,
                        VertexData* vData, int* number_of_neighbors

)
{
    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    int* neighborList;
    hipMallocManaged(&neighborList, sizeof(int) * numberOfSamples);

    for (int i = 0; i < numberOfSamples; i++)
        neighborList[i] = 0;
    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         //int currentCluster = vertexClusters[number];
                         int currentCluster = vData[number].cluster;

                         //neighbors[currentCluster].getNeighbors(neighborList);
                         for (int i = csr.row_ptr[number]; i < csr.row_ptr[number+1];i++) {
                             int currentNode = csr.value_ptr[i];
                             if (vData[currentNode].cluster != currentCluster) {
                                 //neighbors
                                 neighbors[currentCluster].addNeighbor(vData[currentNode].cluster);
                             }
                         }

                         // std::cout <<
                         // neighbors[0].getNeighbors().size();//this is the
                         // neighbor count
                     });


     thrust::device_vector<int> samples2(numberOfSamples);
    thrust::sequence(samples2.begin(), samples2.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {

                        number_of_neighbors[number] =
                             neighbors[number].getNumberOfNeighbors();

                     });


}




void setCluster(int    n,
                float* distance,
    int currentLevel,
    VertexData* vertex_data)
{
    thrust::device_vector<int> samples(n);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         // take bitmask
                         // if sample, the cluster is its own
                         // if not a sample, set cluster as -1
                         // set distance as infinity or 0 based on whether it is
                         // not or is a sample

                         if ((vertex_data[number].bitmask & (1 << currentLevel-1)) != 0) 
                         {
                            distance[number]        = 0;
                             vertex_data[number].cluster =
                                 vertex_data[number].sample_number;
                            
                             printf(
                                "\n%d which is sample %d is now a cluster vertex",
                                number,
                                vertex_data[number].sample_number);
                                
                         }
                         else 
                         {
                             vertex_data[number].cluster =-1;
                             /*
                             printf(
                                 "\n%d which is sample %d is not a cluster "
                                 "vertex",
                                 number,
                                 vertex_data[number].sample_number);
                                 */
                             distance[number]        = INFINITY;
                         }
        });
}







int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    auto vertex_pos = *rx.get_input_vertex_coordinates();

    //attribute to sample,store and order samples
    auto sample_number = *rx.add_vertex_attribute<int>("sample_number", 1);
    auto distance      = *rx.add_vertex_attribute<float>("distance", 1);


    auto sample_level_bitmask = *rx.add_vertex_attribute<
        uint16_t>("bitmask", 1);
    auto clustered_vertex = *rx.add_vertex_attribute<int>("clustering", 1);

    auto number_of_neighbors_coarse = *rx.add_vertex_attribute<int>(
        "number of neighbors",
        1);


    int* flagger;
    hipMallocManaged(&flagger, sizeof(int));
    *flagger = 0;

    auto context = rx.get_context();

    constexpr uint32_t               CUDABlockSize = 512;
    rxmesh::LaunchBox<CUDABlockSize> lb;
    rx.prepare_launch_box({rxmesh::Op::VV},
                          lb,
                          (void*)sample_points<float, CUDABlockSize>);


    float ratio           = 8;
    int   N               = rx.get_num_vertices();
    int   numberOfLevels  = 4;
    int   currentLevel    = 1; // first coarse mesh
    int   numberOfSamplesForFirstLevel = N / powf(ratio, 1);//start
    int   numberOfSamples = N / powf(ratio, currentLevel);//start


    std::random_device rd;
    // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_int_distribution<> dist(0, N - 1);
    // From 0 to (number of points - 1)
    int seed = dist(gen);

    std::cout << "\nSeed: " << seed;

    VertexReduceHandle<float>              reducer(distance);
    hipcub::KeyValuePair<VertexHandle, float> farthestPoint;

    Vec3* vertices;
    Vec3* sample_pos;
    float*    distanceArray;
    int*    clusterVertices;

    
    // Allocate unified memory
    hipMallocManaged(&sample_pos, numberOfSamples * sizeof(Vec3));
    hipMallocManaged(&vertices, N * sizeof(Vec3));
    hipMallocManaged(&distanceArray, N * sizeof(int));
    hipMallocManaged(&clusterVertices, N * sizeof(int));

    hipDeviceSynchronize();


    // pre processing step
    //gathers samples for every level
    int j = 0;
    int currentSampleLevel = numberOfLevels;
    std::cout << "levels:";

    for (int q=0;q<numberOfLevels;q++) {
        std::cout << "\n  level " << q << " : " << N / powf(ratio, q);
    }
    for (int i = 0; i < numberOfSamplesForFirstLevel; i++) {
        if (i == N / (int)powf(ratio,  currentSampleLevel)) {
            currentSampleLevel--;
            std::cout << "\nNext sample level: " << currentSampleLevel;
        }

        rx.for_each_vertex(rxmesh::DEVICE,
                           [seed,
                               context,
                               sample_number,
                               sample_level_bitmask,
                               distance,
                               i,
                               currentSampleLevel,
             sample_pos,
                               vertex_pos] __device__(
                           const rxmesh::VertexHandle vh) {
                               if (seed == context.linear_id(vh)) {
                                   sample_number(vh, 0) = i;
                                   //sample_number_point
                                   distance(vh, 0)      = 0;
                                   sample_pos[i].x        = vertex_pos(vh, 0);
                                   sample_pos[i].y      = vertex_pos(vh, 1);
                                   sample_pos[i].z        = vertex_pos(vh, 2);

                                   for (int k = 0; k < currentSampleLevel;
                                        k++) {
                                       sample_level_bitmask(vh, 0) |= (1 << k);
                                   }

                                   /* printf(
                                       "\n\n Sample %d, vertex %d is level %d",
                                          i, 
                                          seed,
                                          currentSampleLevel);
                                          */
                               } else {
                                   if (i == 0) {
                                       distance(vh, 0)      = INFINITY;
                                       sample_number(vh, 0) = -1;
                                   }
                               }
                           });

        do {
            hipDeviceSynchronize();
            *flagger = 0;
            sample_points<float, CUDABlockSize>
                <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                    rx.get_context(),
                    vertex_pos,
                    distance,
                    flagger);
            hipDeviceSynchronize();
            //std::cout << "\nflag: "<<*flagger
            //          << "\n\niteration: " << j << std::endl;

            j++;

        } while (*flagger != 0);


        // reduction step
        farthestPoint = reducer.arg_max(distance, 0);
        seed          = rx.linear_id(farthestPoint.key);
    }

    std::cout << "\nSampling iterations: " << j; 


    sample_number.move(DEVICE, HOST);
    distance.move(DEVICE, HOST);
    sample_level_bitmask.move(DEVICE, HOST);
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///
    /// first level

    rxmesh::LaunchBox<CUDABlockSize> cb;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        cb,
        (void*)cluster_points<float, CUDABlockSize>);



    //clustering step
    j = 0;
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [
            sample_number,
            sample_level_bitmask,
            distance,
            currentLevel,
            clustered_vertex, context, vertices, vertex_pos ]
        __device__(const rxmesh::VertexHandle vh) {

                vertices[context.linear_id(vh)].x = vertex_pos(vh, 0);
                vertices[context.linear_id(vh)].y = vertex_pos(vh, 1);
                vertices[context.linear_id(vh)].z = vertex_pos(vh, 2);

            //if (sample_number(vh, 0) > -1)
            if ((sample_level_bitmask(vh,0) & (1 << (currentLevel-1))) !=0)
            {
                clustered_vertex(vh, 0) = sample_number(vh, 0);
                distance(vh, 0)         = 0;
            } else {
                distance(vh, 0)         = INFINITY;
                clustered_vertex(vh, 0) = -1;
            }
        });

    do {
        hipDeviceSynchronize();
        *flagger = 0;
        cluster_points<float, CUDABlockSize>
            <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                rx.get_context(),
                vertex_pos,
                distance,
                clustered_vertex,
                flagger);
        hipDeviceSynchronize();
        j++;
    } while (*flagger != 0);

    clustered_vertex.move(DEVICE, HOST);
    std::cout << "\Clustering iterations: " << j;

    int* vertexCluster;
    hipMallocManaged(&vertexCluster, sizeof(int) * N);
    rx.for_each_vertex(rxmesh::DEVICE,
                       [
                           clustered_vertex,
                           context,vertexCluster] __device__(
                       const rxmesh::VertexHandle vh) {
                           vertexCluster[context.linear_id(vh)] =
                               clustered_vertex(vh, 0);


                       });
    hipDeviceSynchronize();

    int* number_of_neighbors;
    hipMallocManaged(&number_of_neighbors, numberOfSamples * sizeof(int));
    for (int i = 0; i < numberOfSamples; i++) {
        number_of_neighbors[i] = 0;
    }
    hipDeviceSynchronize();

    rxmesh::LaunchBox<CUDABlockSize> nn;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        nn,
        (void*)findNumberOfCoarseNeighbors<float, CUDABlockSize>);


    //find number of neighbors without the bit matrix


    // Allocate memory for vertex neighbors
    VertexNeighbors* vertexNeighbors;
    hipError_t      err = hipMallocManaged(&vertexNeighbors,
                                             numberOfSamples * sizeof(
                                                 VertexNeighbors));

    findNumberOfCoarseNeighbors<float, CUDABlockSize>
        <<<nn.blocks, nn.num_threads, nn.smem_bytes_dyn>>>(
            rx.get_context(),
            clustered_vertex,
            number_of_neighbors,
            vertexNeighbors);
    hipDeviceSynchronize();


    rx.for_each_vertex(rxmesh::DEVICE,
                       [numberOfSamples,
                           context,
                           vertexNeighbors,
                           clustered_vertex,
                           sample_number,
                           number_of_neighbors] __device__(
                       const rxmesh::VertexHandle vh) {

                           if (clustered_vertex(vh, 0) ==
                               sample_number(vh, 0)) {
                               number_of_neighbors[clustered_vertex(vh, 0)] =
                                   vertexNeighbors[sample_number(vh, 0)].
                                   getNumberOfNeighbors();


                               /* printf("\n vertex %d : %d neighbors",
                       sample_number(vh, 0),
                       vertexNeighbors[sample_number(vh, 0)].getNumberOfNeighbors());
                       */
                           }
                       });
    hipDeviceSynchronize();

    int num_rows = numberOfSamples; // Set this appropriately
    CSR csr(num_rows, number_of_neighbors, vertexNeighbors, N);

    //csr.printCSR();

    hipDeviceSynchronize(); // Ensure data is synchronized before accessing

   
    //for debug purposes
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
            clustered_vertex,
            number_of_neighbors,
            number_of_neighbors_coarse,
            context] __device__(const rxmesh::VertexHandle vh) {
            number_of_neighbors_coarse(vh, 0) = number_of_neighbors[sample_number(vh, 0)];

        });


    float* prolongation_operator;
    hipMallocManaged(&prolongation_operator,
                      N * numberOfSamples * sizeof(float));
    hipDeviceSynchronize();
   createProlongationOperator(csr.num_rows,
                                csr.row_ptr,
                                csr.value_ptr,
                                csr.number_of_neighbors,
                                N,
                                vertexCluster,
                                vertices, sample_pos,
                                prolongation_operator);
                                
    hipDeviceSynchronize();
   /*
    for (int i=0;i<N;i++) {
        std::cout << "\n" << i << " ";
        for (int k=0;k<numberOfSamples;k++) {
            std::cout << prolongation_operator[i * numberOfSamples + k] << " ";
        }
    }
    */

    Eigen::MatrixXd verts;
    Eigen::MatrixXi faces;
    std::vector<std::array<double, 3>>
        vertexPositions;                            // To store vertex positions
        std::vector<std::vector<size_t>> faceIndices;  // To store face indices
 
    csr.GetRenderData(vertexPositions, faceIndices, sample_pos);

    polyscope::registerSurfaceMesh("mesh level 1", vertexPositions, faceIndices);


    //set 1st level node data
    VertexData* oldVdata;
    hipMallocManaged(&oldVdata, sizeof(VertexData) * numberOfSamples);

    rx.for_each_vertex(rxmesh::DEVICE,
                       [sample_number,
                        oldVdata,
                        clustered_vertex,
                        vertex_pos,
         sample_level_bitmask,
                        context] __device__(const rxmesh::VertexHandle vh) {


            if (sample_number(vh, 0) != -1) {
                //printf("\nputting data for sample %d", sample_number(vh, 0));

                oldVdata[sample_number(vh, 0)].distance = 0;
                oldVdata[sample_number(vh, 0)].linear_id =
                    context.linear_id(vh);
                oldVdata[sample_number(vh, 0)].sample_number =
                    sample_number(vh, 0);
                oldVdata[sample_number(vh, 0)].bitmask =
                    sample_level_bitmask(vh, 0);
                oldVdata[sample_number(vh, 0)].position.x = vertex_pos(vh, 0);
                oldVdata[sample_number(vh, 0)].position.y = vertex_pos(vh, 1);
                oldVdata[sample_number(vh, 0)].position.z = vertex_pos(vh, 2);
                oldVdata[sample_number(vh, 0)].cluster =
                    clustered_vertex(vh, 0);
            }

                       });
    

    ////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///next levels




    CSR lastCSR                 = csr;
    CSR currentCSR              = csr;
    int currentNumberOfVertices = numberOfSamples;
    int currentNumberOfSamples  = numberOfSamples / ratio;
    std::vector<float*> prolongationOperators;
    prolongationOperators.resize(numberOfLevels-1);

    prolongationOperators[0] = prolongation_operator;

    std::vector<Eigen::MatrixXd> vertsArray;
    std::vector<Eigen::MatrixXi> facesArray;
    std::vector<std::vector<std::array<double, 3>>>
        vertexPositionsArray;  // To store vertex positions
    std::vector<std::vector<std::vector<size_t>>>
        faceIndicesArray;  // To store face indices

    std::vector<std::vector<int>> clustering;



    vertsArray.resize(numberOfLevels);
    facesArray.resize(numberOfLevels);
    vertexPositionsArray.resize(numberOfLevels);
    faceIndicesArray.resize(numberOfLevels);
    clustering.resize(numberOfLevels);

    for (int level = 1; level < numberOfLevels - 1; level++) {

        std::cout << "\nlevel : " << level;
        std::cout << "\n current number of samples: " << currentNumberOfSamples;
        std::cout << "\n current number of vertices: "
                  << currentNumberOfVertices;
        setCluster(currentNumberOfVertices, distanceArray, level + 1, oldVdata);

        do {
            *flagger = 0;
            clusterCSR(currentNumberOfVertices,
                       sample_pos,
                       distanceArray,
                       vertexCluster,
                       flagger,
                       lastCSR,
                       oldVdata);
            hipDeviceSynchronize();
        } while (*flagger != 0);

        //clustering[level - 1].resize(currentNumberOfVertices);
        //clustering[level-1]=intPointerArrayToVector(vertexCluster, currentNumberOfVertices);


        //polyscope::getSurfaceMesh("mesh level " + std::to_string(level))
        //->addVertexScalarQuantity("clustered vertices", clustering[level-1]);


        VertexNeighbors* vertexNeighbors2;
        err = hipMallocManaged(
            &vertexNeighbors2,
            currentNumberOfVertices * sizeof(VertexNeighbors));

        int* number_of_neighbors2;
        hipMallocManaged(&number_of_neighbors2,
                          currentNumberOfVertices * sizeof(int));


        numberOfNeighbors(currentNumberOfSamples,
                          vertexNeighbors2,
                          currentNumberOfVertices,
                          lastCSR,
                          oldVdata,
                          number_of_neighbors2);


        hipDeviceSynchronize();
        currentCSR = CSR(currentNumberOfSamples,
                         number_of_neighbors2,
                         vertexNeighbors2,
                         currentNumberOfVertices);

        // currentCSR.printCSR();

        currentCSR.GetRenderData(vertexPositionsArray[level - 1],
                                 faceIndicesArray[level - 1],
                                 sample_pos);


        polyscope::registerSurfaceMesh(
            "mesh level " + std::to_string(level + 1),
            vertexPositionsArray[level - 1],
            faceIndicesArray[level - 1]);



        float* prolongationOperator2;
        hipMallocManaged(&prolongationOperator2,
                          sizeof(float) * currentNumberOfSamples * currentNumberOfVertices);
        hipDeviceSynchronize();
        prolongationOperators[level] = prolongationOperator2;
        hipDeviceSynchronize();

        createProlongationOperator(currentNumberOfSamples,
                                   currentCSR.row_ptr,
                                   currentCSR.value_ptr,
                                   number_of_neighbors2,
                                   currentNumberOfVertices,
                                   oldVdata,
                                   prolongationOperator2);

        hipDeviceSynchronize();  // Ensure data is synchronized before
                                  // accessing


        currentNumberOfSamples /= 8;
        currentNumberOfVertices /= 8;
        lastCSR = currentCSR;

    

    }

    hipDeviceSynchronize();
    for (int q = 2; q < numberOfLevels - 1; q++) {
        std::cout << "\n\n Level: " << q << " to " << q + 1;
        int rows = N / static_cast<int>(std::round(powf(ratio, q)));
        int cols =
            numberOfSamples / static_cast<int>(std::round(powf(ratio, q)));
        std::cout << " Rows: " << rows;
        std::cout << " Cols: " << cols;

        for (int l = 0; l < rows; l++) {
            std::cout << "\n" << l << ": ";
            for (int m = 0; m < cols; m++) {
                int index = l * cols + m;
                auto a     = prolongationOperators[q][index];
                std::cout << prolongationOperators[q][index] << " ";
            }
        }
    }

   
    /*
    setCluster(numberOfSamples, distanceArray, currentLevel+1, oldVdata);

    do {
        *flagger = 0;
        clusterCSR(numberOfSamples,
                   sample_pos,
                   distanceArray,
                   vertexCluster,
                   flagger,
                   csr,oldVdata);
        hipDeviceSynchronize();
    } while (*flagger != 0);

    std::vector<int> a = intPointerArrayToVector(vertexCluster,numberOfSamples);

    polyscope::getSurfaceMesh("mesh level 1")
        ->addVertexScalarQuantity("cluster Vertex", a);

    //neighbor stuff
    
    VertexNeighbors* vertexNeighbors2;
    err = hipMallocManaged(&vertexNeighbors2,
                            numberOfSamples * sizeof(VertexNeighbors));

    int* number_of_neighbors2;
    hipMallocManaged(&number_of_neighbors2,
                      numberOfSamples * sizeof(int));


    numberOfNeighbors(numberOfSamples / 8,
                      vertexNeighbors2,
                      numberOfSamples,
                      csr,
                      oldVdata,number_of_neighbors2);


        hipDeviceSynchronize();
    num_rows = numberOfSamples / 8;  // Set this appropriately
    CSR csr2(num_rows,
             number_of_neighbors2,
             vertexNeighbors2,
             numberOfSamples);

    csr2.printCSR();


    float* prolongationOperator2;
    hipMallocManaged(&prolongationOperator2,
                      sizeof(float) * numberOfSamples * num_rows);


    createProlongationOperator(num_rows,
                               csr2.row_ptr,
                               csr2.value_ptr,
                               number_of_neighbors2,
                               numberOfSamples,
                               oldVdata,
                               prolongationOperator2);

    hipDeviceSynchronize();  // Ensure data is synchronized before accessing


     Eigen::MatrixXd verts2;
    Eigen::MatrixXi faces2;
    std::vector<std::array<double, 3>>
        vertexPositions2;                           // To store vertex positions
    std::vector<std::vector<size_t>> faceIndices2;  // To store face indices

    csr2.GetRenderData(vertexPositions2, faceIndices2, sample_pos);

    polyscope::registerSurfaceMesh(
        "mesh level 2", vertexPositions2, faceIndices2);
        
        */


    //////////////////////////////////////////////////////////////////

    /*
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
         clustered_vertex,
         number_of_neighbors,
         row_ptr,value_ptr,
         context,
        vertices,vertex_pos, prolongation_operator,numberOfSamples] __device__(const rxmesh::VertexHandle vh) {

        //go through every triangle of my cluster
        const int cluster_point = clustered_vertex(vh, 0);
        const int start_pointer = row_ptr[clustered_vertex(vh,0)];
        const int end_pointer = row_ptr[clustered_vertex(vh,0)+1];

        float min_distance = 99999;
        Eigen::Vector3<float> selectedv1{0,0,0}, selectedv2{0, 0, 0},
            selectedv3{0, 0, 0};
        const Eigen::Vector3<float> q{
            vertex_pos(vh, 0), vertex_pos(vh, 1), vertex_pos(vh, 2)};

        int neighbor=0;
        int selected_neighbor=0;
        int neighbor_of_neighbor=0;
        int selected_neighbor_of_neighbor=0;


        for (int i=start_pointer;i<end_pointer;i++) {

            float distance;
             // get the neighbor vertex
            neighbor = value_ptr[i];  // assuming col_idx stores column
                                        // indices of neighbors in csr.

            // get the range of neighbors for this neighbor
            const int neighbor_start = row_ptr[neighbor];
            const int neighbor_end   = row_ptr[neighbor + 1];

            for (int j = neighbor_start; j < neighbor_end; j++) {
                neighbor_of_neighbor = value_ptr[j];

                for (int k=i+1;k<end_pointer;k++)
                {
                    if (value_ptr[k]==neighbor_of_neighbor) 
                    { 


                        
                        Eigen::Vector3<float> v1{vertices[cluster_point].x,
                                                 vertices[cluster_point].y,
                                                 vertices[cluster_point].z};
                        Eigen::Vector3<float> v2{vertices[neighbor].x,
                                                 vertices[neighbor].y,
                                                 vertices[neighbor].z};
                        Eigen::Vector3<float> v3{
                            vertices[neighbor_of_neighbor].x,
                            vertices[neighbor_of_neighbor].y,
                            vertices[neighbor_of_neighbor].z};

                        //find distance , if less than min dist, find bary coords, save them
                        float distance = projectedDistance(v1, v2, v3, q);
                        if (distance<min_distance) {
                            
                            min_distance = distance;
                            selectedv1   = v1;
                            selectedv2   = v2;
                            selectedv3   = v3;
                            selected_neighbor = neighbor;
                            selected_neighbor_of_neighbor =neighbor_of_neighbor;
                        }
                    }
                }
            }
        }
        // take the best bary coords
        auto [b1, b2, b3] = computeBarycentricCoordinates(
            selectedv1, selectedv2, selectedv3, q);
        // put it inside prolongation row, it will be unique so no race
        // condition
        int l = context.linear_id(vh);

        printf("\n %d final coords: %f %f %f", l, b1, b2, b3);


        //prolongation_operator[l * numberOfSamples + cluster_point]        = b1;
        //prolongation_operator[l * numberOfSamples + selected_neighbor] = b2;
        //prolongation_operator[l * numberOfSamples + selected_neighbor_of_neighbor]              = b3;

        
        prolongation_operator[l * numberOfSamples + cluster_point] =
            cluster_point;
        prolongation_operator[l * numberOfSamples + selected_neighbor] =
            selected_neighbor;
        prolongation_operator[l * numberOfSamples +
                              selected_neighbor_of_neighbor] =
            selected_neighbor_of_neighbor;

        //printf("\n%d at %d", l, l * numberOfSamples);



    });
    */


    std::cout << std::endl;
    std::cout << std::endl;

    hipDeviceSynchronize();




    hipFree(vertices);
    hipMallocManaged(&vertices, sizeof(Vec3) * csr.num_rows);

    float* distances;
    hipMallocManaged(&distances, sizeof(float) * csr.num_rows);


    number_of_neighbors_coarse.move(DEVICE, HOST);

    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_number",
        sample_number);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("distance", distance);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_level_bitmask",
        sample_level_bitmask);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "clusterPoint",
        clustered_vertex);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "number of neighbors",
        number_of_neighbors_coarse);


#if USE_POLYSCOPE
    polyscope::show();
#endif
}

