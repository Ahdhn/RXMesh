#include "hip/hip_runtime.h"

#include "include/NeighborHandling.h"


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "include/GMGProlongatorCreation.h"



void numberOfNeighbors(int              numberOfSamples,
                       VertexNeighbors* neighbors,
                       int*             vertexClusters,
                       int              N,
                       CSR              csr

)
{
    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    int* neighborList;
    hipMallocManaged(&neighborList, sizeof(int) * numberOfSamples);

    for (int i = 0; i < numberOfSamples; i++)
        neighborList[i] = 0;
    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         int currentCluster = vertexClusters[number];


                         //neighbors[currentCluster].getNeighbors(neighborList);
                         for (int i = csr.row_ptr[number]; i < csr.row_ptr[number+1];i++) {
                             int currentNode = csr.value_ptr[i];
                             if (vertexClusters[currentNode]!=currentCluster) {
                                 //neighbors
                                 neighbors[currentCluster].addNeighbor(
                                     currentNode);
                             }
                         }

                         // std::cout <<
                         // neighbors[0].getNeighbors().size();//this is the
                         // neighbor count
                     });
}




void setCluster(int    n,
                float* distance,
    int* clusterVertices,            
    uint8_t* bitmask,
    int currentLevel)
{
    thrust::device_vector<int> samples(n);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         // take bitmask
                         // if sample, the cluster is its own
                         // if not a sample, set cluster as -1
                         // set distance as infinity or 0 based on whether it is
                         // not or is a sample

                         if ((bitmask[number] & (1 << currentLevel-1)) != 0) 
                         {
                            printf( "\nvertex %d is being used as a cluster point",number);
                             clusterVertices[number] = clusterVertices[number]; //dont change from previous level
                            distance[number]        = 0;

                         }
                         else 
                         {
                             clusterVertices[number] = -1;
                             distance[number]        = INFINITY;
                         }
        });
}





int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere.obj");

    auto vertex_pos = *rx.get_input_vertex_coordinates();

    //attribute to sample,store and order samples
    auto sample_number = *rx.add_vertex_attribute<int>("sample_number", 1);
    auto distance      = *rx.add_vertex_attribute<float>("distance", 1);


    auto sample_level_bitmask = *rx.add_vertex_attribute<
        uint16_t>("bitmask", 1);
    auto clustered_vertex = *rx.add_vertex_attribute<int>("clustering", 1);

    auto number_of_neighbors_coarse = *rx.add_vertex_attribute<int>(
        "number of neighbors",
        1);


    int* flagger;
    hipMallocManaged(&flagger, sizeof(int));
    *flagger = 0;

    auto context = rx.get_context();

    constexpr uint32_t               CUDABlockSize = 512;
    rxmesh::LaunchBox<CUDABlockSize> lb;
    rx.prepare_launch_box({rxmesh::Op::VV},
                          lb,
                          (void*)sample_points<float, CUDABlockSize>);


    float ratio           = 8;
    int   N               = rx.get_num_vertices();
    int   numberOfLevels  = 3;
    int   currentLevel    = 1; // first coarse mesh
    int   numberOfSamplesForFirstLevel = N / powf(ratio, 1);//start
    int   numberOfSamples = N / powf(ratio, currentLevel);//start


    std::random_device rd;
    // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_int_distribution<> dist(0, N - 1);
    // From 0 to (number of points - 1)
    int seed = dist(gen);

    std::cout << "\nSeed: " << seed;

    VertexReduceHandle<float>              reducer(distance);
    hipcub::KeyValuePair<VertexHandle, float> farthestPoint;

    Vec3* vertices;
    Vec3* sample_pos;
    uint8_t* bitmask;
    float*    distanceArray;
    int*    clusterVertices;
    int*      sample_number_pointer;

    
    // Allocate unified memory
    hipMallocManaged(&sample_pos, numberOfSamples * sizeof(Vec3));
    hipMallocManaged(&vertices, N * sizeof(Vec3));
    hipMallocManaged(&bitmask, N * sizeof(int));
    hipMallocManaged(&distanceArray, N * sizeof(int));
    hipMallocManaged(&clusterVertices, N * sizeof(int));
    hipMallocManaged(&sample_number_pointer, sizeof(int) * numberOfSamples);

    hipDeviceSynchronize();

    for (int i=0;i<N;i++) {
        bitmask[i] = 0;
    }


    // pre processing step
    //gathers samples for every level
    int j = 0;
    int currentSampleLevel = numberOfLevels;
    std::cout << "levels:";

    for (int q=0;q<numberOfLevels;q++) {
        std::cout << "\n  level " << q << " : " << N / powf(ratio, q);
    }
    for (int i = 0; i < numberOfSamplesForFirstLevel; i++) {
        if (i == N / (int)powf(ratio,  currentSampleLevel)) {
            currentSampleLevel--;
            std::cout << "\nNext sample level: " << currentSampleLevel;
        }

        rx.for_each_vertex(rxmesh::DEVICE,
                           [seed,
                               context,
                               sample_number,
                               sample_level_bitmask,
                               bitmask,
                               distance,
                               i,
                               currentSampleLevel,
             sample_pos,
                               vertex_pos, sample_number_pointer] __device__(
                           const rxmesh::VertexHandle vh) {
                               if (seed == context.linear_id(vh)) {
                                   sample_number(vh, 0) = i;
                                   //sample_number_point
                                   distance(vh, 0)      = 0;
                                   sample_pos[i].x        = vertex_pos(vh, 0);
                                   sample_pos[i].y      = vertex_pos(vh, 1);
                                   sample_pos[i].z        = vertex_pos(vh, 2);

                                   for (int k = 0; k < currentSampleLevel;
                                        k++) {
                                       sample_level_bitmask(vh, 0) |= (1 << k);
                                       bitmask[seed] |= (1 << k);
                                   }
                               } else {
                                   if (i == 0) {
                                       distance(vh, 0)      = INFINITY;
                                       sample_number(vh, 0) = -1;
                                   }
                               }
                           });

        do {
            hipDeviceSynchronize();
            *flagger = 0;
            sample_points<float, CUDABlockSize>
                <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                    rx.get_context(),
                    vertex_pos,
                    distance,
                    flagger);
            hipDeviceSynchronize();
            //std::cout << "\nflag: "<<*flagger
            //          << "\n\niteration: " << j << std::endl;

            j++;

        } while (*flagger != 0);


        // reduction step
        farthestPoint = reducer.arg_max(distance, 0);
        seed          = rx.linear_id(farthestPoint.key);
    }

    std::cout << "\nSampling iterations: " << j; 


    sample_number.move(DEVICE, HOST);
    distance.move(DEVICE, HOST);
    sample_level_bitmask.move(DEVICE, HOST);
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///
    /// first level

    rxmesh::LaunchBox<CUDABlockSize> cb;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        cb,
        (void*)cluster_points<float, CUDABlockSize>);



    //clustering step
    j = 0;
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [
            sample_number,
            sample_level_bitmask,
            distance,
            currentLevel,
            clustered_vertex, context, vertices, vertex_pos ]
        __device__(const rxmesh::VertexHandle vh) {

                vertices[context.linear_id(vh)].x = vertex_pos(vh, 0);
                vertices[context.linear_id(vh)].y = vertex_pos(vh, 1);
                vertices[context.linear_id(vh)].z = vertex_pos(vh, 2);

            //if (sample_number(vh, 0) > -1)
            if ((sample_level_bitmask(vh,0) & (1 << (currentLevel-1))) !=0)
            {
                clustered_vertex(vh, 0) = sample_number(vh, 0);
                distance(vh, 0)         = 0;
            } else {
                distance(vh, 0)         = INFINITY;
                clustered_vertex(vh, 0) = -1;
            }
        });

    do {
        hipDeviceSynchronize();
        *flagger = 0;
        cluster_points<float, CUDABlockSize>
            <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                rx.get_context(),
                vertex_pos,
                distance,
                clustered_vertex,
                flagger);
        hipDeviceSynchronize();
        j++;
    } while (*flagger != 0);

    clustered_vertex.move(DEVICE, HOST);
    std::cout << "\Clustering iterations: " << j;

    int* vertexCluster;
    hipMallocManaged(&vertexCluster, sizeof(int) * N);
    rx.for_each_vertex(rxmesh::DEVICE,
                       [
                           clustered_vertex,
                           context,vertexCluster] __device__(
                       const rxmesh::VertexHandle vh) {
                           vertexCluster[context.linear_id(vh)] =
                               clustered_vertex(vh, 0);


                       });
    hipDeviceSynchronize();

    int* number_of_neighbors;
    hipMallocManaged(&number_of_neighbors, numberOfSamples * sizeof(int));
    for (int i = 0; i < numberOfSamples; i++) {
        number_of_neighbors[i] = 0;
    }
    hipDeviceSynchronize();

    rxmesh::LaunchBox<CUDABlockSize> nn;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        nn,
        (void*)findNumberOfCoarseNeighbors<float, CUDABlockSize>);


    //find number of neighbors without the bit matrix


    // Allocate memory for vertex neighbors
    VertexNeighbors* vertexNeighbors;
    hipError_t      err = hipMallocManaged(&vertexNeighbors,
                                             numberOfSamples * sizeof(
                                                 VertexNeighbors));

    findNumberOfCoarseNeighbors<float, CUDABlockSize>
        <<<nn.blocks, nn.num_threads, nn.smem_bytes_dyn>>>(
            rx.get_context(),
            clustered_vertex,
            number_of_neighbors,
            vertexNeighbors);
    hipDeviceSynchronize();


    rx.for_each_vertex(rxmesh::DEVICE,
                       [numberOfSamples,
                           context,
                           vertexNeighbors,
                           clustered_vertex,
                           sample_number,
                           number_of_neighbors] __device__(
                       const rxmesh::VertexHandle vh) {

                           if (clustered_vertex(vh, 0) ==
                               sample_number(vh, 0)) {
                               number_of_neighbors[clustered_vertex(vh, 0)] =
                                   vertexNeighbors[sample_number(vh, 0)].
                                   getNumberOfNeighbors();


                               /* printf("\n vertex %d : %d neighbors",
                       sample_number(vh, 0),
                       vertexNeighbors[sample_number(vh, 0)].getNumberOfNeighbors());
                       */
                           }
                       });
    hipDeviceSynchronize();

    int num_rows = numberOfSamples; // Set this appropriately
    CSR csr(num_rows, number_of_neighbors, vertexNeighbors, N);

    csr.printCSR();

    hipDeviceSynchronize(); // Ensure data is synchronized before accessing

   
    //for debug purposes
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,sample_number_pointer,
            clustered_vertex,
            number_of_neighbors,
            number_of_neighbors_coarse,
            context] __device__(const rxmesh::VertexHandle vh) {
            number_of_neighbors_coarse(vh, 0) = number_of_neighbors[sample_number(vh, 0)];

        });


    float* prolongation_operator;
    hipMallocManaged(&prolongation_operator,
                      N * numberOfSamples * sizeof(float));
    hipDeviceSynchronize();
   createProlongationOperator(csr.num_rows,
                                csr.row_ptr,
                                csr.value_ptr,
                                csr.number_of_neighbors,
                                N,
                                vertexCluster,
                                vertices, sample_pos,
                                prolongation_operator);
                                
    hipDeviceSynchronize();



    Eigen::MatrixXd verts;
    Eigen::MatrixXi faces;
    std::vector<std::array<double, 3>>
        vertexPositions;                            // To store vertex positions
        std::vector<std::vector<size_t>> faceIndices;  // To store face indices
 
    csr.GetRenderData(vertexPositions, faceIndices, sample_pos);

    polyscope::registerSurfaceMesh("dragon level 1", vertexPositions, faceIndices);

    

    ////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///next levels

    setCluster(numberOfSamples,
               distanceArray,
               clusterVertices,
               bitmask,
               currentLevel);


    do {

        *flagger = 0;
        clusterCSR(numberOfSamples,
                   sample_pos,
                   distanceArray,
                   clusterVertices,
                   flagger,
                   csr);
        hipDeviceSynchronize();
    } while (*flagger != 0);


    /*
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
         clustered_vertex,
         number_of_neighbors,
         row_ptr,value_ptr,
         context,
        vertices,vertex_pos, prolongation_operator,numberOfSamples] __device__(const rxmesh::VertexHandle vh) {

        //go through every triangle of my cluster
        const int cluster_point = clustered_vertex(vh, 0);
        const int start_pointer = row_ptr[clustered_vertex(vh,0)];
        const int end_pointer = row_ptr[clustered_vertex(vh,0)+1];

        float min_distance = 99999;
        Eigen::Vector3<float> selectedv1{0,0,0}, selectedv2{0, 0, 0},
            selectedv3{0, 0, 0};
        const Eigen::Vector3<float> q{
            vertex_pos(vh, 0), vertex_pos(vh, 1), vertex_pos(vh, 2)};

        int neighbor=0;
        int selected_neighbor=0;
        int neighbor_of_neighbor=0;
        int selected_neighbor_of_neighbor=0;


        for (int i=start_pointer;i<end_pointer;i++) {

            float distance;
             // get the neighbor vertex
            neighbor = value_ptr[i];  // assuming col_idx stores column
                                        // indices of neighbors in csr.

            // get the range of neighbors for this neighbor
            const int neighbor_start = row_ptr[neighbor];
            const int neighbor_end   = row_ptr[neighbor + 1];

            for (int j = neighbor_start; j < neighbor_end; j++) {
                neighbor_of_neighbor = value_ptr[j];

                for (int k=i+1;k<end_pointer;k++)
                {
                    if (value_ptr[k]==neighbor_of_neighbor) 
                    { 


                        
                        Eigen::Vector3<float> v1{vertices[cluster_point].x,
                                                 vertices[cluster_point].y,
                                                 vertices[cluster_point].z};
                        Eigen::Vector3<float> v2{vertices[neighbor].x,
                                                 vertices[neighbor].y,
                                                 vertices[neighbor].z};
                        Eigen::Vector3<float> v3{
                            vertices[neighbor_of_neighbor].x,
                            vertices[neighbor_of_neighbor].y,
                            vertices[neighbor_of_neighbor].z};

                        //find distance , if less than min dist, find bary coords, save them
                        float distance = projectedDistance(v1, v2, v3, q);
                        if (distance<min_distance) {
                            
                            min_distance = distance;
                            selectedv1   = v1;
                            selectedv2   = v2;
                            selectedv3   = v3;
                            selected_neighbor = neighbor;
                            selected_neighbor_of_neighbor =neighbor_of_neighbor;
                        }
                    }
                }
            }
        }
        // take the best bary coords
        auto [b1, b2, b3] = computeBarycentricCoordinates(
            selectedv1, selectedv2, selectedv3, q);
        // put it inside prolongation row, it will be unique so no race
        // condition
        int l = context.linear_id(vh);

        printf("\n %d final coords: %f %f %f", l, b1, b2, b3);


        //prolongation_operator[l * numberOfSamples + cluster_point]        = b1;
        //prolongation_operator[l * numberOfSamples + selected_neighbor] = b2;
        //prolongation_operator[l * numberOfSamples + selected_neighbor_of_neighbor]              = b3;

        
        prolongation_operator[l * numberOfSamples + cluster_point] =
            cluster_point;
        prolongation_operator[l * numberOfSamples + selected_neighbor] =
            selected_neighbor;
        prolongation_operator[l * numberOfSamples +
                              selected_neighbor_of_neighbor] =
            selected_neighbor_of_neighbor;

        //printf("\n%d at %d", l, l * numberOfSamples);



    });
    */


    std::cout << std::endl;
    std::cout << std::endl;

    hipDeviceSynchronize();




    hipFree(vertices);
    hipMallocManaged(&vertices, sizeof(Vec3) * csr.num_rows);

    float* distances;
    hipMallocManaged(&distances, sizeof(float) * csr.num_rows);


    number_of_neighbors_coarse.move(DEVICE, HOST);

    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_number",
        sample_number);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("distance", distance);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_level_bitmask",
        sample_level_bitmask);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "clusterPoint",
        clustered_vertex);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "number of neighbors",
        number_of_neighbors_coarse);


#if USE_POLYSCOPE
    polyscope::show();
#endif
}

