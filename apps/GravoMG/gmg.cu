#include "hip/hip_runtime.h"

#include "include/NeighborHandling.h"


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "include/GMGProlongatorCreation.h"

#include "include/VCycle.h"
#include "include/interactive.h"

#include "include/RXMeshMCFSetup.h"
#include "rxmesh/geometry_factory.h"

std::vector<int> intPointerArrayToVector(int* array, size_t size)
{
    return std::vector<int>(array, array + size);
}


void CreateNextLevelData(int      N,
                         int      numberOfSamples,
                         VertexData* vData_old,
                         VertexData* vData_new)
{

    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
        [=] __device__(int number) { vData_new[number].distance = 0;
                     });
}




void numberOfNeighbors(int              numberOfSamples,
                       VertexNeighbors* neighbors ,
    int              N,
                       CSR              csr,
                        VertexData* vData, int* number_of_neighbors

)
{
    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    int* neighborList;
    hipMallocManaged(&neighborList, sizeof(int) * numberOfSamples);

    for (int i = 0; i < numberOfSamples; i++)
        neighborList[i] = 0;
    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         //int currentCluster = vertexClusters[number];
                         int currentCluster = vData[number].cluster;

                         //neighbors[currentCluster].getNeighbors(neighborList);
                         for (int i = csr.row_ptr[number]; i < csr.row_ptr[number+1];i++) {
                             int currentNode = csr.value_ptr[i];
                             if (vData[currentNode].cluster != currentCluster) {
                                 //neighbors
                                 neighbors[currentCluster].addNeighbor(vData[currentNode].cluster);
                             }
                         }

                         // std::cout <<
                         // neighbors[0].getNeighbors().size();//this is the
                         // neighbor count
                     });


     thrust::device_vector<int> samples2(numberOfSamples);
    thrust::sequence(samples2.begin(), samples2.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {

                        number_of_neighbors[number] =
                             neighbors[number].getNumberOfNeighbors();

                     });


}




void setCluster(int    n,
                float* distance,
    int currentLevel,
    VertexData* vertex_data)
{
    thrust::device_vector<int> samples(n);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         // take bitmask
                         // if sample, the cluster is its own
                         // if not a sample, set cluster as -1
                         // set distance as infinity or 0 based on whether it is
                         // not or is a sample

                         if ((vertex_data[number].bitmask & (1 << currentLevel-1)) != 0) 
                         {
                            distance[number]        = 0;
                             vertex_data[number].cluster =
                                 vertex_data[number].sample_number;
                            
                             printf(
                                "\n%d which is sample %d is now a cluster vertex",
                                number,
                                vertex_data[number].sample_number);
                                
                         }
                         else 
                         {
                             vertex_data[number].cluster =-1;
                             /*
                             printf(
                                 "\n%d which is sample %d is not a cluster "
                                 "vertex",
                                 number,
                                 vertex_data[number].sample_number);
                                 */
                             distance[number]        = INFINITY;
                         }
        });
}







int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "torus.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bunnyhead.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bumpy-cube.obj");
    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    /*
    std::vector<std::vector<float>>    planeVerts;
    std::vector<std::vector<uint32_t>> planeFaces;
    uint32_t                           nx = 7;
    uint32_t                           ny = 6;
    create_plane(planeVerts, planeFaces, nx, ny);
    RXMeshStatic rx(planeFaces);
    rx.add_vertex_coordinates(planeVerts, "plane");
    */
    
    auto vertex_pos = *rx.get_input_vertex_coordinates();

    //attribute to sample,store and order samples
    auto sample_number = *rx.add_vertex_attribute<int>("sample_number", 1);
    auto distance      = *rx.add_vertex_attribute<float>("distance", 1);



    auto sample_level_bitmask = *rx.add_vertex_attribute<
        uint16_t>("bitmask", 1);
    auto clustered_vertex = *rx.add_vertex_attribute<int>("clustering", 1);

    auto number_of_neighbors_coarse = *rx.add_vertex_attribute<int>(
        "number of neighbors",
        1);


    int* flagger;
    hipMallocManaged(&flagger, sizeof(int));
    *flagger = 0;

    auto context = rx.get_context();

    constexpr uint32_t               CUDABlockSize = 512;
    rxmesh::LaunchBox<CUDABlockSize> lb;
    rx.prepare_launch_box({rxmesh::Op::VV},
                          lb,
                          (void*)sample_points<float, CUDABlockSize>);




    float ratio           = 8;
    int   N               = rx.get_num_vertices();
    int   numberOfLevels  = 0;
    for (int i=0;i<16;i++) {
        if ((int)N / (int)powf(ratio, i) > 8) {
            numberOfLevels++;
        }
    }
    std::cout << "\n Mesh can have " << numberOfLevels << " levels";


    int   currentLevel    = 1; // first coarse mesh
    int   numberOfSamplesForFirstLevel = N / powf(ratio, 1);//start
    int   numberOfSamples = N / powf(ratio, currentLevel);//start


    std::random_device rd;
    // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_int_distribution<> dist(0, N - 1);
    // From 0 to (number of points - 1)
    int seed = dist(gen);

    std::cout << "\nSeed: " << seed;

    VertexReduceHandle<float>              reducer(distance);
    hipcub::KeyValuePair<VertexHandle, float> farthestPoint;

    Vec3* vertices;
    Vec3* sample_pos;
    float*    distanceArray;
    int*    clusterVertices;

    
    // Allocate unified memory
    hipMallocManaged(&sample_pos, numberOfSamples * sizeof(Vec3));
    hipMallocManaged(&vertices, N * sizeof(Vec3));
    hipMallocManaged(&distanceArray, N * sizeof(int));
    hipMallocManaged(&clusterVertices, N * sizeof(int));

    hipDeviceSynchronize();

    
    // pre processing step
    //gathers samples for every level
    int j = 0;
    int currentSampleLevel = numberOfLevels;
    std::cout << "levels:";

    for (int q=0;q<numberOfLevels;q++) {
        std::cout << "\n  level " << q << " : " << N / powf(ratio, q);
    }
    for (int i = 0; i < numberOfSamplesForFirstLevel; i++) {
        if (i == N / (int)powf(ratio,  currentSampleLevel)) {
            currentSampleLevel--;
            std::cout << "\nNext sample level: " << currentSampleLevel;
        }

        rx.for_each_vertex(rxmesh::DEVICE,
                           [seed,
                               context,
                               sample_number,
                               sample_level_bitmask,
                               distance,
                               i,
                               currentSampleLevel,
                               sample_pos,
                               vertex_pos] __device__(
                           const rxmesh::VertexHandle vh) {
                               if (seed == context.linear_id(vh)) {
                                   sample_number(vh, 0) = i;
                                   //sample_number_point
                                   distance(vh, 0)      = 0;
                                   sample_pos[i].x        = vertex_pos(vh, 0);
                                   sample_pos[i].y      = vertex_pos(vh, 1);
                                   sample_pos[i].z        = vertex_pos(vh, 2);

                                   for (int k = 0; k < currentSampleLevel;
                                        k++) {
                                       sample_level_bitmask(vh, 0) |= (1 << k);
                                   }
                               } else {
                                   if (i == 0) {
                                       distance(vh, 0)      = INFINITY;
                                       sample_number(vh, 0) = -1;
                                   }
                               }
                           });

        do {
            hipDeviceSynchronize();
            *flagger = 0;
            sample_points<float, CUDABlockSize>
                <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                    rx.get_context(),
                    vertex_pos,
                    distance,
                    flagger);
            hipDeviceSynchronize();
            j++;

        } while (*flagger != 0);


        // reduction step
        farthestPoint = reducer.arg_max(distance, 0);
        seed          = rx.linear_id(farthestPoint.key);
    }

    std::cout << "\nSampling iterations: " << j; 


    sample_number.move(DEVICE, HOST);
    distance.move(DEVICE, HOST);
    sample_level_bitmask.move(DEVICE, HOST);
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///
    /// first level

    rxmesh::LaunchBox<CUDABlockSize> cb;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        cb,
        (void*)cluster_points<float, CUDABlockSize>);



    //clustering step
    j = 0;
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [
            sample_number,
            sample_level_bitmask,
            distance,
            currentLevel,
            clustered_vertex, context, vertices, vertex_pos ]
        __device__(const rxmesh::VertexHandle vh) {

                vertices[context.linear_id(vh)].x = vertex_pos(vh, 0);
                vertices[context.linear_id(vh)].y = vertex_pos(vh, 1);
                vertices[context.linear_id(vh)].z = vertex_pos(vh, 2);

            //if (sample_number(vh, 0) > -1)
            if ((sample_level_bitmask(vh,0) & (1 << (currentLevel-1))) !=0)
            {
                clustered_vertex(vh, 0) = sample_number(vh, 0);
                distance(vh, 0)         = 0;
            } else {
                distance(vh, 0)         = INFINITY;
                clustered_vertex(vh, 0) = -1;
            }
        });

    do {
        hipDeviceSynchronize();
        *flagger = 0;
        cluster_points<float, CUDABlockSize>
            <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                rx.get_context(),
                vertex_pos,
                distance,
                clustered_vertex,
                flagger);
        hipDeviceSynchronize();
        j++;
    } while (*flagger != 0);

    clustered_vertex.move(DEVICE, HOST);
    std::cout << "\Clustering iterations: " << j;

    int* vertexCluster;
    hipMallocManaged(&vertexCluster, sizeof(int) * N);
    rx.for_each_vertex(rxmesh::DEVICE,
                       [
                           clustered_vertex,
                           context,vertexCluster] __device__(
                       const rxmesh::VertexHandle vh) {
                           vertexCluster[context.linear_id(vh)] =
                               clustered_vertex(vh, 0);


                       });
    hipDeviceSynchronize();

    int* number_of_neighbors;
    hipMallocManaged(&number_of_neighbors, numberOfSamples * sizeof(int));
    for (int i = 0; i < numberOfSamples; i++) {
        number_of_neighbors[i] = 0;
    }
    hipDeviceSynchronize();

    rxmesh::LaunchBox<CUDABlockSize> nn;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        nn,
        (void*)findNumberOfCoarseNeighbors<float, CUDABlockSize>);


    //find number of neighbors without the bit matrix


    // Allocate memory for vertex neighbors
    VertexNeighbors* vertexNeighbors;
    hipError_t      err = hipMallocManaged(&vertexNeighbors,
                                             numberOfSamples * sizeof(
                                                 VertexNeighbors));

    findNumberOfCoarseNeighbors<float, CUDABlockSize>
        <<<nn.blocks, nn.num_threads, nn.smem_bytes_dyn>>>(
            rx.get_context(),
            clustered_vertex,
            number_of_neighbors,
            vertexNeighbors);
    hipDeviceSynchronize();


    rx.for_each_vertex(rxmesh::DEVICE,
                       [numberOfSamples,
                           context,
                           vertexNeighbors,
                           clustered_vertex,
                           sample_number,
                           number_of_neighbors] __device__(
                       const rxmesh::VertexHandle vh) {

                           if (clustered_vertex(vh, 0) ==
                               sample_number(vh, 0)) {
                               number_of_neighbors[clustered_vertex(vh, 0)] =
                                   vertexNeighbors[sample_number(vh, 0)].
                                   getNumberOfNeighbors();
                           }
                       });
    hipDeviceSynchronize();

    int num_rows = numberOfSamples; // Set this appropriately
    CSR csr(num_rows, number_of_neighbors, vertexNeighbors, N);

    hipDeviceSynchronize(); // Ensure data is synchronized before accessing

   
    //for debug purposes
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
            clustered_vertex,
            number_of_neighbors,
            number_of_neighbors_coarse,
            context] __device__(const rxmesh::VertexHandle vh) {
            number_of_neighbors_coarse(vh, 0) = number_of_neighbors[sample_number(vh, 0)];

        });


    float* prolongation_operator;
    hipMallocManaged(&prolongation_operator,
                      N * numberOfSamples * sizeof(float));
    hipDeviceSynchronize();
    std::vector<CSR> operatorsCSR;
    operatorsCSR.push_back(CSR(N));

    std::vector<CSR> prolongationOperatorCSR;  // work on allocating these for v cycle

    CSR firstOperator(N);

   createProlongationOperator(csr.num_rows,
                                csr.row_ptr,
                                csr.value_ptr,
                                csr.number_of_neighbors,
                                N,
                                vertexCluster,
                                vertices, sample_pos, firstOperator.value_ptr, firstOperator.data_ptr,
                                prolongation_operator);

   prolongationOperatorCSR.push_back(firstOperator);

    hipDeviceSynchronize();

    Eigen::MatrixXd verts;
    Eigen::MatrixXi faces;
    std::vector<std::array<double, 3>>
        vertexPositions;                            // To store vertex positions
        std::vector<std::vector<size_t>> faceIndices;  // To store face indices

    csr.GetRenderData(vertexPositions, faceIndices, sample_pos);

    polyscope::registerSurfaceMesh(
        "mesh level 1", vertexPositions, faceIndices);

    
    //set 1st level node data
    VertexData* oldVdata;
    hipMallocManaged(&oldVdata, sizeof(VertexData) * numberOfSamples);

    rx.for_each_vertex(rxmesh::DEVICE,
                       [sample_number,
                        oldVdata,
                        clustered_vertex,
                        vertex_pos,
         sample_level_bitmask,
                        context] __device__(const rxmesh::VertexHandle vh) {


            if (sample_number(vh, 0) != -1) {
                //printf("\nputting data for sample %d", sample_number(vh, 0));

                oldVdata[sample_number(vh, 0)].distance = 0;
                oldVdata[sample_number(vh, 0)].linear_id =
                    context.linear_id(vh);
                oldVdata[sample_number(vh, 0)].sample_number =
                    sample_number(vh, 0);
                oldVdata[sample_number(vh, 0)].bitmask =
                    sample_level_bitmask(vh, 0);
                oldVdata[sample_number(vh, 0)].position.x = vertex_pos(vh, 0);
                oldVdata[sample_number(vh, 0)].position.y = vertex_pos(vh, 1);
                oldVdata[sample_number(vh, 0)].position.z = vertex_pos(vh, 2);
                oldVdata[sample_number(vh, 0)].cluster =
                    clustered_vertex(vh, 0);
            }

    });
    


    ////////////////////////////////////////////////////////////////////////////////////////
    ///
    ///next levels




    CSR lastCSR                 = csr;
    CSR currentCSR              = csr;
    int currentNumberOfVertices = N;
    //numberOfSamples;
    int currentNumberOfSamples = numberOfSamples;
    /// ratio;
    std::vector<float*> prolongationOperators;
    prolongationOperators.resize(numberOfLevels-1);

    prolongationOperators[0] = prolongation_operator;

    std::vector<Eigen::MatrixXd> vertsArray;
    std::vector<Eigen::MatrixXi> facesArray;
    std::vector<std::vector<std::array<double, 3>>>
        vertexPositionsArray;  // To store vertex positions
    std::vector<std::vector<std::vector<size_t>>>
        faceIndicesArray;  // To store face indices

    std::vector<std::vector<int>> clustering;



    vertsArray.resize(numberOfLevels);
    facesArray.resize(numberOfLevels);
    vertexPositionsArray.resize(numberOfLevels);
    faceIndicesArray.resize(numberOfLevels);
    clustering.resize(numberOfLevels);


   CSR a(numberOfSamples);

    //operatorsCSR.resize(numberOfLevels-1);

    for (int level = 1; level < numberOfLevels - 1; level++) {

        currentNumberOfSamples /= 8;
        currentNumberOfVertices /= 8;
        a = CSR(currentNumberOfVertices);

        std::cout << "\nlevel : " << level;
        std::cout << "\n current number of samples: " << currentNumberOfSamples;
        std::cout << "\n current number of vertices: "
                  << currentNumberOfVertices;
        setCluster(currentNumberOfVertices, distanceArray, level + 1, oldVdata);

        do {

            *flagger = 0;
            clusterCSR(currentNumberOfVertices,
                       sample_pos,
                       distanceArray,
                       vertexCluster,
                       flagger,
                       lastCSR,
                       oldVdata);
            hipDeviceSynchronize();
        } while (*flagger != 0);

        clustering[level - 1].resize(currentNumberOfVertices);
        clustering[level - 1] = intPointerArrayToVector(vertexCluster, currentNumberOfVertices);


        polyscope::getSurfaceMesh("mesh level " 
            + std::to_string(level))
        ->addVertexScalarQuantity("clustered vertices", clustering[level - 1]);


        VertexNeighbors* vertexNeighbors2;
        err = hipMallocManaged(
            &vertexNeighbors2,
            currentNumberOfVertices * sizeof(VertexNeighbors));

        int* number_of_neighbors2;
        hipMallocManaged(&number_of_neighbors2,
                          currentNumberOfVertices * sizeof(int));


        numberOfNeighbors(currentNumberOfSamples,
                          vertexNeighbors2,
                          currentNumberOfVertices,
                          lastCSR,
                          oldVdata,
                          number_of_neighbors2);


        hipDeviceSynchronize();
        currentCSR = CSR(currentNumberOfSamples,
                         number_of_neighbors2,
                         vertexNeighbors2,
                         currentNumberOfVertices);

        //currentCSR.printCSR();

        currentCSR.GetRenderData(vertexPositionsArray[level - 1],
                                 faceIndicesArray[level - 1],
                                 sample_pos);


        polyscope::registerSurfaceMesh(
            "mesh level " + std::to_string(level + 1),
            vertexPositionsArray[level - 1],
            faceIndicesArray[level - 1]);

        float* prolongationOperator2;
        hipMallocManaged(&prolongationOperator2,
                          sizeof(float) * currentNumberOfSamples * currentNumberOfVertices);
        hipDeviceSynchronize();
        prolongationOperators[level] = prolongationOperator2;
        hipDeviceSynchronize();

        createProlongationOperator(currentNumberOfSamples,
                                   currentCSR.row_ptr,
                                   currentCSR.value_ptr,
                                   number_of_neighbors2,
                                   currentNumberOfVertices,
                                   oldVdata,
                                   prolongationOperator2);


        createProlongationOperator(currentNumberOfSamples,
                                   currentCSR.row_ptr,
                                   currentCSR.value_ptr,
                                   a.value_ptr,
                                   a.data_ptr,
                                   number_of_neighbors2,
                                   currentNumberOfVertices,
                                   oldVdata);
        prolongationOperatorCSR.push_back(a);

        hipDeviceSynchronize();  // Ensure data is synchronized before accessing
        
        lastCSR = currentCSR; //next mesh level
    }

    hipDeviceSynchronize();

    ////////////////////////////////////////////////////
    /*
    std::cout << "\n\n\n\n\n\n";
    std::cout << "first\n";

    CSR t1(3, 3,3);

    VectorCSR3D b(3);
    VectorCSR3D vectorResult(3);
    for (int i = 0; i < vectorResult.n * 3; i++) {
        vectorResult.vector[i] = 0.0f;
    }
    // Set right hand side - different for each component of each point
    b.vector[0] = 4.0;  // Point 1 (x,y,z)
    b.vector[1] = 2.0;
    b.vector[2] = 1.0;

    b.vector[3] = 3.0;  // Point 2 (x,y,z)
    b.vector[4] = 5.0;
    b.vector[5] = 2.0;

    b.vector[6] = 1.0;  // Point 3 (x,y,z)
    b.vector[7] = 2.0;
    b.vector[8] = 6.0;


    t1.printCSR();

    gauss_jacobi_CSR_3D(t1, vectorResult.vector, b.vector, 50);
    
    std::cout << "\nVector result: ";
    for (int i = 0; i < vectorResult.n; i++) {
        std::cout << "\n";
        std::cout << vectorResult.vector[i * 3] << " ";
        std::cout << vectorResult.vector[i * 3 + 1] << " ";
        std::cout << vectorResult.vector[i * 3 + 2] << " ";
    }
    
    */
    //contruct equations as CSR matrices

    
    constexpr uint32_t blockThreads = 256;

    uint32_t num_vertices = rx.get_num_vertices();

    auto coords = rx.get_input_vertex_coordinates();

    SparseMatrix<float> A_mat(rx);
    DenseMatrix<float>  B_mat(rx, rx.get_num_vertices(), 3);


    std::shared_ptr<DenseMatrix<float>> X_mat = coords->to_matrix();
    // B set up
    LaunchBox<blockThreads> launch_box_B;
    rx.prepare_launch_box({Op::VV},
                          launch_box_B,
                          (void*)mcf_B_setup<float, blockThreads>);

    mcf_B_setup<float, blockThreads><<<launch_box_B.blocks,
                                       launch_box_B.num_threads,
                                       launch_box_B.smem_bytes_dyn>>>(
        rx.get_context(), *coords, B_mat,true);

    

    // A and X set up
    LaunchBox<blockThreads> launch_box_A_X;
    rx.prepare_launch_box({Op::VV},
                          launch_box_A_X,
                          (void*)mcf_A_setup<float, blockThreads>,
                          true);

    mcf_A_setup<float, blockThreads>
        <<<launch_box_A_X.blocks,
           launch_box_A_X.num_threads,
           launch_box_A_X.smem_bytes_dyn>>>(rx.get_context(),
                                            *coords,
                                            A_mat,
                                            true,10);

    A_mat.move(DEVICE, HOST);
    B_mat.move(DEVICE, HOST);

    CSR A_csr(A_mat,A_mat.row_ptr(),A_mat.col_idx(),A_mat.non_zeros());
    VectorCSR3D B_v(B_mat.rows());


    //A_csr.printCSR();

    std::cout << "\nRHS:";
    std::cout << "\n Number of rows of B:"<<B_mat.rows();

    for (int i=0;i<B_mat.rows();i++) {
        B_v.vector[i*3] = B_mat(i, 0);
        B_v.vector[i*3+1] = B_mat(i, 1);
        B_v.vector[i*3+2] = B_mat(i, 2);

    }

    
    std::vector<CSR> equationsPerLevel;
    equationsPerLevel.push_back(A_csr);

    currentNumberOfSamples = numberOfSamples;

    CSR result = A_csr;

    //make all the equations for each level
    
    for (int i=0;i<numberOfLevels-1;i++) 
    {

        result = multiplyCSR(result.num_rows,
                                 result.num_rows,
                                 currentNumberOfSamples,
                                 result.row_ptr,
                                 result.value_ptr,
                                 result.data_ptr,
                                 result.non_zeros,
                             prolongationOperatorCSR[i].row_ptr,
                             prolongationOperatorCSR[i].value_ptr,
                             prolongationOperatorCSR[i].data_ptr,
                             prolongationOperatorCSR[i].non_zeros);

        CSR transposeOperator =
        transposeCSR(prolongationOperatorCSR[i], currentNumberOfSamples);

        result = multiplyCSR(transposeOperator.num_rows,
                         prolongationOperatorCSR[i].num_rows,
                         numberOfSamples,
                         transposeOperator.row_ptr,
                         transposeOperator.value_ptr,
                         transposeOperator.data_ptr,
                         transposeOperator.non_zeros,
                         result.row_ptr,
                         result.value_ptr,
                         result.data_ptr,
                         result.non_zeros);

        equationsPerLevel.push_back(result);

        currentNumberOfSamples/=ratio;
        //std::cout << "Equation level " << i << "\n\n";
        //equationsPerLevel[i].printCSR();
    }
   



    
    hipDeviceSynchronize();


    GMGVCycle gmg(N);

    gmg.prolongationOperators = prolongationOperatorCSR;
    gmg.LHS                   = equationsPerLevel;
    gmg.RHS                   = B_v;
    //gmg.max_number_of_levels  = 1;  // numberOfLevels-1;
    gmg.max_number_of_levels  = 0;
    gmg.post_relax_iterations = 5;
    gmg.pre_relax_iterations = 5;


    std::cout << "\nNumber of equations LHS:" << gmg.LHS.size();
    std::cout << "\nNumber of operators:" << gmg.prolongationOperators.size();
    std::cout << "\nMax level:" << gmg.max_number_of_levels;

        

        //gauss_jacobi_CSR_3D(gmg.LHS[0], gmg.X.vector, gmg.RHS.vector, 1000);


    int numberOfVCycles=2;
    for (int i=0;i<numberOfVCycles;i++)
        gmg.VCycle(gmg.LHS[0], gmg.RHS, gmg.X, 0);

    std::cout << "\nFinal output: \n";
    std::vector<std::array<double, 3>> vertexMeshPositions;
    vertexMeshPositions.resize(gmg.X.n);
    
    for (int i = 0; i < gmg.X.n; i++) {

        /*
        std::cout << " \n";
        std::cout << B_v.vector[3 * i] << " ";
        std::cout << B_v.vector[3 * i+1] << " ";
        std::cout << B_v.vector[3 * i+2] << " ";
        std::cout << " | ";
        std::cout << gmg.X.vector[3 * i] << " ";
        std::cout << gmg.X.vector[3 * i+1] << " ";
        std::cout << gmg.X.vector[3 * i+2] << " ";
        */


        vertexMeshPositions[i] = {
            gmg.X.vector[3*i], gmg.X.vector[i * 3 + 1], gmg.X.vector[i * 3 + 2]};
    }
    
     polyscope::registerSurfaceMesh("output mesh", vertexMeshPositions,
        rx.get_polyscope_mesh()->faces);

     
    auto polyscope_callback = [&]() mutable {

//        menu();
         ImGui::Begin("GMG Parameters");

        ImGui::InputInt("Number of Levels", &gmg.max_number_of_levels);
        ImGui::InputInt("Number of V cycles", &numberOfVCycles);
        ImGui::InputInt("Number of pre solve smoothing iterations", &gmg.pre_relax_iterations);
        ImGui::InputInt("Number of post solve smoothing iterations", &gmg.post_relax_iterations);
        ImGui::InputInt("Number of direct solve iterations", &gmg.directSolveIterations);
        ImGui::SliderFloat("Omega", &gmg.omega,0.0,1.0);


        if (ImGui::Button("Run V Cycles again")) {
            std::cout << "\n---------------NEW SOLVE INITIATED--------------------\n";
            gmg.X.reset();

            for (int i=0;i<numberOfVCycles;i++)
                gmg.VCycle(gmg.LHS[0], gmg.RHS, gmg.X, 0);


           // vertexMeshPositions.clear();
           // vertexMeshPositions.resize(gmg.X.n);

            for (int i = 0; i < gmg.X.n; i++)
            {
                /*
                std::cout << " \n";
                std::cout << B_v.vector[3 * i] << " ";
                std::cout << B_v.vector[3 * i + 1] << " ";
                std::cout << B_v.vector[3 * i + 2] << " ";
                std::cout << " | ";
                std::cout << gmg.X.vector[3 * i] << " ";
                std::cout << gmg.X.vector[3 * i + 1] << " ";
                std::cout << gmg.X.vector[3 * i + 2] << " ";
                */
                
                vertexMeshPositions[i] = {gmg.X.vector[3 * i],
                                          gmg.X.vector[3 * i + 1],
                                          gmg.X.vector[3 * i + 2]};
            }
            
            //polyscope::removeSurfaceMesh("output mesh");

            polyscope::registerSurfaceMesh("output mesh 2",
                                           vertexMeshPositions,
                                           rx.get_polyscope_mesh()->faces);
        }

        ImGui::End();
    };
    
      polyscope::state::userCallback = polyscope_callback;

    

    //////////////////////////////////////////////////////////////////

    /*
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
         clustered_vertex,
         number_of_neighbors,
         row_ptr,value_ptr,
         context,
        vertices,vertex_pos, prolongation_operator,numberOfSamples] __device__(const rxmesh::VertexHandle vh) {

        //go through every triangle of my cluster
        const int cluster_point = clustered_vertex(vh, 0);
        const int start_pointer = row_ptr[clustered_vertex(vh,0)];
        const int end_pointer = row_ptr[clustered_vertex(vh,0)+1];

        float min_distance = 99999;
        Eigen::Vector3<float> selectedv1{0,0,0}, selectedv2{0, 0, 0},
            selectedv3{0, 0, 0};
        const Eigen::Vector3<float> q{
            vertex_pos(vh, 0), vertex_pos(vh, 1), vertex_pos(vh, 2)};

        int neighbor=0;
        int selected_neighbor=0;
        int neighbor_of_neighbor=0;
        int selected_neighbor_of_neighbor=0;


        for (int i=start_pointer;i<end_pointer;i++) {

            float distance;
             // get the neighbor vertex
            neighbor = value_ptr[i];  // assuming col_idx stores column
                                        // indices of neighbors in csr.

            // get the range of neighbors for this neighbor
            const int neighbor_start = row_ptr[neighbor];
            const int neighbor_end   = row_ptr[neighbor + 1];

            for (int j = neighbor_start; j < neighbor_end; j++) {
                neighbor_of_neighbor = value_ptr[j];

                for (int k=i+1;k<end_pointer;k++)
                {
                    if (value_ptr[k]==neighbor_of_neighbor) 
                    { 


                        
                        Eigen::Vector3<float> v1{vertices[cluster_point].x,
                                                 vertices[cluster_point].y,
                                                 vertices[cluster_point].z};
                        Eigen::Vector3<float> v2{vertices[neighbor].x,
                                                 vertices[neighbor].y,
                                                 vertices[neighbor].z};
                        Eigen::Vector3<float> v3{
                            vertices[neighbor_of_neighbor].x,
                            vertices[neighbor_of_neighbor].y,
                            vertices[neighbor_of_neighbor].z};

                        //find distance , if less than min dist, find bary coords, save them
                        float distance = projectedDistance(v1, v2, v3, q);
                        if (distance<min_distance) {
                            
                            min_distance = distance;
                            selectedv1   = v1;
                            selectedv2   = v2;
                            selectedv3   = v3;
                            selected_neighbor = neighbor;
                            selected_neighbor_of_neighbor =neighbor_of_neighbor;
                        }
                    }
                }
            }
        }
        // take the best bary coords
        auto [b1, b2, b3] = computeBarycentricCoordinates(
            selectedv1, selectedv2, selectedv3, q);
        // put it inside prolongation row, it will be unique so no race
        // condition
        int l = context.linear_id(vh);

        printf("\n %d final coords: %f %f %f", l, b1, b2, b3);


        //prolongation_operator[l * numberOfSamples + cluster_point]        = b1;
        //prolongation_operator[l * numberOfSamples + selected_neighbor] = b2;
        //prolongation_operator[l * numberOfSamples + selected_neighbor_of_neighbor]              = b3;

        
        prolongation_operator[l * numberOfSamples + cluster_point] =
            cluster_point;
        prolongation_operator[l * numberOfSamples + selected_neighbor] =
            selected_neighbor;
        prolongation_operator[l * numberOfSamples +
                              selected_neighbor_of_neighbor] =
            selected_neighbor_of_neighbor;

        //printf("\n%d at %d", l, l * numberOfSamples);



    });
    


    std::cout << std::endl;
    std::cout << std::endl;

    hipDeviceSynchronize();




    hipFree(vertices);
    hipMallocManaged(&vertices, sizeof(Vec3) * csr.num_rows);

    float* distances;
    hipMallocManaged(&distances, sizeof(float) * csr.num_rows);


    number_of_neighbors_coarse.move(DEVICE, HOST);

    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_number",
        sample_number);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("distance", distance);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "sample_level_bitmask",
        sample_level_bitmask);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "clusterPoint",
        clustered_vertex);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "number of neighbors",
        number_of_neighbors_coarse);
        */





#if USE_POLYSCOPE
    polyscope::show();
#endif
}



    ////////////////////////////////////
/*
std::cout << "\n\n\n\n\n\n";
std::cout << "first\n";

CSR t1(3, 3);

VectorCSR3D vectorTest(3);
VectorCSR3D vectorResult(3);

for (int i=0;i<3;i++) {
    vectorTest.vector[i*3] = i+1;
    vectorTest.vector[i*3 + 1] = i + 1;
    vectorTest.vector[i*3+2] = i+1;
}

SpMV_CSR_3D(t1.row_ptr,
         t1.value_ptr,
         t1.data_ptr,
         vectorTest.vector,
         vectorResult.vector,
         3);

std::cout << "\nVector result: ";
for (int i=0;i<vectorResult.n;i++) {
    std::cout << "\n";
    std::cout << vectorResult.vector[i*3] << " ";
    std::cout << vectorResult.vector[i*3+1] << " ";
    std::cout << vectorResult.vector[i*3+2] << " ";
}
*/
//////////////////////////////////////////////