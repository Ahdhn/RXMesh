#include "hip/hip_runtime.h"

#include "include/NeighborHandling.h"


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/sequence.h>


#include "include/GMGProlongatorCreation.h"

#include "include/VCycle.h"
#include "include/interactive.h"

#include "include/RXMeshMCFSetup.h"
#include "rxmesh/geometry_factory.h"

std::vector<int> intPointerArrayToVector(int* array, size_t size)
{
    return std::vector<int>(array, array + size);
}

void numberOfNeighbors(int              numberOfSamples,
                       VertexNeighbors* neighbors ,
    int              N,
                       CSR              csr,
                        VertexData* vData, int* number_of_neighbors

)
{
    thrust::device_vector<int> samples(N);
    thrust::sequence(samples.begin(), samples.end());

    int* neighborList;
    hipMallocManaged(&neighborList, sizeof(int) * numberOfSamples);

    for (int i = 0; i < numberOfSamples; i++)
        neighborList[i] = 0;
    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         //int currentCluster = vertexClusters[number];
                         int currentCluster = vData[number].cluster;

                         //neighbors[currentCluster].getNeighbors(neighborList);
                         for (int i = csr.row_ptr[number]; i < csr.row_ptr[number+1];i++) {
                             int currentNode = csr.value_ptr[i];
                             if (vData[currentNode].cluster != currentCluster) {
                                 //neighbors
                                 neighbors[currentCluster].addNeighbor(vData[currentNode].cluster);
                             }
                         }

                         // std::cout <<
                         // neighbors[0].getNeighbors().size();//this is the
                         // neighbor count
                     });


     thrust::device_vector<int> samples2(numberOfSamples);
    thrust::sequence(samples2.begin(), samples2.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {

                        number_of_neighbors[number] =
                             neighbors[number].getNumberOfNeighbors();

                     });


}




void setCluster(int    n,
                float* distance,
    int currentLevel,
    VertexData* vertex_data)
{
    thrust::device_vector<int> samples(n);
    thrust::sequence(samples.begin(), samples.end());

    thrust::for_each(thrust::device,
                     samples.begin(),
                     samples.end(),
                     [=] __device__(int number) {
                         // take bitmask
                         // if sample, the cluster is its own
                         // if not a sample, set cluster as -1
                         // set distance as infinity or 0 based on whether it is
                         // not or is a sample

                         if ((vertex_data[number].bitmask & (1 << currentLevel-1)) != 0) 
                         {
                            distance[number]        = 0;
                             vertex_data[number].cluster =
                                 vertex_data[number].sample_number;
                            
                             printf(
                                "\n%d which is sample %d is now a cluster vertex",
                                number,
                                vertex_data[number].sample_number);
                                
                         }
                         else 
                         {
                             vertex_data[number].cluster =-1;
                             /*
                             printf(
                                 "\n%d which is sample %d is not a cluster "
                                 "vertex",
                                 number,
                                 vertex_data[number].sample_number);
                                 */
                             distance[number]        = INFINITY;
                         }
        });
}


void createProlongationOperators(int N, int numberOfSamples, int numberOfLevels, float ratio, Vec3* sample_pos, CSR csr, std::vector<CSR>& prolongationOperatorCSR, VertexData* oldVdata, float* distanceArray, int* vertexCluster)
{
    hipDeviceSynchronize();
    int* flag;
    hipMallocManaged(&flag, sizeof(int));
    *flag = 0;

    hipError_t err;
    CSR         lastCSR                 = csr;
    CSR         currentCSR              = csr;
    int         currentNumberOfVertices = N;
    //numberOfSamples;
    int currentNumberOfSamples = numberOfSamples;
    /// ratio;

    std::vector<Eigen::MatrixXd> vertsArray;
    std::vector<Eigen::MatrixXi> facesArray;
    std::vector<std::vector<std::array<double, 3>>> vertexPositionsArray;  // To store vertex positions
    std::vector<std::vector<std::vector<size_t>>> faceIndicesArray;  // To store face indices

    std::vector<std::vector<int>> clustering;

    vertsArray.resize(numberOfLevels);
    facesArray.resize(numberOfLevels);
    vertexPositionsArray.resize(numberOfLevels);
    faceIndicesArray.resize(numberOfLevels);
    clustering.resize(numberOfLevels);


    CSR a(numberOfSamples);

    //operatorsCSR.resize(numberOfLevels-1);

    for (int level = 1; level < numberOfLevels - 1; level++) {

        currentNumberOfSamples /= ratio;
        currentNumberOfVertices /= ratio;
        a = CSR(currentNumberOfVertices);

        std::cout << "\nlevel : " << level;
        std::cout << "\n current number of samples: " << currentNumberOfSamples;
        std::cout << "\n current number of vertices: "
            << currentNumberOfVertices;
        setCluster(currentNumberOfVertices, distanceArray, level + 1, oldVdata);

        do {

            *flag = 0;
            clusterCSR(currentNumberOfVertices,
                       sample_pos,
                       distanceArray,
                       vertexCluster,
                       flag,
                       lastCSR,
                       oldVdata);
            hipDeviceSynchronize();
        } while (*flag != 0);

        clustering[level - 1].resize(currentNumberOfVertices);
        clustering[level - 1] = intPointerArrayToVector(vertexCluster, currentNumberOfVertices);


        polyscope::getSurfaceMesh("mesh level " 
                                  + std::to_string(level))
            ->addVertexScalarQuantity("clustered vertices", clustering[level - 1]);


        VertexNeighbors* vertexNeighbors2;
        err = hipMallocManaged(
            &vertexNeighbors2,
            currentNumberOfVertices * sizeof(VertexNeighbors));

        int* number_of_neighbors2;
        hipMallocManaged(&number_of_neighbors2,
                          currentNumberOfVertices * sizeof(int));


        numberOfNeighbors(currentNumberOfSamples,
                          vertexNeighbors2,
                          currentNumberOfVertices,
                          lastCSR,
                          oldVdata,
                          number_of_neighbors2);


        hipDeviceSynchronize();
        currentCSR = CSR(currentNumberOfSamples,
                         number_of_neighbors2,
                         vertexNeighbors2,
                         currentNumberOfVertices);

        currentCSR.printCSR();

        currentCSR.GetRenderData(vertexPositionsArray[level - 1],
                                 faceIndicesArray[level - 1],
                                 sample_pos);


        polyscope::registerSurfaceMesh(
            "mesh level " + std::to_string(level + 1),
            vertexPositionsArray[level - 1],
            faceIndicesArray[level - 1]);


        createProlongationOperator(currentNumberOfSamples,
                                   currentCSR.row_ptr,
                                   currentCSR.value_ptr,
                                   a.value_ptr,
                                   a.data_ptr,
                                   number_of_neighbors2,
                                   currentNumberOfVertices,
                                   oldVdata);
        prolongationOperatorCSR.push_back(a);

        hipDeviceSynchronize(); // Ensure data is synchronized before accessing
        
        lastCSR = currentCSR; //next mesh level
    }
    hipFree(flag);
    hipDeviceSynchronize();
}

void constructLHS(CSR A_csr, std::vector<CSR> prolongationOperatorCSR, std::vector<CSR>& equationsPerLevel, int numberOfLevels, int numberOfSamples, float ratio)
{
    int currentNumberOfSamples = numberOfSamples;

    CSR result = A_csr;

    //make all the equations for each level
    
    for (int i =0;i<numberOfLevels-1;i++) 
    {
        result = multiplyCSR(result.num_rows,
                             result.num_rows,
                             currentNumberOfSamples,
                             result.row_ptr,
                             result.value_ptr,
                             result.data_ptr,
                             result.non_zeros,
                             prolongationOperatorCSR[i].row_ptr,
                             prolongationOperatorCSR[i].value_ptr,
                             prolongationOperatorCSR[i].data_ptr,
                             prolongationOperatorCSR[i].non_zeros);

        CSR transposeOperator =
            transposeCSR(prolongationOperatorCSR[i], currentNumberOfSamples);

        result = multiplyCSR(transposeOperator.num_rows,
                             prolongationOperatorCSR[i].num_rows,
                             numberOfSamples,
                             transposeOperator.row_ptr,
                             transposeOperator.value_ptr,
                             transposeOperator.data_ptr,
                             transposeOperator.non_zeros,
                             result.row_ptr,
                             result.value_ptr,
                             result.data_ptr,
                             result.non_zeros);

        equationsPerLevel.push_back(result);

        currentNumberOfSamples/=ratio;
        std::cout << "Equation level " << i << "\n\n";
        equationsPerLevel[i].printCSR();
    }
}

void setVertexData(RXMeshStatic &rx, Context &context, VertexData* oldVdata, Attribute<float, VertexHandle> vertex_pos, Attribute<int, VertexHandle> sample_number, Attribute<unsigned short, VertexHandle> sample_level_bitmask, Attribute<int, VertexHandle> clustered_vertex)
{
    rx.for_each_vertex(rxmesh::DEVICE,
                       [sample_number,
                           oldVdata,
                           clustered_vertex,
                           vertex_pos,
                           sample_level_bitmask,
                           context] __device__(const rxmesh::VertexHandle vh) {


                           if (sample_number(vh, 0) != -1) {
                               //printf("\nputting data for sample %d", sample_number(vh, 0));

                               oldVdata[sample_number(vh, 0)].distance  = 0;
                               oldVdata[sample_number(vh, 0)].linear_id =
                                   context.linear_id(vh);
                               oldVdata[sample_number(vh, 0)].sample_number =
                                   sample_number(vh, 0);
                               oldVdata[sample_number(vh, 0)].bitmask =
                                   sample_level_bitmask(vh, 0);
                               oldVdata[sample_number(vh, 0)].position.x = vertex_pos(vh, 0);
                               oldVdata[sample_number(vh, 0)].position.y = vertex_pos(vh, 1);
                               oldVdata[sample_number(vh, 0)].position.z = vertex_pos(vh, 2);
                               oldVdata[sample_number(vh, 0)].cluster =
                                   clustered_vertex(vh, 0);
                           }

                       });
}

int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "torus.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bunnyhead.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bumpy-cube.obj");
    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    
    std::vector<std::vector<float>>    planeVerts;
    std::vector<std::vector<uint32_t>> planeFaces;
    uint32_t                           nx = 15;
    uint32_t                           ny = 15;
    create_plane(planeVerts, planeFaces, nx, ny);
    RXMeshStatic rx(planeFaces);
    rx.add_vertex_coordinates(planeVerts, "plane");
    
    
    auto vertex_pos = *rx.get_input_vertex_coordinates();

    //attribute to sample,store and order samples
    auto sample_number = *rx.add_vertex_attribute<int>("sample_number", 1);
    auto distance      = *rx.add_vertex_attribute<float>("distance", 1);



    auto sample_level_bitmask = *rx.add_vertex_attribute<
        uint16_t>("bitmask", 1);
    auto clustered_vertex = *rx.add_vertex_attribute<int>("clustering", 1);

    auto number_of_neighbors_coarse = *rx.add_vertex_attribute<int>(
        "number of neighbors",
        1);


    int* flagger;
    hipMallocManaged(&flagger, sizeof(int));
    *flagger = 0;

    auto context = rx.get_context();

    constexpr uint32_t               CUDABlockSize = 512;
    rxmesh::LaunchBox<CUDABlockSize> lb;
    rx.prepare_launch_box({rxmesh::Op::VV},
                          lb,
                          (void*)sample_points<float, CUDABlockSize>);




    float ratio           = 5;
    int   N               = rx.get_num_vertices();
    int   numberOfLevels  = 0;
    for (int i=0;i<16;i++) {
        if ((int)N / (int)powf(ratio, i) > 6) {
            numberOfLevels++;
        }
    }
    
    std::cout << "\n Mesh can have " << numberOfLevels << " levels";


    int   currentLevel    = 1; // first coarse mesh
    int   numberOfSamplesForFirstLevel = N / powf(ratio, 1);//start
    int   numberOfSamples = N / powf(ratio, currentLevel);//start


    std::random_device rd;
    // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
    std::uniform_int_distribution<> dist(0, N - 1);
    // From 0 to (number of points - 1)
    int seed = 20;
    //dist(gen);

    std::cout << "\nSeed: " << seed;

    VertexReduceHandle<float>              reducer(distance);
    hipcub::KeyValuePair<VertexHandle, float> farthestPoint;

    Vec3* vertices;
    Vec3* sample_pos;
    float*    distanceArray;
    int*    clusterVertices;

    
    // Allocate unified memory
    hipMallocManaged(&sample_pos, numberOfSamples * sizeof(Vec3));
    hipMallocManaged(&vertices, N * sizeof(Vec3));
    hipMallocManaged(&distanceArray, N * sizeof(int));
    hipMallocManaged(&clusterVertices, N * sizeof(int));

    hipDeviceSynchronize();

    
    // pre processing step
    //gathers samples for every level
    int j = 0;
    int currentSampleLevel = numberOfLevels;
    std::cout << "levels:";

    for (int q=0;q<numberOfLevels;q++) {
        std::cout << "\n  level " << q << " : " << N / powf(ratio, q);
    }
    for (int i = 0; i < numberOfSamplesForFirstLevel; i++) {
        if (i == N / (int)powf(ratio,  currentSampleLevel)) {
            currentSampleLevel--;
            std::cout << "\nNext sample level: " << currentSampleLevel;
        }

        rx.for_each_vertex(rxmesh::DEVICE,
                           [seed,
                               context,
                               sample_number,
                               sample_level_bitmask,
                               distance,
                               i,
                               currentSampleLevel,
                               sample_pos,
                               vertex_pos] __device__(
                           const rxmesh::VertexHandle vh) {
                               if (seed == context.linear_id(vh)) {
                                   sample_number(vh, 0) = i;
                                   //sample_number_point
                                   distance(vh, 0)      = 0;
                                   sample_pos[i].x        = vertex_pos(vh, 0);
                                   sample_pos[i].y      = vertex_pos(vh, 1);
                                   sample_pos[i].z        = vertex_pos(vh, 2);

                                   for (int k = 0; k < currentSampleLevel;
                                        k++) {
                                       sample_level_bitmask(vh, 0) |= (1 << k);
                                   }
                               } else {
                                   if (i == 0) {
                                       distance(vh, 0)      = INFINITY;
                                       sample_number(vh, 0) = -1;
                                   }
                               }
                           });

        do {
            hipDeviceSynchronize();
            *flagger = 0;
            sample_points<float, CUDABlockSize>
                <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                    rx.get_context(),
                    vertex_pos,
                    distance,
                    flagger);
            hipDeviceSynchronize();
            j++;

        } while (*flagger != 0);


        // reduction step
        farthestPoint = reducer.arg_max(distance, 0);
        seed          = rx.linear_id(farthestPoint.key);
    }

    std::cout << "\nSampling iterations: " << j; 


    sample_number.move(DEVICE, HOST);
    distance.move(DEVICE, HOST);
    sample_level_bitmask.move(DEVICE, HOST);


    rxmesh::LaunchBox<CUDABlockSize> cb;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        cb,
        (void*)cluster_points<float, CUDABlockSize>);



    //clustering step
    j = 0;
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [
            sample_number,
            sample_level_bitmask,
            distance,
            currentLevel,
            clustered_vertex, context, vertices, vertex_pos ]
        __device__(const rxmesh::VertexHandle vh) {

                vertices[context.linear_id(vh)].x = vertex_pos(vh, 0);
                vertices[context.linear_id(vh)].y = vertex_pos(vh, 1);
                vertices[context.linear_id(vh)].z = vertex_pos(vh, 2);

            //if (sample_number(vh, 0) > -1)
            if ((sample_level_bitmask(vh,0) & (1 << (currentLevel-1))) !=0)
            {
                clustered_vertex(vh, 0) = sample_number(vh, 0);
                distance(vh, 0)         = 0;
            } else {
                distance(vh, 0)         = INFINITY;
                clustered_vertex(vh, 0) = -1;
            }
        });

    do {
        hipDeviceSynchronize();
        *flagger = 0;
        cluster_points<float, CUDABlockSize>
            <<<lb.blocks, lb.num_threads, lb.smem_bytes_dyn>>>(
                rx.get_context(),
                vertex_pos,
                distance,
                clustered_vertex,
                flagger);
        hipDeviceSynchronize();
        j++;
    } while (*flagger != 0);

    clustered_vertex.move(DEVICE, HOST);
    std::cout << "\Clustering iterations: " << j;

    int* vertexCluster;
    hipMallocManaged(&vertexCluster, sizeof(int) * N);
    rx.for_each_vertex(rxmesh::DEVICE,
                       [
                           clustered_vertex,
                           context,vertexCluster] __device__(
                       const rxmesh::VertexHandle vh) {
                           vertexCluster[context.linear_id(vh)] = clustered_vertex(vh, 0);
                       });
    hipDeviceSynchronize();




    int* number_of_neighbors;
    hipMallocManaged(&number_of_neighbors, numberOfSamples * sizeof(int));
    for (int i = 0; i < numberOfSamples; i++) {
        number_of_neighbors[i] = 0;
    }
    hipDeviceSynchronize();

    rxmesh::LaunchBox<CUDABlockSize> nn;
    rx.prepare_launch_box(
        {rxmesh::Op::VV},
        nn,
        (void*)findNumberOfCoarseNeighbors<float, CUDABlockSize>);



    // Allocate memory for vertex neighbors
    VertexNeighbors* vertexNeighbors;
    hipError_t      err = hipMallocManaged(&vertexNeighbors,
                                             numberOfSamples * sizeof(
                                                 VertexNeighbors));

    findNumberOfCoarseNeighbors<float, CUDABlockSize>
        <<<nn.blocks, nn.num_threads, nn.smem_bytes_dyn>>>(
            rx.get_context(),
            clustered_vertex,
            number_of_neighbors,
            vertexNeighbors);
    hipDeviceSynchronize();


    rx.for_each_vertex(rxmesh::DEVICE,
                       [numberOfSamples,
                           context,
                           vertexNeighbors,
                           clustered_vertex,
                           sample_number,
                           number_of_neighbors] __device__(
                       const rxmesh::VertexHandle vh) {

                           if (clustered_vertex(vh, 0) ==
                               sample_number(vh, 0)) {
                               number_of_neighbors[clustered_vertex(vh, 0)] =
                                   vertexNeighbors[sample_number(vh, 0)].
                                   getNumberOfNeighbors();
                           }
                       });
    hipDeviceSynchronize();

    int num_rows = numberOfSamples; // Set this appropriately
    CSR csr(num_rows, number_of_neighbors, vertexNeighbors, N);

    hipDeviceSynchronize(); // Ensure data is synchronized before accessing

   
    //for debug purposes
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [sample_number,
            clustered_vertex,
            number_of_neighbors,
            number_of_neighbors_coarse,
            context] __device__(const rxmesh::VertexHandle vh) {
            number_of_neighbors_coarse(vh, 0) = number_of_neighbors[sample_number(vh, 0)];

        });


   
    std::vector<CSR> operatorsCSR;
    operatorsCSR.push_back(CSR(N));

    std::vector<CSR> prolongationOperatorCSR;  // work on allocating these for v cycle

    CSR firstOperator(N);

   createProlongationOperator(csr.num_rows,
                                csr.row_ptr,
                                csr.value_ptr,
                                csr.number_of_neighbors,
                                N,
                                vertexCluster,
                                vertices, sample_pos, firstOperator.value_ptr, firstOperator.data_ptr);

   prolongationOperatorCSR.push_back(firstOperator);

   std::cout << "\nFIRST OPERATOR:";
    firstOperator.printCSR();
    hipDeviceSynchronize();

    Eigen::MatrixXd verts;
    Eigen::MatrixXi faces;
    std::vector<std::array<double, 3>>
        vertexPositions;                            // To store vertex positions
        std::vector<std::vector<size_t>> faceIndices;  // To store face indices

    csr.GetRenderData(vertexPositions, faceIndices, sample_pos);

    polyscope::registerSurfaceMesh(
        "mesh level 1", vertexPositions, faceIndices);




    //set 1st level node data
    VertexData* oldVdata;
    hipMallocManaged(&oldVdata, sizeof(VertexData) * numberOfSamples);
    setVertexData(rx,
            context,
            oldVdata,
            vertex_pos,
            sample_number,
            sample_level_bitmask,
            clustered_vertex);

    createProlongationOperators(N,
                                numberOfSamples,
                                numberOfLevels,
                                ratio,
                                sample_pos,
                                csr,
                                prolongationOperatorCSR,
                                oldVdata,
                                distanceArray,
                                vertexCluster);

    //contruct equations as CSR matrices
    SparseMatrix<float> A_mat(rx);
    DenseMatrix<float>  B_mat(rx, rx.get_num_vertices(), 3);
    setupMCF(rx, A_mat, B_mat);
    A_mat.move(DEVICE, HOST);
    B_mat.move(DEVICE, HOST);
    CSR A_csr(A_mat,A_mat.row_ptr(),A_mat.col_idx(),A_mat.non_zeros());
    VectorCSR3D B_v(B_mat.rows());

    std::cout << "\nRHS:";
    std::cout << "\n Number of rows of B:"<<B_mat.rows();

    for (int i=0;i<B_mat.rows();i++) {
        B_v.vector[i*3] = B_mat(i, 0);
        B_v.vector[i*3+1] = B_mat(i, 1);
        B_v.vector[i*3+2] = B_mat(i, 2);
    }
    std::vector<CSR> equationsPerLevel;
    equationsPerLevel.push_back(A_csr);

    constructLHS(A_csr,
                 prolongationOperatorCSR,
                 equationsPerLevel,
                 numberOfLevels,
                 numberOfSamples,
                 ratio);

    hipDeviceSynchronize();

    
    GMGVCycle gmg(N);

    gmg.prolongationOperators = prolongationOperatorCSR;
    gmg.LHS                   = equationsPerLevel;
    gmg.RHS                   = B_v;
    //gmg.max_number_of_levels  = 1;  // numberOfLevels-1;
    gmg.max_number_of_levels  = 0;
    gmg.post_relax_iterations = 5;
    gmg.pre_relax_iterations = 5;
    gmg.ratio                 = ratio;


    std::cout << "\nNumber of equations LHS:" << gmg.LHS.size();
    std::cout << "\nNumber of operators:" << gmg.prolongationOperators.size();
    std::cout << "\nMax level:" << gmg.max_number_of_levels;


    int numberOfVCycles=2;
    for (int i=0;i<numberOfVCycles;i++)
        gmg.VCycle(gmg.LHS[0], gmg.RHS, gmg.X, 0);

    std::cout << "\nFinal output: \n";
    std::vector<std::array<double, 3>> vertexMeshPositions;
    vertexMeshPositions.resize(gmg.X.n);
    
    for (int i = 0; i < gmg.X.n; i++) {

        /*
        std::cout << " \n";
        std::cout << B_v.vector[3 * i] << " ";
        std::cout << B_v.vector[3 * i+1] << " ";
        std::cout << B_v.vector[3 * i+2] << " ";
        std::cout << " | ";
        std::cout << gmg.X.vector[3 * i] << " ";
        std::cout << gmg.X.vector[3 * i+1] << " ";
        std::cout << gmg.X.vector[3 * i+2] << " ";
        */


        vertexMeshPositions[i] = {
            gmg.X.vector[3*i], gmg.X.vector[i * 3 + 1], gmg.X.vector[i * 3 + 2]};
    }
    
     polyscope::registerSurfaceMesh("output mesh", vertexMeshPositions,
        rx.get_polyscope_mesh()->faces);

     
    auto polyscope_callback = [&]() mutable {

        ImGui::Begin("GMG Parameters");

        ImGui::InputInt("Number of Levels", &gmg.max_number_of_levels);
        ImGui::InputInt("Number of V cycles", &numberOfVCycles);
        ImGui::InputInt("Number of pre solve smoothing iterations", &gmg.pre_relax_iterations);
        ImGui::InputInt("Number of post solve smoothing iterations", &gmg.post_relax_iterations);
        ImGui::InputInt("Number of direct solve iterations", &gmg.directSolveIterations);
        ImGui::SliderFloat("Omega", &gmg.omega,0.0,1.0);


        if (ImGui::Button("Run V Cycles again")) {
            std::cout << "\n---------------NEW SOLVE INITIATED--------------------\n";
            gmg.X.reset();

            for (int i=0;i<numberOfVCycles;i++)
                gmg.VCycle(gmg.LHS[0], gmg.RHS, gmg.X, 0);


           // vertexMeshPositions.clear();
           // vertexMeshPositions.resize(gmg.X.n);

            for (int i = 0; i < gmg.X.n; i++)
            {
                /*
                std::cout << " \n";
                std::cout << B_v.vector[3 * i] << " ";
                std::cout << B_v.vector[3 * i + 1] << " ";
                std::cout << B_v.vector[3 * i + 2] << " ";
                std::cout << " | ";
                std::cout << gmg.X.vector[3 * i] << " ";
                std::cout << gmg.X.vector[3 * i + 1] << " ";
                std::cout << gmg.X.vector[3 * i + 2] << " ";
                */
                
                vertexMeshPositions[i] = {gmg.X.vector[3 * i],
                                          gmg.X.vector[3 * i + 1],
                                          gmg.X.vector[3 * i + 2]};
            }
            
            //polyscope::removeSurfaceMesh("output mesh");

            polyscope::registerSurfaceMesh("output mesh 2",
                                           vertexMeshPositions,
                                           rx.get_polyscope_mesh()->faces);
        }

        ImGui::End();
    };

    polyscope::state::userCallback = polyscope_callback;

    number_of_neighbors_coarse.move(DEVICE, HOST);

    rx.get_polyscope_mesh()->addVertexScalarQuantity("sample_number",
                                                     sample_number);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("distance", distance);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("sample_level_bitmask",
                                                     sample_level_bitmask);
    rx.get_polyscope_mesh()->addVertexScalarQuantity("clusterPoint",
                                                     clustered_vertex);
    rx.get_polyscope_mesh()->addVertexScalarQuantity(
        "number of neighbors", number_of_neighbors_coarse);



#if USE_POLYSCOPE
    polyscope::show();
#endif
}



    ////////////////////////////////////
/*
std::cout << "\n\n\n\n\n\n";
std::cout << "first\n";

CSR t1(3, 3);

VectorCSR3D vectorTest(3);
VectorCSR3D vectorResult(3);

for (int i=0;i<3;i++) {
    vectorTest.vector[i*3] = i+1;
    vectorTest.vector[i*3 + 1] = i + 1;
    vectorTest.vector[i*3+2] = i+1;
}

SpMV_CSR_3D(t1.row_ptr,
         t1.value_ptr,
         t1.data_ptr,
         vectorTest.vector,
         vectorResult.vector,
         3);

std::cout << "\nVector result: ";
for (int i=0;i<vectorResult.n;i++) {
    std::cout << "\n";
    std::cout << vectorResult.vector[i*3] << " ";
    std::cout << vectorResult.vector[i*3+1] << " ";
    std::cout << vectorResult.vector[i*3+2] << " ";
}
*/

    /*
std::cout << "\n\n\n\n\n\n";
std::cout << "first\n";

CSR t1(3, 3,3);

VectorCSR3D b(3);
VectorCSR3D vectorResult(3);
for (int i = 0; i < vectorResult.n * 3; i++) {
    vectorResult.vector[i] = 0.0f;
}
// Set right hand side - different for each component of each point
b.vector[0] = 4.0;  // Point 1 (x,y,z)
b.vector[1] = 2.0;
b.vector[2] = 1.0;

b.vector[3] = 3.0;  // Point 2 (x,y,z)
b.vector[4] = 5.0;
b.vector[5] = 2.0;

b.vector[6] = 1.0;  // Point 3 (x,y,z)
b.vector[7] = 2.0;
b.vector[8] = 6.0;


t1.printCSR();

gauss_jacobi_CSR_3D(t1, vectorResult.vector, b.vector, 50);

std::cout << "\nVector result: ";
for (int i = 0; i < vectorResult.n; i++) {
    std::cout << "\n";
    std::cout << vectorResult.vector[i * 3] << " ";
    std::cout << vectorResult.vector[i * 3 + 1] << " ";
    std::cout << vectorResult.vector[i * 3 + 2] << " ";
}

*/
//////////////////////////////////////////////