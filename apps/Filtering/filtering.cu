#include "hip/hip_runtime.h"
// Parallel version of
// Fleishman, Shachar, Iddo Drori, and Daniel Cohen-Or.
//"Bilateral mesh denoising." ACM SIGGRAPH 2003 Papers.2003. 950-953.

#include <omp.h>

#include "../common/openmesh_trimesh.h"
#include "gtest/gtest.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/export_tools.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/log.h"

struct arg
{
    std::string obj_file_name   = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder   = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id       = 0;
    uint32_t    num_filter_iter = 5;
    char**      argv;
    int         argc;
} Arg;

#include "filtering_openmesh.h"
#include "filtering_rxmesh.cuh"

TEST(App, Filtering)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);


    // Load mesh
    std::vector<std::vector<uint32_t>> Faces;
    std::vector<std::vector<dataT>>    Verts;
    ASSERT_TRUE(import_obj(Arg.obj_file_name, Verts, Faces));

    RXMeshStatic rxmesh_static(Faces, false);

    TriMesh input_mesh;
    ASSERT_TRUE(OpenMesh::IO::read_mesh(input_mesh, Arg.obj_file_name));


    // OpenMesh Impl
    rxmesh::RXMeshAttribute<dataT> ground_truth;
    size_t                         max_neighbour_size = 0;
    filtering_openmesh(
        omp_get_max_threads(), input_mesh, ground_truth, max_neighbour_size);


    // RXMesh Impl
    filtering_rxmesh(rxmesh_static, Verts, ground_truth, max_neighbour_size);

    // Release allocation
    ground_truth.release();
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;

    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: Filtering.exe < -option X>\n"
                        " -h:                Display this massage and exits\n"
                        " -input:            Input file. Input file should under the input/ subdirectory\n"
                        "                    Default is {} \n"
                        "                    Hint: Only accepts OBJ files\n"
                        " -o:                JSON file output folder. Default is {} \n"
                        " -num_filter_iter:  Iteration count. Default is {} \n"
                        " -device_id:        GPU device ID. Default is {}",
             Arg.obj_file_name, Arg.output_folder ,Arg.num_filter_iter ,Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-num_filter_iter")) {
            Arg.num_filter_iter =
                atoi(get_cmd_option(argv, argv + argc, "-num_filter_iter"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("num_filter_iter= {}", Arg.num_filter_iter);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}