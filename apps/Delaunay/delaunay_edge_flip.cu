#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/util/log.h"
#include "rxmesh/util/macros.h"

struct arg
{
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "torus.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    uint32_t    device_id     = 0;
    char**      argv;
    int         argc;
} Arg;

#include "delaunay_rxmesh.cuh"

TEST(Apps, DelaunayEdgeFlip)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);

    // RXMeshDynamic rx(Arg.obj_file_name);
    // rx.save(STRINGIFY(OUTPUT_DIR) "torus_patches");

    RXMeshDynamic rx(
        Arg.obj_file_name, false, STRINGIFY(INPUT_DIR) "torus_patches");

    ASSERT_TRUE(rx.is_edge_manifold());

    ASSERT_TRUE(delaunay_rxmesh(rx));
}


int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;


    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: DelaunayEdgeFlip.exe < -option X>\n"
                        " -h:          Display this massage and exit\n"
                        " -input:      Input file. Input file should be under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accept OBJ files\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -device_id:  GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}