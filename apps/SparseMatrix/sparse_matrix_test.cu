#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"
#include "rxmesh/attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/report.h"
#include "rxmesh/util/timer.h"
#include "sparse_matrix.cuh"

struct arg
{
    std::string obj_file_name = STRINGIFY(INPUT_DIR) "sphere3.obj";
    std::string output_folder = STRINGIFY(OUTPUT_DIR);
    uint32_t    num_run       = 1;
    uint32_t    device_id     = 0;
    char**      argv;
    int         argc;
} Arg;

TEST(Apps, SparseMatrix)
{
    using namespace rxmesh;
    using dataT = float;

    // Select device
    cuda_query(Arg.device_id);

    // Load mesh
    std::vector<std::vector<dataT>>    Verts;
    std::vector<std::vector<uint32_t>> Faces;

    ASSERT_TRUE(import_obj(Arg.obj_file_name, Verts, Faces));


    RXMeshStatic rxmesh(Faces, false);

    // TODO: fillin the spmat test

    SparseMatInfo<int> spmat(rxmesh);
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);
    Arg.argv = argv;
    Arg.argc = argc;

    if (argc > 1) {
        if (cmd_option_exists(argv, argc + argv, "-h")) {
            // clang-format off
            RXMESH_INFO("\nUsage: SparseMatrix.exe < -option X>\n"
                        " -h:          Display this massage and exit\n"
                        " -input:      Input file. Input file should be under the input/ subdirectory\n"
                        "              Default is {} \n"
                        "              Hint: Only accept OBJ files\n"
                        " -o:          JSON file output folder. Default is {} \n"
                        " -num_run:    Number of iterations for performance testing. Default is {} \n"                        
                        " -device_id:  GPU device ID. Default is {}",
            Arg.obj_file_name, Arg.output_folder, Arg.num_run, Arg.device_id);
            // clang-format on
            exit(EXIT_SUCCESS);
        }

        if (cmd_option_exists(argv, argc + argv, "-num_run")) {
            Arg.num_run = atoi(get_cmd_option(argv, argv + argc, "-num_run"));
        }

        if (cmd_option_exists(argv, argc + argv, "-input")) {
            Arg.obj_file_name =
                std::string(get_cmd_option(argv, argv + argc, "-input"));
        }
        if (cmd_option_exists(argv, argc + argv, "-o")) {
            Arg.output_folder =
                std::string(get_cmd_option(argv, argv + argc, "-o"));
        }
        if (cmd_option_exists(argv, argc + argv, "-device_id")) {
            Arg.device_id =
                atoi(get_cmd_option(argv, argv + argc, "-device_id"));
        }
    }

    RXMESH_TRACE("input= {}", Arg.obj_file_name);
    RXMESH_TRACE("output_folder= {}", Arg.output_folder);
    RXMESH_TRACE("num_run= {}", Arg.num_run);
    RXMESH_TRACE("device_id= {}", Arg.device_id);

    return RUN_ALL_TESTS();
}
