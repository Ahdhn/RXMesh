#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"
#include "matrix_operation.cuh"
#include "rxmesh/attribute.h"
#include "rxmesh/rxmesh_static.h"
#include "rxmesh/util/import_obj.h"
#include "rxmesh/util/report.h"
#include "rxmesh/util/timer.h"
#include "sparse_matrix_mcf.cuh"

template <uint32_t blockThreads, typename IndexT = int>
__global__ static void sparse_mat_test(const rxmesh::Context context,
                                       IndexT*               patch_ptr_v,
                                       IndexT*               vet_degree)
{
    using namespace rxmesh;

    auto init_lambda = [&](VertexHandle& v_id, const VertexIterator& iter) {
        // printf(" %" PRIu32 " - %" PRIu32 " - %" PRIu32 " - %" PRIu32 " \n",
        //        row_ptr[0],
        //        row_ptr[1],
        //        row_ptr[2],
        //        row_ptr[3]);
        auto     ids                                 = v_id.unpack();
        uint32_t patch_id                            = ids.first;
        uint16_t local_id                            = ids.second;
        vet_degree[patch_ptr_v[patch_id] + local_id] = iter.size() + 1;
    };

    query_block_dispatcher<Op::VV, blockThreads>(context, init_lambda);
}

template <uint32_t blockThreads>
__global__ static void sparse_mat_query_test(
    const rxmesh::Context      context,
    rxmesh::SparseMatInfo<int> sparse_mat)
{
    using namespace rxmesh;
    auto init_lambda = [&](VertexHandle& v_id, const VertexIterator& iter) {
        sparse_mat(v_id, v_id) = 2;
        for (uint32_t v = 0; v < iter.size(); ++v) {
            sparse_mat(v_id, iter[v]) = 2;
            sparse_mat(iter[v], v_id) = 2;
        }
    };

    query_block_dispatcher<Op::VV, blockThreads>(context, init_lambda);
}

template <typename T, uint32_t blockThreads>
__global__ static void sparse_mat_edge_len_test(
    const rxmesh::Context      context,
    rxmesh::VertexAttribute<T> coords,
    rxmesh::SparseMatInfo<T>   sparse_mat,
    T*                         arr_ref)
{
    using namespace rxmesh;
    auto init_lambda = [&](VertexHandle& v_id, const VertexIterator& iter) {
        // reference value calculation
        auto     r_ids      = v_id.unpack();
        uint32_t r_patch_id = r_ids.first;
        uint16_t r_local_id = r_ids.second;

        uint32_t row_index =
            sparse_mat.m_d_patch_ptr_v[r_patch_id] + r_local_id;

        arr_ref[row_index]     = 0;
        sparse_mat(v_id, v_id) = 0;

        Vector<3, T> v_coord(coords(v_id, 0), coords(v_id, 1), coords(v_id, 2));
        for (uint32_t v = 0; v < iter.size(); ++v) {
            Vector<3, T> vi_coord(
                coords(iter[v], 0), coords(iter[v], 1), coords(iter[v], 2));

            sparse_mat(v_id, iter[v]) = dist(v_coord, vi_coord);

            arr_ref[row_index] += dist(v_coord, vi_coord);
        }
    };

    query_block_dispatcher<Op::VV, blockThreads>(context, init_lambda);
}

template <typename T>
__global__ void spmat_multi_hardwired_kernel(
    T*                       vec,
    rxmesh::SparseMatInfo<T> sparse_mat,
    T*                       out,
    const int                N)
{
    int   tid = threadIdx.x + blockIdx.x * blockDim.x;
    float sum = 0;
    if (tid < N) {
        uint32_t start = sparse_mat.m_d_row_ptr[tid];
        uint32_t end   = sparse_mat.m_d_row_ptr[tid + 1];
        for (int i = 0; i < end - start; i++) {
            sum += vec[sparse_mat.m_d_col_idx[start + i]] *
                   sparse_mat.m_d_val[start + i];
        }
        out[tid] = sum;
    }
}

template <typename T, uint32_t blockThreads>
__global__ static void simple_A_X_B_setup(const rxmesh::Context      context,
                                          rxmesh::VertexAttribute<T> coords,
                                          rxmesh::SparseMatInfo<T>   A_mat,
                                          rxmesh::DenseMatInfo<T>    X_mat,
                                          rxmesh::DenseMatInfo<T>    B_mat,
                                          const T                    time_step)
{
    using namespace rxmesh;
    auto init_lambda = [&](VertexHandle& v_id, const VertexIterator& iter) {
        T sum_e_weight(0);

        T v_weight = iter.size();

        // reference value calculation
        auto     r_ids      = v_id.unpack();
        uint32_t r_patch_id = r_ids.first;
        uint16_t r_local_id = r_ids.second;

        uint32_t row_index = A_mat.m_d_patch_ptr_v[r_patch_id] + r_local_id;

        B_mat(row_index, 0) = threadIdx.x;
        B_mat(row_index, 1) = blockIdx.x;
        B_mat(row_index, 2) = row_index * 3;

        X_mat(row_index, 0) = coords(v_id, 0) * v_weight;
        X_mat(row_index, 1) = coords(v_id, 1) * v_weight;
        X_mat(row_index, 2) = coords(v_id, 2) * v_weight;

        Vector<3, float> vi_coord(
            coords(v_id, 0), coords(v_id, 1), coords(v_id, 2));
        for (uint32_t v = 0; v < iter.size(); ++v) {
            T e_weight           = 1;
            A_mat(v_id, iter[v]) = -time_step * e_weight;

            sum_e_weight += e_weight;
        }

        A_mat(v_id, v_id) = v_weight + time_step * sum_e_weight;
    };

    query_block_dispatcher<Op::VV, blockThreads>(context, init_lambda);
}


TEST(Apps, PatchPointer)
{
    using namespace rxmesh;

    cuda_query(0);

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    // move the patch ptr to the host so we can test it
    std::vector<uint32_t> h_ptchptr(rx.get_num_patches() + 1);

    SparseMatInfo<int> spmat(rx);

    // vertices
    CUDA_ERROR(hipMemcpy(h_ptchptr.data(),
                          spmat.m_d_patch_ptr_v,
                          h_ptchptr.size() * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    EXPECT_EQ(h_ptchptr.back(), rx.get_num_vertices());

    for (uint32_t i = 0; i < rx.get_num_patches(); ++i) {
        EXPECT_EQ(h_ptchptr[i + 1] - h_ptchptr[i],
                  rx.get_patches_info()[i].num_owned_vertices);
    }

    // edges
    CUDA_ERROR(hipMemcpy(h_ptchptr.data(),
                          spmat.m_d_patch_ptr_e,
                          h_ptchptr.size() * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    EXPECT_EQ(h_ptchptr.back(), rx.get_num_edges());

    for (uint32_t i = 0; i < rx.get_num_patches(); ++i) {
        EXPECT_EQ(h_ptchptr[i + 1] - h_ptchptr[i],
                  rx.get_patches_info()[i].num_owned_edges);
    }

    // faces
    CUDA_ERROR(hipMemcpy(h_ptchptr.data(),
                          spmat.m_d_patch_ptr_f,
                          h_ptchptr.size() * sizeof(uint32_t),
                          hipMemcpyDeviceToHost));
    EXPECT_EQ(h_ptchptr.back(), rx.get_num_faces());

    for (uint32_t i = 0; i < rx.get_num_patches(); ++i) {
        EXPECT_EQ(h_ptchptr[i + 1] - h_ptchptr[i],
                  rx.get_patches_info()[i].num_owned_faces);
    }

    spmat.free();
}

TEST(Apps, SparseMatrix)
{
    using namespace rxmesh;

    // Select device
    cuda_query(0);

    // generate rxmesh obj
    std::string  obj_path = STRINGIFY(INPUT_DIR) "dragon.obj";
    RXMeshStatic rxmesh(obj_path);

    uint32_t num_vertices = rxmesh.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    int* d_arr_ones;
    int* d_result;

    std::vector<int> init_tmp_arr(num_vertices, 1);
    CUDA_ERROR(hipMalloc((void**)&d_arr_ones, (num_vertices) * sizeof(int)));
    CUDA_ERROR(hipMemcpy(d_arr_ones,
                          init_tmp_arr.data(),
                          num_vertices * sizeof(int),
                          hipMemcpyHostToDevice));

    CUDA_ERROR(hipMalloc((void**)&d_result, (num_vertices) * sizeof(int)));

    SparseMatInfo<int> spmat(rxmesh);
    spmat.set_ones();

    spmat_multi_hardwired_kernel<<<blocks, threads>>>(
        d_arr_ones, spmat, d_result, num_vertices);

    std::vector<int> h_result(num_vertices);
    CUDA_ERROR(hipMemcpy(
        h_result.data(), d_result, num_vertices, hipMemcpyDeviceToHost));

    // get reference result
    int* vet_degree;
    CUDA_ERROR(hipMalloc((void**)&vet_degree, (num_vertices) * sizeof(int)));

    LaunchBox<threads> launch_box;
    rxmesh.prepare_launch_box(
        {Op::VV}, launch_box, (void*)sparse_mat_test<threads>);

    sparse_mat_test<threads><<<launch_box.blocks,
                               launch_box.num_threads,
                               launch_box.smem_bytes_dyn>>>(
        rxmesh.get_context(), spmat.m_d_patch_ptr_v, vet_degree);

    std::vector<int> h_vet_degree(num_vertices);
    CUDA_ERROR(hipMemcpy(
        h_vet_degree.data(), vet_degree, num_vertices, hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        EXPECT_EQ(h_result[i], h_vet_degree[i]);
    }


    CUDA_ERROR(hipFree(d_arr_ones));
    CUDA_ERROR(hipFree(d_result));
    CUDA_ERROR(hipFree(vet_degree));
    spmat.free();
}

TEST(Apps, SparseMatrixQuery)
{
    using namespace rxmesh;

    // Select device
    cuda_query(0);

    // generate rxmesh obj
    std::string  obj_path = STRINGIFY(INPUT_DIR) "cube.obj";
    RXMeshStatic rxmesh(obj_path);

    uint32_t num_vertices = rxmesh.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    SparseMatInfo<int> spmat(rxmesh);
    spmat.set_ones();

    LaunchBox<threads> launch_box;
    rxmesh.prepare_launch_box(
        {Op::VV}, launch_box, (void*)sparse_mat_query_test<threads>);

    sparse_mat_query_test<threads>
        <<<launch_box.blocks,
           launch_box.num_threads,
           launch_box.smem_bytes_dyn>>>(rxmesh.get_context(), spmat);

    std::vector<uint32_t> h_result(spmat.m_nnz);
    CUDA_ERROR(hipMemcpy(h_result.data(),
                          spmat.m_d_val,
                          spmat.m_nnz * sizeof(int),
                          hipMemcpyDeviceToHost));

    std::vector<uint32_t> h_ref(spmat.m_nnz, 2);

    for (int i = 0; i < spmat.m_nnz; ++i) {
        EXPECT_EQ(h_result[i], h_ref[i]);
    }

    spmat.free();
}

TEST(Apps, SparseMatrixEdgeLen)
{
    using namespace rxmesh;

    // Select device
    cuda_query(0);

    // generate rxmesh obj
    std::string  obj_path = STRINGIFY(INPUT_DIR) "dragon.obj";
    RXMeshStatic rxmesh(obj_path);

    uint32_t num_vertices = rxmesh.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto coords = rxmesh.get_input_vertex_coordinates();

    float* d_arr_ones;

    std::vector<float> init_tmp_arr(num_vertices, 1.f);
    CUDA_ERROR(hipMalloc((void**)&d_arr_ones, (num_vertices) * sizeof(float)));
    CUDA_ERROR(hipMemcpy(d_arr_ones,
                          init_tmp_arr.data(),
                          num_vertices * sizeof(float),
                          hipMemcpyHostToDevice));

    SparseMatInfo<float> spmat(rxmesh);

    float* d_arr_ref;
    float* d_result;

    CUDA_ERROR(hipMalloc((void**)&d_arr_ref, (num_vertices) * sizeof(float)));
    CUDA_ERROR(hipMalloc((void**)&d_result, (num_vertices) * sizeof(float)));

    LaunchBox<threads> launch_box;
    rxmesh.prepare_launch_box(
        {Op::VV}, launch_box, (void*)sparse_mat_edge_len_test<float, threads>);

    sparse_mat_edge_len_test<float, threads><<<launch_box.blocks,
                                               launch_box.num_threads,
                                               launch_box.smem_bytes_dyn>>>(
        rxmesh.get_context(), *coords, spmat, d_arr_ref);

    // Spmat matrix multiply

    spmat_multi_hardwired_kernel<float>
        <<<blocks, threads>>>(d_arr_ones, spmat, d_result, num_vertices);

    // copy the value back to host
    std::vector<float> h_arr_ref(num_vertices);
    CUDA_ERROR(hipMemcpy(h_arr_ref.data(),
                          d_arr_ref,
                          num_vertices * sizeof(float),
                          hipMemcpyDeviceToHost));

    std::vector<float> h_result(num_vertices);
    CUDA_ERROR(hipMemcpy(h_result.data(),
                          d_result,
                          num_vertices * sizeof(float),
                          hipMemcpyDeviceToHost));

    for (uint32_t i = 0; i < num_vertices; ++i) {
        // printf("Idx: %" PRIu32 " %f %f \n", i, h_result[i], h_arr_ref[i]);
        EXPECT_FLOAT_EQ(h_result[i], h_arr_ref[i]);
    }

    CUDA_ERROR(hipFree(d_arr_ref));
    CUDA_ERROR(hipFree(d_arr_ones));
    CUDA_ERROR(hipFree(d_result));
    spmat.free();
}

TEST(Apps, SparseMatrixSimpleSolve)
{
    using namespace rxmesh;

    // Select device
    cuda_query(0);

    // generate rxmesh obj
    std::string  obj_path = STRINGIFY(INPUT_DIR) "cube.obj";
    RXMeshStatic rxmesh(obj_path);

    uint32_t num_vertices = rxmesh.get_num_vertices();

    const uint32_t threads = 256;
    const uint32_t blocks  = DIVIDE_UP(num_vertices, threads);

    auto                 coords = rxmesh.get_input_vertex_coordinates();
    SparseMatInfo<float> A_mat(rxmesh);
    DenseMatInfo<float>  X_mat(num_vertices, 3);
    DenseMatInfo<float>  B_mat(num_vertices, 3);

    float time_step = 1.f;

    LaunchBox<threads> launch_box;
    rxmesh.prepare_launch_box(
        {Op::VV}, launch_box, (void*)simple_A_X_B_setup<float, threads>);

    simple_A_X_B_setup<float, threads><<<launch_box.blocks,
                                         launch_box.num_threads,
                                         launch_box.smem_bytes_dyn>>>(
        rxmesh.get_context(), *coords, A_mat, X_mat, B_mat, time_step);

    spmat_linear_solve(A_mat, X_mat, B_mat, Solver::CHOL, Reorder::NONE);

    /* Wrap raw data into cuSPARSE generic API objects */
    hipsparseSpMatDescr_t matA = NULL;
    hipsparseCreateCsr(&matA,
                      A_mat.m_row_size,
                      A_mat.m_col_size,
                      A_mat.m_nnz,
                      A_mat.m_d_row_ptr,
                      A_mat.m_d_col_idx,
                      A_mat.m_d_val,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO,
                      HIP_R_64F);

    hipsparseDnVecDescr_t vecx = NULL;

    hipsparseCreateDnVec(&vecx, A_mat.m_col_size, X_mat.data(), HIP_R_64F);
    hipsparseDnVecDescr_t vecAx = NULL;
    hipsparseCreateDnVec(&vecAx, A_mat.m_row_size, B_mat.data(), HIP_R_64F);

    // const double minus_one  = -1.0;
    // const double one        = 1.0;
    // size_t       bufferSize = 0;
    // checkCudaErrors(hipsparseSpMV_bufferSize(cusparseHandle,
    //                                         HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                                         &minus_one,
    //                                         matA,
    //                                         vecx,
    //                                         &one,
    //                                         vecAx,
    //                                         HIP_R_64F,
    //                                         HIPSPARSE_SPMV_ALG_DEFAULT,
    //                                         &bufferSize));
    // void* buffer = NULL;
    // checkCudaErrors(hipMalloc(&buffer, bufferSize));

    // checkCudaErrors(hipsparseSpMV(cusparseHandle,
    // HIPSPARSE_OPERATION_NON_TRANSPOSE,
    //                            &minus_one, matA, vecx, &one, vecAx,
    //                            HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
    //                            buffer));
}

int main(int argc, char** argv)
{
    using namespace rxmesh;
    Log::init();

    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
